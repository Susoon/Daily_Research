#include "hip/hip_runtime.h"
#include "ipsec.h"
#include "packet_man.h"
#include "gdnio.h"

#define TOTAL_T_NUM 4 * 7 * 128
#define AES_T_NUM 7
#define PPB 128
#define HMAC_T_NUM 2
#define PKT_SIZE 128
#define PAD_LEN 0


__device__ void sha1_kernel_global_128(unsigned char *data, sha1_gpu_context *ctx, unsigned int *extended, int len, int pkt_idx)
{
	int thread_index = threadIdx.x%AES_T_NUM;
	
	if(thread_index >= HMAC_T_NUM)
		return;

	int e_index = thread_index * 80;
	int block_index = thread_index * 64;
	unsigned int temp, t;

	if(thread_index == 0){
		/* Initialization vector for SHA-1 */
		ctx->state[0] = 0x67452301;           
		ctx->state[1] = 0xEFCDAB89;           
		ctx->state[2] = 0x98BADCFE;           
		ctx->state[3] = 0x10325476;           
		ctx->state[4] = 0xC3D2E1F0; 
	}
	__syncthreads();

	/*
	 * Extend 32 block byte block into 80 byte block.
	 */

//sh_kim 20.03.11 : when data length is 20byte, we need padding
	if(len == 20 && threadIdx.x == 0)
	{
		memset(data + len - 1, 0, 44);
	}

	GET_UINT32_BE( extended[pkt_idx*e_index + 0], data + block_index,  0 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 1], data + block_index,  4 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 2], data + block_index,  8 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 3], data + block_index, 12 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 4], data + block_index, 16 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 5], data + block_index, 20 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 6], data + block_index, 24 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 7], data + block_index, 28 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 8], data + block_index, 32 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 9], data + block_index, 36 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 10], data + block_index, 40 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 11], data + block_index, 44 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 12], data + block_index, 48 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 13], data + block_index, 52 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 14], data + block_index, 56 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 15], data + block_index, 60 );

	// Same as "blk(i)" macro in openssl source.
	for (t = 16; t < 80; t++) {
		temp = extended[pkt_idx*e_index + t - 3] ^ extended[pkt_idx*e_index + t - 8] ^ extended[pkt_idx*e_index + t - 14] ^ extended[pkt_idx*e_index + t - 16];
		extended[pkt_idx*e_index + t] = S(temp,1);
	}

	__syncthreads();
	if(thread_index == 0){
		for(t = 0; t < HMAC_T_NUM; t++) 
			sha1_gpu_process(ctx, (unsigned int*)&extended[pkt_idx * t * 80]);
	}
}

//SHKIM, ipsec_128 ver.
__global__ void nf_ipsec_128(struct pkt_buf *p_buf, int* pkt_cnt, unsigned int* ctr, unsigned char* d_nounce, unsigned int* d_key, unsigned char* d_sbox, unsigned char* d_GF2, int chain_seq, unsigned int* seq, unsigned int* extended)
{
	// <<< 4, 896 >>> threads. 
	//	7 threads for 1 pkt. (124B pkt)
	// 896 / 7 = 128, 1TB has 896 threads each and manages 128 pkts.
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int cur_tid = threadIdx.x / AES_T_NUM;
	// 4 x 896 = 3,584
	// tid : 0 - 3,583 (3,584 threads)

	__shared__ unsigned char IV[PPB][16];
	__shared__ unsigned char aes_tmp[PPB][16*AES_T_NUM]; 
	__shared__ sha1_gpu_context ictx[PPB];
	__shared__ sha1_gpu_context octx[PPB];
	// IV : 128 * 16 =  2,048
	// aes_tmp : 128 * 16 * 7 = 14,336
	// ictx : 24 * 128 = 3,072
	// octx : 24 * 128 = 3,072
	//-------------------------- Total __shared__ mem Usage : 22,528 / 49,152 (48KB per TB)

	if(tid == TOTAL_T_NUM - 1){
		START_RED
		printf("[%s] tid %d is alive!\n", __FUNCTION__, tid);
		END
	}

	__syncthreads();

	while(true){ // Persistent Kernel (for every threads)
		// 7-threads to be grouped. 
		//-------------------------- Multi threads Job --------------------------------------------
		if(readNoCache(&p_buf->rx_buf_idx[tid/AES_T_NUM]) == chain_seq){

			//-------------------------- Single threads Job --------------------------------------------
			if(tid % AES_T_NUM == 0){ 
				///////////////////// ESP Tailer, padlen, next-hdr /////////////////////////
#if PAD_LEN
				int i;
				for(i = 1; i <= PAD_LEN; i++)
					p_buf->rx_buf[0x1000 * (tid/AES_T_NUM) + i] = 0; // padding
#endif	
		
				p_buf->rx_buf[0x1000 * (tid/AES_T_NUM) + (PKT_SIZE - 4) + PAD_LEN] = PAD_LEN; // padlen 

				p_buf->rx_buf[0x1000 * (tid/AES_T_NUM) + (PKT_SIZE - 4) + PAD_LEN + 1] = IPPROTO_IPIP; // next-hdr (Meaning "IP within IP)

				/* For Reference...
					 IPPROTO_IP = 0
					 IPPROTO_ICMP = 1
					 IPPROTO_IPIP = 4
					 IPPROTO_TCP = 6
					 IPPROTO_UDP = 17
					 IPPROTO_ESP = 50
				 */
				atomicAdd(ctr, 1); // same "ctr" value for grouped 7-threads. (counter) AES-CTR Mode
				IV[cur_tid][15] = *ctr & 0xFF;
				IV[cur_tid][14] = (*ctr >> 8) & 0xFF; // CKJUNG, 1 Byte = 8bits means, Octal notation
				IV[cur_tid][13] = (*ctr >> 16) & 0xFF;
				IV[cur_tid][12] = (*ctr >> 24) & 0xFF;
				for(int i = 0; i < 12; i++)
					IV[cur_tid][i] = 0;

				// Copy our state into private memory
				unsigned char temp, temp2;
				unsigned char overflow = 0;
				char tmp[16];
				for(int i = 15; i != -1; i--) {
					temp = d_nounce[i];
					temp2 = IV[cur_tid][i];
					IV[cur_tid][i] += temp + overflow;
					overflow = ((int)temp2 + (int)temp + (int)overflow > 255);
				}

				AddRoundKey(IV[cur_tid], &d_key[0]);

				for(int i = 1; i < 10; i++)
				{
					SubBytes(IV[cur_tid], d_sbox);
					ShiftRows(IV[cur_tid]);
					MixColumns(IV[cur_tid], d_GF2, tmp);
					AddRoundKey(IV[cur_tid], &d_key[4 * i]);
				}
				SubBytes(IV[cur_tid], d_sbox);
				ShiftRows(IV[cur_tid]);
				AddRoundKey(IV[cur_tid], &d_key[4 * 10]);
			}
			__syncthreads();
			//-------------------------- Multi threads Job --------------------------------------------
			////////////////// Locating AES Encrypted parts into a pkt  ///////////////////////////////
			for(int i = 0; i < 16; i++){
				aes_tmp[cur_tid][((tid%AES_T_NUM)*16) + i] = p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM)) + sizeof(struct ethhdr) + ((tid%AES_T_NUM) * 16) + i] ^ IV[cur_tid][i];
			}
			for(int i = 0; i < 16; i++){
				p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM)) + sizeof(struct ethhdr) + sizeof(struct iphdr) + sizeof(struct esphdr) + ((tid%AES_T_NUM) * 16) + i] = aes_tmp[cur_tid][((tid%AES_T_NUM)*16) + i]; 
			}
			__syncthreads();
#if 1
			//-------------------------- Single threads Job --------------------------------------------
			if(tid % AES_T_NUM == 0){
			//////////// Proto_type = ESP set! ///////////
				p_buf->rx_buf[0x1000 * (tid/AES_T_NUM) + sizeof(struct ethhdr) + 9] = IPPROTO_ESP; // IPPROTO_ESP = 50
				struct ethhdr* ethh;
				struct iphdr* iph;
				struct esphdr* esph;

				ethh = (struct ethhdr *)&p_buf->rx_buf[0x1000 * (tid/AES_T_NUM)];
				iph = (struct iphdr *)(ethh + 1);
				esph = (struct esphdr *)((uint32_t *)iph + iph->ihl);

				// SPI (Security Parameter Index)
				uint32_t spi = 1085899777;
				HTONS32(spi);

				////////// Set ESP header SPI value ///////////////////
				memcpy(&esph->spi, &spi, 4);
				atomicAdd(seq, 1);

				//////////// Set ESP header SEQ value //////////
				memcpy(&esph->seq, seq, 4);
			}
			__syncthreads();
#endif
				// CKJUNG, HMAC-SHA1 From here! /////////////////////////////
				// RFC 2104, H(K XOR opad, H(K XOR ipad, text))
				/**** Inner Digest ****/
				// H(K XOR ipad, text) : 64 Bytes
				sha1_kernel_global_128(&p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM)) + sizeof(struct ethhdr) + sizeof(struct iphdr)], &ictx[cur_tid], extended, 64, (tid/AES_T_NUM));
				/**** Outer Digest ****/
				// H(K XOR opad, H(K XOR ipad, text)) : 20 Bytes
				sha1_kernel_global_128(&(ictx[cur_tid].c_state[0]), &octx[cur_tid], extended, 20, (tid/AES_T_NUM));
			__syncthreads();
			//-------------------------- Multi threads Job --------------------------------------------
			// Attach 20-bytes HMAC-SHA authentication digest to packet.
			if(tid % AES_T_NUM < 3)
				memcpy(&p_buf->rx_buf[0x1000 * (tid/AES_T_NUM) + (PKT_SIZE - 4) + 30 + ((tid%AES_T_NUM) * 8)], &(octx[cur_tid].c_state[((tid%AES_T_NUM)*8)]), 8);
			__syncthreads();
			//-------------------------- Single threads Job --------------------------------------------
			if(tid % AES_T_NUM == 0){
				atomicAdd(&pkt_cnt[1], 1);	
				p_buf->rx_buf_idx[tid/AES_T_NUM] = chain_seq+1;
			}
		}
		__syncthreads();
	}
}
