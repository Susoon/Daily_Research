#include "hip/hip_runtime.h"
#include "ipsec.h"
#include "packet_man.h"
#include "gdnio.h"

extern struct pkt_buf *p_buf;
extern int *pkt_cnt;
extern unsigned int *ctr;

__device__ void AddRoundKey(unsigned char *state, unsigned *w)
{
	int i;                                                              
	for(i = 0; i < BLOCK_SIZE; i++) { // column
		state[i * 4 + 0] = state[i * 4 + 0] ^ ((w[i] >> (8 * 3)) & 0xFF);
		state[i * 4 + 1] = state[i * 4 + 1] ^ ((w[i] >> (8 * 2)) & 0xFF);
		state[i * 4 + 2] = state[i * 4 + 2] ^ ((w[i] >> (8 * 1)) & 0xFF);
		state[i * 4 + 3] = state[i * 4 + 3] ^ ((w[i] >> (8 * 0)) & 0xFF);
	}                                                                   
}

__device__ void SubBytes(unsigned char *state, unsigned char* sbox) //state = 16 chars
{ 
	int i;
	for(i = 0; i < 4 * BLOCK_SIZE; i++) {
		state[i] = sbox[state[i]];
	}
} 

__device__ void ShiftRows(unsigned char *state)
{ 
	// NOTE: For whatever reason the standard uses column-major ordering ?
	// 0 1 2 3 --> 0 1 2 3  | 0  4  8  12 --> 0   4  8 12
	// 0 1 2 3 --> 1 2 3 0  | 1  5  9  13 --> 5   9 13  1
	// 0 1 2 3 --> 2 3 0 1  | 2  6  10 14 --> 10 14  2  6
	// 0 1 2 3 --> 3 0 1 2  | 3  7  11 15 --> 15  3  7 11
	unsigned char temp = state[1];

	state[1] = state[5];
	state[5] = state[9];
	state[9] = state[13];
	state[13] = temp;

	temp = state[2];
	state[2] = state[10];
	state[10] = temp;
	temp = state[6];
	state[6] = state[14];
	state[14] = temp;

	temp = state[3];
	state[3] = state[15];
	state[15] = state[11];
	state[11] = state[7];
	state[7] = temp;
}

// See "Efficient Software Implementation of AES on 32-bit platforms"
__device__ void MixColumns(unsigned char *state, unsigned char* GF_2, char* s) 
{
//[TODO] malloc!!!!!! is the criminal!!! CKJUNG, 18.10.26 
	memcpy(s, state, 4 * BLOCK_SIZE);
	int i;
#if 1
	for(i = 0; i < BLOCK_SIZE; i++) { // column
		unsigned char * x = (unsigned char*)&s[i*4];
		unsigned char * y = (unsigned char*)&state[i*4];
		y[0] = x[1] ^ x[2] ^ x[3];
		y[1] = x[0] ^ x[2] ^ x[3];
		y[2] = x[0] ^ x[1] ^ x[3];
		y[3] = x[0] ^ x[1] ^ x[2];
		x[0] = GF_2[x[0]];
		x[1] = GF_2[x[1]];
		x[2] = GF_2[x[2]];
		x[3] = GF_2[x[3]];
		y[0] ^= x[0] ^ x[1];
		y[1] ^= x[1] ^ x[2];
		y[2] ^= x[2] ^ x[3];
		y[3] ^= x[3] ^ x[0];
	}
#endif
} 

/**                                           
 * Initialize new context                      
 *                                             
 * @param context SHA1-Context                 
 */                                            
/*
 * Process extended block.
 */
__device__ void sha1_gpu_process (sha1_gpu_context *ctx, uint32_t W[80])
{
	uint32_t A, B, C, D, E;
	A = ctx->state[0];
	B = ctx->state[1];
	C = ctx->state[2];
	D = ctx->state[3];
	E = ctx->state[4];

#define P(a,b,c,d,e,x)\
{\
	e += S(a,5) + F(b,c,d) + K + x; b = S(b,30);\
}


#define F(x,y,z) (z ^ (x & (y ^ z)))
#define K 0x5A827999

	P( A, B, C, D, E, W[0]  );
	P( E, A, B, C, D, W[1]  );
	P( D, E, A, B, C, W[2]  );
	P( C, D, E, A, B, W[3]  );
	P( B, C, D, E, A, W[4]  );
	P( A, B, C, D, E, W[5]  );
	P( E, A, B, C, D, W[6]  );
	P( D, E, A, B, C, W[7]  );
	P( C, D, E, A, B, W[8]  );
	P( B, C, D, E, A, W[9]  );
	P( A, B, C, D, E, W[10] );
	P( E, A, B, C, D, W[11] );
	P( D, E, A, B, C, W[12] );
	P( C, D, E, A, B, W[13] );
	P( B, C, D, E, A, W[14] );
	P( A, B, C, D, E, W[15] );
	P( E, A, B, C, D, W[16] );
	P( D, E, A, B, C, W[17] );
	P( C, D, E, A, B, W[18] );
	P( B, C, D, E, A, W[19] );

#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0x6ED9EBA1

	P( A, B, C, D, E, W[20] );
	P( E, A, B, C, D, W[21] );
	P( D, E, A, B, C, W[22] );
	P( C, D, E, A, B, W[23] );
	P( B, C, D, E, A, W[24] );
	P( A, B, C, D, E, W[25] ); // w[25] is the problem.
	P( E, A, B, C, D, W[26] );
	P( D, E, A, B, C, W[27] );
	P( C, D, E, A, B, W[28] );
	P( B, C, D, E, A, W[29] );
	P( A, B, C, D, E, W[30] );
	P( E, A, B, C, D, W[31] );
	P( D, E, A, B, C, W[32] );
	P( C, D, E, A, B, W[33] );
	P( B, C, D, E, A, W[34] );
	P( A, B, C, D, E, W[35] );
	P( E, A, B, C, D, W[36] );
	P( D, E, A, B, C, W[37] );
	P( C, D, E, A, B, W[38] );
	P( B, C, D, E, A, W[39] );


#undef K
#undef F

#define F(x,y,z) ((x & y) | (z & (x | y)))
#define K 0x8F1BBCDC

	P( A, B, C, D, E, W[40] );
	P( E, A, B, C, D, W[41] );
	P( D, E, A, B, C, W[42] );
	P( C, D, E, A, B, W[43] );
	P( B, C, D, E, A, W[44] );
	P( A, B, C, D, E, W[45] );
	P( E, A, B, C, D, W[46] );
	P( D, E, A, B, C, W[47] );
	P( C, D, E, A, B, W[48] );
	P( B, C, D, E, A, W[49] );
	P( A, B, C, D, E, W[50] );
	P( E, A, B, C, D, W[51] );
	P( D, E, A, B, C, W[52] );
	P( C, D, E, A, B, W[53] );
	P( B, C, D, E, A, W[54] );
	P( A, B, C, D, E, W[55] );
	P( E, A, B, C, D, W[56] );
	P( D, E, A, B, C, W[57] );
	P( C, D, E, A, B, W[58] );
	P( B, C, D, E, A, W[59] );

#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0xCA62C1D6

	P( A, B, C, D, E, W[60] );
	P( E, A, B, C, D, W[61] );
	P( D, E, A, B, C, W[62] );
	P( C, D, E, A, B, W[63] );
	P( B, C, D, E, A, W[64] );
	P( A, B, C, D, E, W[65] );
	P( E, A, B, C, D, W[66] );
	P( D, E, A, B, C, W[67] );
	P( C, D, E, A, B, W[68] );
	P( B, C, D, E, A, W[69] );
	P( A, B, C, D, E, W[70] );
	P( E, A, B, C, D, W[71] );
	P( D, E, A, B, C, W[72] );
	P( C, D, E, A, B, W[73] );
	P( B, C, D, E, A, W[74] );
	P( A, B, C, D, E, W[75] );
	P( E, A, B, C, D, W[76] );
	P( D, E, A, B, C, W[77] );
	P( C, D, E, A, B, W[78] );
	P( B, C, D, E, A, W[79] );
#undef K
#undef F

	ctx->state[0] += A;
	ctx->state[1] += B;
	ctx->state[2] += C;
	ctx->state[3] += D;
	ctx->state[4] += E;
}


__device__ void sha1_kernel_global_1514(unsigned char *data, sha1_gpu_context *ctx, unsigned int *extended, int len, int pkt_idx)
{
	int thread_index = threadIdx.x%94;
	
	if(thread_index >= 24)
		return;

	int e_index = thread_index * 80;
	int block_index = thread_index * 64;
	unsigned int temp, t;

	if(thread_index == 0){
		/* Initialization vector for SHA-1 */
		ctx->state[0] = 0x67452301;           
		ctx->state[1] = 0xEFCDAB89;           
		ctx->state[2] = 0x98BADCFE;           
		ctx->state[3] = 0x10325476;           
		ctx->state[4] = 0xC3D2E1F0; 
	}
	__syncthreads();

	/*
	 * Extend 32 block byte block into 80 byte block.
	 */

//sh_kim 20.03.11 : when data length is 20byte, we need padding
	if(len == 20 && threadIdx.x = 0)
	{
		memset(data + len - 1, 0, 44);
	}

	GET_UINT32_BE( extended[pkt_idx*e_index + 0], data + block_index,  0 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 1], data + block_index,  4 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 2], data + block_index,  8 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 3], data + block_index, 12 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 4], data + block_index, 16 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 5], data + block_index, 20 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 6], data + block_index, 24 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 7], data + block_index, 28 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 8], data + block_index, 32 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 9], data + block_index, 36 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 10], data + block_index, 40 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 11], data + block_index, 44 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 12], data + block_index, 48 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 13], data + block_index, 52 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 14], data + block_index, 56 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 15], data + block_index, 60 );

	// Same as "blk(i)" macro in openssl source.
	for (t = 16; t < 80; t++) {
		temp = extended[pkt_idx*e_index + t - 3] ^ extended[pkt_idx*e_index + t - 8] ^ extended[pkt_idx*e_index + t - 14] ^ extended[pkt_idx*e_index + t - 16];
		extended[pkt_idx*e_index + t] = S(temp,1);
	}

	__syncthreads();
	if(thread_index == 0){
		for(t = 0; t < 24; t++) 
			sha1_gpu_process(ctx, (unsigned int*)&extended[pkt_idx * t * 80]);
	}
}


__device__ void sha1_kernel_global(unsigned char *data, sha1_gpu_context *ctx, uint32_t *extended, int len)
{
#if 1
	/* Initialization vector for SHA-1 */
	ctx->state[0] = 0x67452301;           
	ctx->state[1] = 0xEFCDAB89;           
	ctx->state[2] = 0x98BADCFE;           
	ctx->state[3] = 0x10325476;           
	ctx->state[4] = 0xC3D2E1F0;           
#endif

	uint32_t temp, t;

	/*
	 * Extend 32 block byte block into 80 byte block.
	 */

//sh_kim 20.03.11 : when data length is 20byte, we need padding
	if(len == 20)
	{
		memset(data + len - 1, 0, 44);
	}

	GET_UINT32_BE( extended[0], data,  0 );
	GET_UINT32_BE( extended[1], data,  4 );
	GET_UINT32_BE( extended[2], data,  8 );
	GET_UINT32_BE( extended[3], data, 12 );
	GET_UINT32_BE( extended[4], data, 16 );
	GET_UINT32_BE( extended[5], data, 20 );
	GET_UINT32_BE( extended[6], data, 24 );
	GET_UINT32_BE( extended[7], data, 28 );
	GET_UINT32_BE( extended[8], data, 32 );
	GET_UINT32_BE( extended[9], data, 36 );
	GET_UINT32_BE( extended[10], data, 40 );
	GET_UINT32_BE( extended[11], data, 44 );
	GET_UINT32_BE( extended[12], data, 48 );
	GET_UINT32_BE( extended[13], data, 52 );
	GET_UINT32_BE( extended[14], data, 56 );
	GET_UINT32_BE( extended[15], data, 60 );

	// Same as "blk(i)" macro in openssl source.
	for (t = 16; t < 80; t++) {
		temp = extended[t - 3] ^ extended[t - 8] ^ extended[t - 14] ^ extended[t - 16];
		extended[t] = S(temp,1);
	}

	sha1_gpu_process(ctx, extended);
}

// CKJUNG, 18.10.26 [NF#2:IPSec]-------------------------------------
__global__ void nf_ipsec_64(struct pkt_buf *p_buf, int* pkt_cnt, unsigned int* ctr, unsigned char* d_nounce, unsigned int* d_key, unsigned char* d_sbox, unsigned char* d_GF2, int chain_seq, unsigned int* seq)
{
	// <<< 4, 384 >>> threads. 
	//	3 threads for 1 pkt. (60B pkt)
	// 384 / 3 = 128, 1TB has 384 threads each and manages 128 pkts.
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	// tid : 0 - 1,535 (1,536 threads) = 512 * 3

	__shared__ unsigned char IV[128][16];
	__shared__ unsigned char aes_tmp[128][16*3]; 
	__shared__ sha1_gpu_context octx[128];
	// IV : 128 * 16 =  2,048 
	// aes_tmp : 128 * 16 * 3 = 6,144
	// ictx : 24 * 128 = 3,072
	// octx : 24 * 128 = 3,072
	// pkt_len : 4 * 128 = 512
	//-------------------------- Total __shared__ mem Usage : 14,336 + 512

	if(tid == 1535){
		START_RED
		printf("[%s] tid %d is alive!\n", __FUNCTION__, tid);
		END
	}

	__syncthreads();
	while(true){ // Persistent Kernel (for every threads)
		// 3-threads to be grouped. ex) 0,1,2 --> idx 0; 3,4,5 --> idx 1; ...
		//-------------------------- Multi threads Job --------------------------------------------
		__syncthreads();
		if(readNoCache(&p_buf->rx_buf_idx[tid/3]) == chain_seq){
			//-------------------------- Single threads Job --------------------------------------------
			//__syncthreads();
#if 1
			if(tid % 3 == 0){
				p_buf->rx_buf[0x1000 * (tid/3) + 60] = 0; // padlen 
				p_buf->rx_buf[0x1000 * (tid/3) + 60 + 1] = IPPROTO_IPIP; // next-hdr (Meaning "IP within IP)
				/* For Reference...
					 IPPROTO_IP = 0
					 IPPROTO_ICMP = 1
					 IPPROTO_IPIP = 4
					 IPPROTO_TCP = 6
					 IPPROTO_UDP = 17
					 IPPROTO_ESP = 50
				 */
				atomicAdd(ctr, 1); // same "ctr" value for grouped 3-threads. (counter) AES-CTR Mode
				IV[threadIdx.x/3][15] = *ctr & 0xFF;
				IV[threadIdx.x/3][14] = (*ctr >> 8) & 0xFF; // CKJUNG, 1 Byte = 8bits means, Octal notation
				IV[threadIdx.x/3][13] = (*ctr >> 16) & 0xFF;
				IV[threadIdx.x/3][12] = (*ctr >> 24) & 0xFF;
				for(int i = 0; i < 12; i++)
					IV[threadIdx.x/3][i] = 0;

				// Copy our state into private memory
				unsigned char temp, temp2;
				unsigned char overflow = 0;
				char tmp[16];
				for(int i = 15; i != -1; i--) {
					temp = d_nounce[i];
					temp2 = IV[threadIdx.x/3][i];
					IV[threadIdx.x/3][i] += temp + overflow;
					overflow = ((int)temp2 + (int)temp + (int)overflow > 255);
				}

				AddRoundKey(IV[threadIdx.x/3], &d_key[0]);

#if 1
				for(int i = 1; i < 10; i++)
				{
					SubBytes(IV[threadIdx.x/3], d_sbox);
					ShiftRows(IV[threadIdx.x/3]);
					MixColumns(IV[threadIdx.x/3], d_GF2, tmp);
					AddRoundKey(IV[threadIdx.x/3], &d_key[4 * i]);
				}
#endif
				SubBytes(IV[threadIdx.x/3], d_sbox);
				ShiftRows(IV[threadIdx.x/3]);
				AddRoundKey(IV[threadIdx.x/3], &d_key[4 * 10]);
			}
#endif
			__syncthreads();
			//-------------------------- Multi threads Job --------------------------------------------
			////////////////// Locating AES Encrypted parts into a pkt  ///////////////////////////////
			for(int i = 0; i < 16; i++){
				aes_tmp[threadIdx.x/3][((tid%3)*16) + i] = p_buf->rx_buf[(0x1000 * (tid/3)) + sizeof(struct ethhdr) + ((tid%3) * 16) + i] ^ IV[threadIdx.x/3][i];
			}
			for(int i = 0; i < 16; i++){
				p_buf->rx_buf[(0x1000 * (tid/3)) + sizeof(struct ethhdr) + sizeof(struct iphdr) + sizeof(struct esphdr) + ((tid%3) * 16) + i] = aes_tmp[threadIdx.x/3][((tid%3)*16) + i]; 
			}
			__syncthreads();
#if 1
			//-------------------------- Single threads Job --------------------------------------------
			if(tid % 3 == 0){
				//////////// Proto_type = ESP set! ///////////
				p_buf->rx_buf[0x1000 * (tid/3) + sizeof(struct ethhdr) + 9] = IPPROTO_ESP; // IPPROTO_ESP = 50
				struct ethhdr* ethh;
				struct iphdr* iph;
				struct esphdr* esph;

				ethh = (struct ethhdr *)&p_buf->rx_buf[0x1000 * (tid/3)];
				iph = (struct iphdr *)(ethh + 1);
				esph = (struct esphdr *)((uint32_t *)iph + iph->ihl);

				// SPI (Security Parameter Index)
				uint32_t spi = 1085899777;
				HTONS32(spi);

				////////// Set ESP header SPI value ///////////////////
				memcpy(&esph->spi, &spi, 4);
				atomicAdd(seq, 1);

				//////////// Set ESP header SEQ value //////////
				memcpy(&esph->seq, seq, 4);

				// CKJUNG, HMAC-SHA1 From here! /////////////////////////////
				// RFC 2104, H(K XOR opad, H(K XOR ipad, text))
				/**** Inner Digest ****/
				// H(K XOR ipad, text) : 64 Bytes
				uint32_t extended[80];
				sha1_gpu_context ictx;

				sha1_kernel_global(&p_buf->rx_buf[(0x1000 * (tid/3)) + sizeof(struct ethhdr) + sizeof(struct iphdr)], &ictx, extended, 64);
				/**** Outer Digest ****/
				// H(K XOR opad, H(K XOR ipad, text)) : 20 Bytes
				sha1_kernel_global(&(ictx.c_state[0]), &octx[threadIdx.x/3], extended, 20);
			}
#endif
			__syncthreads();
			//-------------------------- Multi threads Job --------------------------------------------
			// Attach 20-bytes HMAC-SHA authentication digest to packet.
			memcpy(&p_buf->rx_buf[0x1000 * (tid/3) + 90 + ((tid%3) * 8)], &(octx[threadIdx.x/3].c_state[((tid%3)*8)]), 8);
			__syncthreads();
			//-------------------------- Single threads Job --------------------------------------------
			if(tid % 3 == 0){
				atomicAdd(&pkt_cnt[1], 1);	
				p_buf->rx_buf_idx[tid/3] = chain_seq+1;
			}
		}
	}
}

//CKJUNG, ipsec_1514 ver.
__global__ void nf_ipsec_1514(struct pkt_buf *p_buf, int* pkt_cnt, unsigned int* ctr, unsigned char* d_nounce, unsigned int* d_key, unsigned char* d_sbox, unsigned char* d_GF2, int chain_seq, unsigned int* seq, unsigned int* extended)
{
	// <<< 13, 940 >>> threads. 
	//	94 threads for 1 pkt. (1510B pkt)
	// 940 / 94 = 10, 1TB has 940 threads each and manages 10 pkts.
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	// 13 x 940 = 12,220
	// tid : 0 - 12,219 (12,220 threads)

	__shared__ unsigned char IV[10][16];
	__shared__ unsigned char aes_tmp[10][16*94]; 
	__shared__ unsigned char rot_index; // This index is updated by "the last thread" of each TB to move forward to the NEXT 128 desc.
	// rot_index (0 - 3): {0 x 128(0) ~ 3 x 128(384)} + 127  == 0 ~ 511
	// IV : 10 * 16 =  160
	// aes_tmp : 10 * 16 * 94 = 15,040
	// ictx : 24 * 10 = 240
	// octx : 24 * 10 = 240
	// rot_index : 1
	//-------------------------- Total __shared__ mem Usage : 15,681 / 49,152 (48KB per TB)

	if(threadIdx.x == 0) // The first thread of EACH TB initialize rot_index(rotation_index) to "0". 
		rot_index = 0;

	if(tid == 12219){
		START_RED
		printf("tid %d is alive!\n", tid);
		END
	}

	__syncthreads();

	while(true){ // Persistent Kernel (for every threads)
		// 94-threads to be grouped. 
		if(tid < 128*94){ // Beyond this idx could exceed "rx_buf" boundary.
			//-------------------------- Multi threads Job --------------------------------------------
			if(readNoCache(&p_buf->rx_buf_idx[tid/94 + rot_index*128]) == chain_seq){

				//-------------------------- Single threads Job --------------------------------------------
				if(tid % 94 == 0){ 
					///////////////////// ESP Tailer, padlen, next-hdr /////////////////////////
					int i;
					for(i = 1; i <= 6; i++)
						p_buf->rx_buf[0x1000 * (tid/94 + rot_index*128) + i] = 0; // padding
					
					p_buf->rx_buf[0x1000 * (tid/94 + rot_index*128) + 1510 + 6] = 6; // padlen 

					p_buf->rx_buf[0x1000 * (tid/94 + rot_index*128) + 1510 + 6 + 1] = IPPROTO_IPIP; // next-hdr (Meaning "IP within IP)

					/* For Reference...
						 IPPROTO_IP = 0
						 IPPROTO_ICMP = 1
						 IPPROTO_IPIP = 4
						 IPPROTO_TCP = 6
						 IPPROTO_UDP = 17
						 IPPROTO_ESP = 50
					 */
					atomicAdd(ctr, 1); // same "ctr" value for grouped 4-threads. (counter) AES-CTR Mode
					IV[threadIdx.x/94][15] = *ctr & 0xFF;
					IV[threadIdx.x/94][14] = (*ctr >> 8) & 0xFF; // CKJUNG, 1 Byte = 8bits means, Octal notation
					IV[threadIdx.x/94][13] = (*ctr >> 16) & 0xFF;
					IV[threadIdx.x/94][12] = (*ctr >> 24) & 0xFF;
					for(int i = 0; i < 12; i++)
						IV[threadIdx.x/94][i] = 0;

					// Copy our state into private memory
					unsigned char temp, temp2;
					unsigned char overflow = 0;
					char tmp[16];
					for(int i = 15; i != -1; i--) {
						temp = d_nounce[i];
						temp2 = IV[threadIdx.x/94][i];
						IV[threadIdx.x/94][i] += temp + overflow;
						overflow = ((int)temp2 + (int)temp + (int)overflow > 255);
					}

					AddRoundKey(IV[threadIdx.x/94], &d_key[0]);

					for(int i = 1; i < 10; i++)
					{
						SubBytes(IV[threadIdx.x/94], d_sbox);
						ShiftRows(IV[threadIdx.x/94]);
						MixColumns(IV[threadIdx.x/94], d_GF2, tmp);
						AddRoundKey(IV[threadIdx.x/94], &d_key[4 * i]);
					}
					SubBytes(IV[threadIdx.x/94], d_sbox);
					ShiftRows(IV[threadIdx.x/94]);
					AddRoundKey(IV[threadIdx.x/94], &d_key[4 * 10]);
				}
				__syncthreads();
				//-------------------------- Multi threads Job --------------------------------------------
				////////////////// Locating AES Encrypted parts into a pkt  ///////////////////////////////
				for(int i = 0; i < 16; i++){
					aes_tmp[threadIdx.x/94][((tid%94)*16) + i] = p_buf->rx_buf[(0x1000 * (tid/94 + rot_index*128)) + sizeof(struct ethhdr) + ((tid%94) * 16) + i] ^ IV[threadIdx.x/94][i];
				}
				for(int i = 0; i < 16; i++){
					p_buf->rx_buf[(0x1000 * (tid/94 + rot_index*128)) + sizeof(struct ethhdr) + sizeof(struct iphdr) + sizeof(struct esphdr) + ((tid%94) * 16) + i] = aes_tmp[threadIdx.x/94][((tid%94)*16) + i]; 
				}
				__syncthreads();
#if 1
				//-------------------------- Single threads Job --------------------------------------------
				if(tid % 94 == 0){
					//////////// Proto_type = ESP set! ///////////
					p_buf->rx_buf[0x1000 * (tid/94 + rot_index*128) + sizeof(struct ethhdr) + 9] = IPPROTO_ESP; // IPPROTO_ESP = 50
					struct ethhdr* ethh;
					struct iphdr* iph;
					struct esphdr* esph;

					ethh = (struct ethhdr *)&p_buf->rx_buf[0x1000 * (tid/94 + rot_index*128)];
					iph = (struct iphdr *)(ethh + 1);
					esph = (struct esphdr *)((uint32_t *)iph + iph->ihl);

					// SPI (Security Parameter Index)
					uint32_t spi = 1085899777;
					HTONS32(spi);

					////////// Set ESP header SPI value ///////////////////
					memcpy(&esph->spi, &spi, 4);
					atomicAdd(seq, 1);

					//////////// Set ESP header SEQ value //////////
					memcpy(&esph->seq, seq, 4);
				}
				__syncthreads();
#endif
					// CKJUNG, HMAC-SHA1 From here! /////////////////////////////
					// RFC 2104, H(K XOR opad, H(K XOR ipad, text))
					/**** Inner Digest ****/
					// H(K XOR ipad, text) : 64 Bytes
					sha1_kernel_global_1514(&p_buf->rx_buf[(0x1000 * (tid/94 + rot_index*128)) + sizeof(struct ethhdr) + sizeof(struct iphdr)], &ictx[threadIdx.x/94], extended, 64, (tid/94 + rot_index*128));
					/**** Outer Digest ****/
					// H(K XOR opad, H(K XOR ipad, text)) : 20 Bytes
					sha1_kernel_global_1514(&(ictx[threadIdx.x/94].c_state[0]), &octx[threadIdx.x/94], extended, 20, (tid/94 + rot_index*128));
			
				//-------------------------- Single threads Job --------------------------------------------
				if(tid % 94 == 0){
					atomicAdd(&pkt_cnt[1], 1);	
					p_buf->rx_buf_idx[tid/94 + rot_index*128] = chain_seq+1;
					if(tid/94 == 127)
						rot_index += 1;
					if(rot_index == 4)
						rot_index = 0;
				}
			}
		}
		__syncthreads();
	}
}

unsigned int SubWord(unsigned int w) {                                              
	unsigned int i = (sbox[(w >> 24) & 0xFF] << 24) | (sbox[(w >> 16) & 0xFF] << 16); 
	i |= (sbox[(w >> 8) & 0xFF] << 8) | sbox[w & 0xFF];                               
	return i;                                                                         
}                                                                                   

unsigned int RotWord(unsigned int w) {                                              
	unsigned char temp = (w >> 24) & 0xFF;                                            
	return ((w << 8) | temp);                                                         
}                                                                                   

void KeyExpansion(unsigned char* key, unsigned int* w) {
	unsigned int temp;
	int i = 0;
	
	for(i = 0; i < KEY_SIZE; i++) {
		w[i] = (key[4*i] << 24) | (key[4*i + 1] << 16) | (key[4*i + 2] << 8) | key[4*i + 3];
	}
	
	for(; i < BLOCK_SIZE * (NUM_ROUNDS + 1); i++) {
		temp = w[i - 1];
		if(i % KEY_SIZE == 0) {
			temp = SubWord(RotWord(temp)) ^ Rcon[i / KEY_SIZE];
		}
		w[i] = w[i - KEY_SIZE] ^ temp;
	}
}                                                                                                            

extern "C"
void initialize_ipsec(int chain_seq)
{
	// CKJUNG, 18.10.25 [NF #2: IPSec] Setting initial_counter, key /////////////////////////

	unsigned char nounce[16];
	FILE* fnounce = fopen("test.ctr", "rb");
	fread(&nounce, 1, 16, fnounce);
	fclose(fnounce);

	int num_keys = BLOCK_SIZE * (NUM_ROUNDS + 1);
	unsigned char key[16];
	unsigned int* expanded_key = (unsigned int*)malloc(num_keys * sizeof(int));
	FILE* fkey = fopen("test.key", "rb");
	fread(&key, 1, 16, fkey);
	fclose(fkey);
	KeyExpansion(key, expanded_key);

	unsigned char *d_nounce;
	unsigned int *d_key;
	unsigned char *d_sbox;
	unsigned char *d_GF2;
	unsigned int *d_seq; // 20.02.02. CKJUNG

	unsigned int *d_extended;
	
	printf("____[Initialize]__NF #2__IPSec__\n");
	
	ASSERTRT(hipMalloc((void**)&d_nounce, 16*sizeof(unsigned char)));
	ASSERTRT(hipMemset(d_nounce, 0, 16*sizeof(unsigned char)));
	ASSERTRT(hipMalloc((void**)&d_key, num_keys*sizeof(unsigned int)));
	ASSERTRT(hipMemset(d_key, 0, num_keys*sizeof(unsigned int)));
	ASSERTRT(hipMalloc((void**)&d_sbox, 256*sizeof(unsigned char)));
	ASSERTRT(hipMemset(d_sbox, 0, 256*sizeof(unsigned char)));
	ASSERTRT(hipMalloc((void**)&d_GF2, 256*sizeof(unsigned char)));
	ASSERTRT(hipMemset(d_GF2, 0, 256*sizeof(unsigned char)));
	
	ASSERTRT(hipMalloc((void**)&d_seq, sizeof(unsigned int)));
	ASSERTRT(hipMemset(d_GF2, 0, sizeof(unsigned int)));
	
	ASSERTRT(hipMalloc((void**)&d_extended, 512*80*24*sizeof(unsigned int)));
	ASSERTRT(hipMemset(d_GF2, 0, sizeof(unsigned int)));
	
	hipError_t nounce_err = hipMemcpy(d_nounce, nounce, 16*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipError_t key_err = hipMemcpy(d_key, expanded_key, num_keys*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipError_t sbox_err = hipMemcpy(d_sbox, sbox, 256*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipError_t GF2_err = hipMemcpy(d_GF2, GF_2, 256*sizeof(unsigned char), hipMemcpyHostToDevice);
	if(nounce_err != hipSuccess || key_err != hipSuccess || sbox_err != hipSuccess || GF2_err != hipSuccess)
	{
		START_RED
			printf("[Error] hipMemcpy for \"nounce\" or \"key\" or \"sbox\" or \"GF2\" has failed.\n");
		END
	}else{
		START_GRN
			printf("[IPSec] Nounce, Expanded keys, SBOX, and GF2 are ready.\n");
		END
	}

	hipStream_t cuda_stream3;
	ASSERT_CUDA(hipStreamCreateWithFlags(&cuda_stream3,hipStreamNonBlocking));
	
	printf("NF#2: IPsec\n");

	/* 
	 * ipsec for 64B pkt
	 * 1 pkt needs 3 GPU threads.
	 * 512 x 3 = 1,536 threads. (OK)
	 * 384 threads per TB; 384 = 3 * 128; each TB manages 128 pkts; 128 * 4 = 512 Descs 
	 */
	nf_ipsec_64<<< 4, 384, 0, cuda_stream3 >>> (p_buf, pkt_cnt, ctr, d_nounce, d_key, d_sbox, d_GF2, chain_seq, d_seq); 
	ipsec<<< 4, 384, 0, cuda_stream3 >>> (p_buf, pkt_cnt, ctr, d_nounce, d_key, d_sbox, d_GF2, chain_seq, d_seq); 

	/*
	 * ipsec for 1514B pkt
	 * 1 pkt needs 94 GPU threads.
	 * 512 x 94 = 48,128 threads. (Impossible)
	 * (Persistent kernel arch max; 1,024 threads per 1 SM) 14 x 1,024 = 14,336 threads is MAXIMUM 
	 * We can only afford 128 desc. 128 x 94 = 12,032 threads (OK)
	 * So, 12.8 x 940 is our choice here. ==> 13 x 940 (13 SM x 940 threads/SM)
	 */
	nf_ipsec_1514<<< 13, 940, 0, cuda_stream3 >>> (p_buf, pkt_cnt, ctr, d_nounce, d_key, d_sbox, d_GF2, chain_seq, d_seq, d_extended); 

	START_GRN
	printf("[Done]____[Initialize]__NF #2__IPSec__\n");
	END	

	free(expanded_key);
	// ~ CKJUNG /////////////////////////////////////////////////////////////////////////////
}

