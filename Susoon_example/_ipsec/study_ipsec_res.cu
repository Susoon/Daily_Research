#include "hip/hip_runtime.h"
#include "ipsec.h"
#include "packet_man.h"
#include "gdnio.h"

extern struct pkt_buf *p_buf;
extern int *pkt_cnt;
extern unsigned int *ctr;

__device__ void AddRoundKey(unsigned char *state, unsigned *w)
{
	int i;                                                              
	for(i = 0; i < BLOCK_SIZE; i++) { // column
		state[i * 4 + 0] = state[i * 4 + 0] ^ ((w[i] >> (8 * 3)) & 0xFF);
		state[i * 4 + 1] = state[i * 4 + 1] ^ ((w[i] >> (8 * 2)) & 0xFF);
		state[i * 4 + 2] = state[i * 4 + 2] ^ ((w[i] >> (8 * 1)) & 0xFF);
		state[i * 4 + 3] = state[i * 4 + 3] ^ ((w[i] >> (8 * 0)) & 0xFF);
	}                                                                   
}

__device__ void SubBytes(unsigned char *state, unsigned char* sbox) //state = 16 chars
{ 
	int i;
	for(i = 0; i < 4 * BLOCK_SIZE; i++) {
		state[i] = sbox[state[i]];
	}
} 

__device__ void ShiftRows(unsigned char *state)
{ 
	// NOTE: For whatever reason the standard uses column-major ordering ?
	// 0 1 2 3 --> 0 1 2 3  | 0  4  8  12 --> 0   4  8 12
	// 0 1 2 3 --> 1 2 3 0  | 1  5  9  13 --> 5   9 13  1
	// 0 1 2 3 --> 2 3 0 1  | 2  6  10 14 --> 10 14  2  6
	// 0 1 2 3 --> 3 0 1 2  | 3  7  11 15 --> 15  3  7 11
	unsigned char temp = state[1];

	state[1] = state[5];
	state[5] = state[9];
	state[9] = state[13];
	state[13] = temp;

	temp = state[2];
	state[2] = state[10];
	state[10] = temp;
	temp = state[6];
	state[6] = state[14];
	state[14] = temp;

	temp = state[3];
	state[3] = state[15];
	state[15] = state[11];
	state[11] = state[7];
	state[7] = temp;
}

// See "Efficient Software Implementation of AES on 32-bit platforms"
__device__ void MixColumns(unsigned char *state, unsigned char* GF_2, char* s) 
{
//[TODO] malloc!!!!!! is the criminal!!! CKJUNG, 18.10.26 
	memcpy(s, state, 4 * BLOCK_SIZE);
	int i;
#if 1
	for(i = 0; i < BLOCK_SIZE; i++) { // column
		unsigned char * x = (unsigned char*)&s[i*4];
		unsigned char * y = (unsigned char*)&state[i*4];
		y[0] = x[1] ^ x[2] ^ x[3];
		y[1] = x[0] ^ x[2] ^ x[3];
		y[2] = x[0] ^ x[1] ^ x[3];
		y[3] = x[0] ^ x[1] ^ x[2];
		x[0] = GF_2[x[0]];
		x[1] = GF_2[x[1]];
		x[2] = GF_2[x[2]];
		x[3] = GF_2[x[3]];
		y[0] ^= x[0] ^ x[1];
		y[1] ^= x[1] ^ x[2];
		y[2] ^= x[2] ^ x[3];
		y[3] ^= x[3] ^ x[0];
	}
#endif
} 

/**                                           
 * Initialize new context                      
 *                                             
 * @param context SHA1-Context                 
 */                                            
/*
 * Process extended block.
 */
__device__ void sha1_gpu_process (sha1_gpu_context *ctx, uint32_t W[80])
{
	uint32_t A, B, C, D, E;
	A = ctx->state[0];
	B = ctx->state[1];
	C = ctx->state[2];
	D = ctx->state[3];
	E = ctx->state[4];

#define P(a,b,c,d,e,x)\
{\
	e += S(a,5) + F(b,c,d) + K + x; b = S(b,30);\
}


#define F(x,y,z) (z ^ (x & (y ^ z)))
#define K 0x5A827999

	P( A, B, C, D, E, W[0]  );
	P( E, A, B, C, D, W[1]  );
	P( D, E, A, B, C, W[2]  );
	P( C, D, E, A, B, W[3]  );
	P( B, C, D, E, A, W[4]  );
	P( A, B, C, D, E, W[5]  );
	P( E, A, B, C, D, W[6]  );
	P( D, E, A, B, C, W[7]  );
	P( C, D, E, A, B, W[8]  );
	P( B, C, D, E, A, W[9]  );
	P( A, B, C, D, E, W[10] );
	P( E, A, B, C, D, W[11] );
	P( D, E, A, B, C, W[12] );
	P( C, D, E, A, B, W[13] );
	P( B, C, D, E, A, W[14] );
	P( A, B, C, D, E, W[15] );
	P( E, A, B, C, D, W[16] );
	P( D, E, A, B, C, W[17] );
	P( C, D, E, A, B, W[18] );
	P( B, C, D, E, A, W[19] );

#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0x6ED9EBA1

	P( A, B, C, D, E, W[20] );
	P( E, A, B, C, D, W[21] );
	P( D, E, A, B, C, W[22] );
	P( C, D, E, A, B, W[23] );
	P( B, C, D, E, A, W[24] );
	P( A, B, C, D, E, W[25] ); // w[25] is the problem.
	P( E, A, B, C, D, W[26] );
	P( D, E, A, B, C, W[27] );
	P( C, D, E, A, B, W[28] );
	P( B, C, D, E, A, W[29] );
	P( A, B, C, D, E, W[30] );
	P( E, A, B, C, D, W[31] );
	P( D, E, A, B, C, W[32] );
	P( C, D, E, A, B, W[33] );
	P( B, C, D, E, A, W[34] );
	P( A, B, C, D, E, W[35] );
	P( E, A, B, C, D, W[36] );
	P( D, E, A, B, C, W[37] );
	P( C, D, E, A, B, W[38] );
	P( B, C, D, E, A, W[39] );


#undef K
#undef F

#define F(x,y,z) ((x & y) | (z & (x | y)))
#define K 0x8F1BBCDC

	P( A, B, C, D, E, W[40] );
	P( E, A, B, C, D, W[41] );
	P( D, E, A, B, C, W[42] );
	P( C, D, E, A, B, W[43] );
	P( B, C, D, E, A, W[44] );
	P( A, B, C, D, E, W[45] );
	P( E, A, B, C, D, W[46] );
	P( D, E, A, B, C, W[47] );
	P( C, D, E, A, B, W[48] );
	P( B, C, D, E, A, W[49] );
	P( A, B, C, D, E, W[50] );
	P( E, A, B, C, D, W[51] );
	P( D, E, A, B, C, W[52] );
	P( C, D, E, A, B, W[53] );
	P( B, C, D, E, A, W[54] );
	P( A, B, C, D, E, W[55] );
	P( E, A, B, C, D, W[56] );
	P( D, E, A, B, C, W[57] );
	P( C, D, E, A, B, W[58] );
	P( B, C, D, E, A, W[59] );

#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0xCA62C1D6

	P( A, B, C, D, E, W[60] );
	P( E, A, B, C, D, W[61] );
	P( D, E, A, B, C, W[62] );
	P( C, D, E, A, B, W[63] );
	P( B, C, D, E, A, W[64] );
	P( A, B, C, D, E, W[65] );
	P( E, A, B, C, D, W[66] );
	P( D, E, A, B, C, W[67] );
	P( C, D, E, A, B, W[68] );
	P( B, C, D, E, A, W[69] );
	P( A, B, C, D, E, W[70] );
	P( E, A, B, C, D, W[71] );
	P( D, E, A, B, C, W[72] );
	P( C, D, E, A, B, W[73] );
	P( B, C, D, E, A, W[74] );
	P( A, B, C, D, E, W[75] );
	P( E, A, B, C, D, W[76] );
	P( D, E, A, B, C, W[77] );
	P( C, D, E, A, B, W[78] );
	P( B, C, D, E, A, W[79] );
#undef K
#undef F

	ctx->state[0] += A;
	ctx->state[1] += B;
	ctx->state[2] += C;
	ctx->state[3] += D;
	ctx->state[4] += E;
}

unsigned int SubWord(unsigned int w) {                                              
	unsigned int i = (sbox[(w >> 24) & 0xFF] << 24) | (sbox[(w >> 16) & 0xFF] << 16); 
	i |= (sbox[(w >> 8) & 0xFF] << 8) | sbox[w & 0xFF];                               
	return i;                                                                         
}                                                                                   

unsigned int RotWord(unsigned int w) {                                              
	unsigned char temp = (w >> 24) & 0xFF;                                            
	return ((w << 8) | temp);                                                         
}                                                                                   

void KeyExpansion(unsigned char* key, unsigned int* w) {
	unsigned int temp;
	int i = 0;
	
	for(i = 0; i < KEY_SIZE; i++) {
		w[i] = (key[4*i] << 24) | (key[4*i + 1] << 16) | (key[4*i + 2] << 8) | key[4*i + 3];
	}
	
	for(; i < BLOCK_SIZE * (NUM_ROUNDS + 1); i++) {
		temp = w[i - 1];
		if(i % KEY_SIZE == 0) {
			temp = SubWord(RotWord(temp)) ^ Rcon[i / KEY_SIZE];
		}
		w[i] = w[i - KEY_SIZE] ^ temp;
	}
}                                                                                                            

extern "C"
void initialize_ipsec(int chain_seq)
{
	// CKJUNG, 18.10.25 [NF #2: IPSec] Setting initial_counter, key /////////////////////////

	unsigned char nounce[16];
	FILE* fnounce = fopen("test.ctr", "rb");
	fread(&nounce, 1, 16, fnounce);
	fclose(fnounce);

	int num_keys = BLOCK_SIZE * (NUM_ROUNDS + 1);
	unsigned char key[16];
	unsigned int* expanded_key = (unsigned int*)malloc(num_keys * sizeof(int));
	FILE* fkey = fopen("test.key", "rb");
	fread(&key, 1, 16, fkey);
	fclose(fkey);
	KeyExpansion(key, expanded_key);

	unsigned char *d_nounce;
	unsigned int *d_key;
	unsigned char *d_sbox;
	unsigned char *d_GF2;
	unsigned int *d_seq; // 20.02.02. CKJUNG

	unsigned int *d_extended;
	
	printf("____[Initialize]__NF #2__IPSec__\n");
	
	ASSERTRT(hipMalloc((void**)&d_nounce, 16*sizeof(unsigned char)));
	ASSERTRT(hipMemset(d_nounce, 0, 16*sizeof(unsigned char)));
	ASSERTRT(hipMalloc((void**)&d_key, num_keys*sizeof(unsigned int)));
	ASSERTRT(hipMemset(d_key, 0, num_keys*sizeof(unsigned int)));
	ASSERTRT(hipMalloc((void**)&d_sbox, 256*sizeof(unsigned char)));
	ASSERTRT(hipMemset(d_sbox, 0, 256*sizeof(unsigned char)));
	ASSERTRT(hipMalloc((void**)&d_GF2, 256*sizeof(unsigned char)));
	ASSERTRT(hipMemset(d_GF2, 0, 256*sizeof(unsigned char)));
	
	ASSERTRT(hipMalloc((void**)&d_seq, sizeof(unsigned int)));
	ASSERTRT(hipMemset(d_GF2, 0, sizeof(unsigned int)));
	
	ASSERTRT(hipMalloc((void**)&d_extended, 512*80*24*sizeof(unsigned int)));
	ASSERTRT(hipMemset(d_GF2, 0, sizeof(unsigned int)));
	
	hipError_t nounce_err = hipMemcpy(d_nounce, nounce, 16*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipError_t key_err = hipMemcpy(d_key, expanded_key, num_keys*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipError_t sbox_err = hipMemcpy(d_sbox, sbox, 256*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipError_t GF2_err = hipMemcpy(d_GF2, GF_2, 256*sizeof(unsigned char), hipMemcpyHostToDevice);
	if(nounce_err != hipSuccess || key_err != hipSuccess || sbox_err != hipSuccess || GF2_err != hipSuccess)
	{
		START_RED
			printf("[Error] hipMemcpy for \"nounce\" or \"key\" or \"sbox\" or \"GF2\" has failed.\n");
		END
	}else{
		START_GRN
			printf("[IPSec] Nounce, Expanded keys, SBOX, and GF2 are ready.\n");
		END
	}

	hipStream_t cuda_stream3;
	ASSERT_CUDA(hipStreamCreateWithFlags(&cuda_stream3,hipStreamNonBlocking));
	
	printf("NF#2: IPsec\n");

	/* 
	 * ipsec for 64B pkt
	 * 1 pkt needs 3 GPU threads.
	 * 512 x 3 = 1,536 threads. (OK)
	 * 384 threads per TB; 384 = 3 * 128; each TB manages 128 pkts; 128 * 4 = 512 Descs 
	 */
	nf_ipsec_64<<< 4, 384, 0, cuda_stream3 >>> (p_buf, pkt_cnt, ctr, d_nounce, d_key, d_sbox, d_GF2, chain_seq, d_seq); 
	ipsec<<< 4, 384, 0, cuda_stream3 >>> (p_buf, pkt_cnt, ctr, d_nounce, d_key, d_sbox, d_GF2, chain_seq, d_seq); 

	/*
	 * ipsec for 1514B pkt
	 * 1 pkt needs 94 GPU threads.
	 * 512 x 94 = 48,128 threads. (Impossible)
	 * (Persistent kernel arch max; 1,024 threads per 1 SM) 14 x 1,024 = 14,336 threads is MAXIMUM 
	 * We can only afford 128 desc. 128 x 94 = 12,032 threads (OK)
	 * So, 12.8 x 940 is our choice here. ==> 13 x 940 (13 SM x 940 threads/SM)
	 */
	nf_ipsec_1514<<< 13, 940, 0, cuda_stream3 >>> (p_buf, pkt_cnt, ctr, d_nounce, d_key, d_sbox, d_GF2, chain_seq, d_seq, d_extended); 

	START_GRN
	printf("[Done]____[Initialize]__NF #2__IPSec__\n");
	END	

	free(expanded_key);
	// ~ CKJUNG /////////////////////////////////////////////////////////////////////////////
}

