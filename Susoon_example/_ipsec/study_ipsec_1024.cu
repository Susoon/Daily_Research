#include "hip/hip_runtime.h"
#include "ipsec.h"
#include "packet_man.h"
#include "gdnio.h"

#define TOTAL_T_NUM 11 * 63 * 16
#define AES_T_NUM 63
#define PPB 16
#define HMAC_T_NUM 16
#define PKT_SIZE 1024
#define PAD_LEN 0

__device__ void sha1_kernel_global_1024(unsigned char *data, sha1_gpu_context *ctx, unsigned int *extended, int len, int pkt_idx)
{
	int thread_index = threadIdx.x%94;
	
	if(thread_index >= HMAC_T_NUM)
		return;

	int e_index = thread_index * 80;
	int block_index = thread_index * 64;
	unsigned int temp, t;

	if(thread_index == 0){
		/* Initialization vector for SHA-1 */
		ctx->state[0] = 0x67452301;           
		ctx->state[1] = 0xEFCDAB89;           
		ctx->state[2] = 0x98BADCFE;           
		ctx->state[3] = 0x10325476;           
		ctx->state[4] = 0xC3D2E1F0; 
	}
	__syncthreads();

	/*
	 * Extend 32 block byte block into 80 byte block.
	 */

//sh_kim 20.03.11 : when data length is 20byte, we need padding
	if(len == 20 && threadIdx.x = 0)
	{
		memset(data + len - 1, 0, 44);
	}

	GET_UINT32_BE( extended[pkt_idx*e_index + 0], data + block_index,  0 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 1], data + block_index,  4 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 2], data + block_index,  8 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 3], data + block_index, 12 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 4], data + block_index, 16 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 5], data + block_index, 20 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 6], data + block_index, 24 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 7], data + block_index, 28 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 8], data + block_index, 32 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 9], data + block_index, 36 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 10], data + block_index, 40 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 11], data + block_index, 44 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 12], data + block_index, 48 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 13], data + block_index, 52 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 14], data + block_index, 56 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 15], data + block_index, 60 );

	// Same as "blk(i)" macro in openssl source.
	for (t = 16; t < 80; t++) {
		temp = extended[pkt_idx*e_index + t - 3] ^ extended[pkt_idx*e_index + t - 8] ^ extended[pkt_idx*e_index + t - 14] ^ extended[pkt_idx*e_index + t - 16];
		extended[pkt_idx*e_index + t] = S(temp,1);
	}

	__syncthreads();
	if(thread_index == 0){
		for(t = 0; t < HMAC_T_NUM; t++) 
			sha1_gpu_process(ctx, (unsigned int*)&extended[pkt_idx * t * 80]);
	}
}

//CKJUNG, ipsec_1024 ver.
__global__ void nf_ipsec_1024(struct pkt_buf *p_buf, int* pkt_cnt, unsigned int* ctr, unsigned char* d_nounce, unsigned int* d_key, unsigned char* d_sbox, unsigned char* d_GF2, int chain_seq, unsigned int* seq, unsigned int* extended)
{
	// <<< 11, 1008 >>> threads. 
	//	63 threads for 1 pkt. (102B pkt)
	// 1008 / 63 = 16, 1TB has 1008 threads each and manages 16 pkts.
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int cur_tid = threadIdx.x / AES_T_NUM;
	// 11 x 1008 = 11,088
	// tid : 0 - 11,087 (10,087 threads)

	__shared__ unsigned char IV[PPB][16];
	__shared__ unsigned char aes_tmp[PPB][16*AES_T_NUM]; 
	__shared__ unsigned char rot_index; // This index is updated by "the last thread" of each TB to move forward to the NEXT 256 desc.
	// rot_index (0 - 2): {0 x 171(0) ~ 2 x 171(342)} + 170  == 0 ~ 511
	// IV : 16 * 16 =  256
	// aes_tmp : 16 * 16 * 63 = 16,128
	// ictx : 16 * 16 = 256
	// octx : 16 * 16 = 256
	// rot_index : 1
	//-------------------------- Total __shared__ mem Usage : 16,897 / 49,152 (48KB per TB)

	if(threadIdx.x == 0) // The first thread of EACH TB initialize rot_index(rotation_index) to "0". 
		rot_index = 0;

	if(tid == TOTAL_T_NUM - 1){
		START_RED
		printf("tid %d is alive!\n", tid);
		END
	}

	__syncthreads();

	while(true){ // Persistent Kernel (for every threads)
		// 63-threads to be grouped. 
		if(tid < 171*AES_T_NUM && (rot_index == 2 && tid < 171 * AES_T_NUM - 1)){ // Beyond this idx could exceed "rx_buf" boundary.
			//-------------------------- Multi threads Job --------------------------------------------
			if(readNoCache(&p_buf->rx_buf_idx[tid/AES_T_NUM + rot_index*171]) == chain_seq){

				//-------------------------- Single threads Job --------------------------------------------
				if(tid % AES_T_NUM == 0){ 
					///////////////////// ESP Tailer, padlen, next-hdr /////////////////////////
					int i;
					for(i = 1; i <= 6; i++)
						p_buf->rx_buf[0x1000 * (tid/AES_T_NUM + rot_index*171) + i] = 0; // padding
					
					p_buf->rx_buf[0x1000 * (tid/AES_T_NUM + rot_index*171) + 1510 + 6] = 6; // padlen 

					p_buf->rx_buf[0x1000 * (tid/AES_T_NUM + rot_index*171) + 1510 + 6 + 1] = IPPROTO_IPIP; // next-hdr (Meaning "IP within IP)

					/* For Reference...
						 IPPROTO_IP = 0
						 IPPROTO_ICMP = 1
						 IPPROTO_IPIP = 4
						 IPPROTO_TCP = 6
						 IPPROTO_UDP = 17
						 IPPROTO_ESP = 50
					 */
					atomicAdd(ctr, 1); // same "ctr" value for grouped 63-threads. (counter) AES-CTR Mode
					IV[cur_tid][15] = *ctr & 0xFF;
					IV[cur_tid][14] = (*ctr >> 8) & 0xFF; // CKJUNG, 1 Byte = 8bits means, Octal notation
					IV[cur_tid][13] = (*ctr >> 16) & 0xFF;
					IV[cur_tid][12] = (*ctr >> 24) & 0xFF;
					for(int i = 0; i < 12; i++)
						IV[cur_tid][i] = 0;

					// Copy our state into private memory
					unsigned char temp, temp2;
					unsigned char overflow = 0;
					char tmp[16];
					for(int i = 15; i != -1; i--) {
						temp = d_nounce[i];
						temp2 = IV[cur_tid][i];
						IV[cur_tid][i] += temp + overflow;
						overflow = ((int)temp2 + (int)temp + (int)overflow > 255);
					}

					AddRoundKey(IV[cur_tid], &d_key[0]);

					for(int i = 1; i < 10; i++)
					{
						SubBytes(IV[cur_tid], d_sbox);
						ShiftRows(IV[cur_tid]);
						MixColumns(IV[cur_tid], d_GF2, tmp);
						AddRoundKey(IV[cur_tid], &d_key[4 * i]);
					}
					SubBytes(IV[cur_tid], d_sbox);
					ShiftRows(IV[cur_tid]);
					AddRoundKey(IV[cur_tid], &d_key[4 * 10]);
				}
				__syncthreads();
				//-------------------------- Multi threads Job --------------------------------------------
				////////////////// Locating AES Encrypted parts into a pkt  ///////////////////////////////
				for(int i = 0; i < 16; i++){
					aes_tmp[cur_tid][((tid%AES_T_NUM)*16) + i] = p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM + rot_index*171)) + sizeof(struct ethhdr) + ((tid%AES_T_NUM) * 16) + i] ^ IV[cur_tid][i];
				}
				for(int i = 0; i < 16; i++){
					p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM + rot_index*171)) + sizeof(struct ethhdr) + sizeof(struct iphdr) + sizeof(struct esphdr) + ((tid%AES_T_NUM) * 16) + i] = aes_tmp[cur_tid][((tid%AES_T_NUM)*16) + i]; 
				}
				__syncthreads();
#if 1
				//-------------------------- Single threads Job --------------------------------------------
				if(tid % AES_T_NUM == 0){
					//////////// Proto_type = ESP set! ///////////
					p_buf->rx_buf[0x1000 * (tid/AES_T_NUM + rot_index*171) + sizeof(struct ethhdr) + 9] = IPPROTO_ESP; // IPPROTO_ESP = 50
					struct ethhdr* ethh;
					struct iphdr* iph;
					struct esphdr* esph;

					ethh = (struct ethhdr *)&p_buf->rx_buf[0x1000 * (tid/AES_T_NUM + rot_index*171)];
					iph = (struct iphdr *)(ethh + 1);
					esph = (struct esphdr *)((uint32_t *)iph + iph->ihl);

					// SPI (Security Parameter Index)
					uint32_t spi = 1085899777;
					HTONS32(spi);

					////////// Set ESP header SPI value ///////////////////
					memcpy(&esph->spi, &spi, 4);
					atomicAdd(seq, 1);

					//////////// Set ESP header SEQ value //////////
					memcpy(&esph->seq, seq, 4);
				}
				__syncthreads();
#endif
					// CKJUNG, HMAC-SHA1 From here! /////////////////////////////
					// RFC 2104, H(K XOR opad, H(K XOR ipad, text))
					/**** Inner Digest ****/
					// H(K XOR ipad, text) : 64 Bytes
					sha1_kernel_global_1024(&p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM + rot_index*171)) + sizeof(struct ethhdr) + sizeof(struct iphdr)], &ictx[cur_tid], extended, 64, (tid/AES_T_NUM + rot_index*171));
					/**** Outer Digest ****/
					// H(K XOR opad, H(K XOR ipad, text)) : 20 Bytes
					sha1_kernel_global_1024(&(ictx[cur_tid].c_state[0]), &octx[cur_tid], extended, 20, (tid/AES_T_NUM + rot_index*171));
			
				//-------------------------- Single threads Job --------------------------------------------
				if(tid % AES_T_NUM == 0){
					atomicAdd(&pkt_cnt[1], 1);	
					p_buf->rx_buf_idx[tid/AES_T_NUM + rot_index*171] = chain_seq+1;
					if(tid/AES_T_NUM == 255)
						rot_index += 1;
					if(rot_index == 3)
						rot_index = 0;
				}
			}
		}
		__syncthreads();
	}
}
