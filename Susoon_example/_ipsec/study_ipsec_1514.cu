#include "hip/hip_runtime.h"
#include "ipsec.h"
#include "packet_man.h"
#include "gdnio.h"

#define TOTAL_T_NUM 13 * 94 * 10
#define AES_T_NUM 94
#define PPB 10
#define HMAC_T_NUM 24
#define PKT_SIZE 1514
#define PAD_LEN 6

__device__ void sha1_kernel_global_1514(unsigned char *data, sha1_gpu_context *ctx, unsigned int *extended, int len, int pkt_idx)
{
	int thread_index = threadIdx.x%94;
	
	if(thread_index >= HMAC_T_NUM)
		return;

	int e_index = thread_index * 80;
	int block_index = thread_index * 64;
	unsigned int temp, t;

	if(thread_index == 0){
		/* Initialization vector for SHA-1 */
		ctx->state[0] = 0x67452301;           
		ctx->state[1] = 0xEFCDAB89;           
		ctx->state[2] = 0x98BADCFE;           
		ctx->state[3] = 0x10325476;           
		ctx->state[4] = 0xC3D2E1F0; 
	}
	__syncthreads();

	/*
	 * Extend 32 block byte block into 80 byte block.
	 */

//sh_kim 20.03.11 : when data length is 20byte, we need padding
	if(len == 20 && threadIdx.x = 0)
	{
		memset(data + len - 1, 0, 44);
	}

	GET_UINT32_BE( extended[pkt_idx*e_index + 0], data + block_index,  0 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 1], data + block_index,  4 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 2], data + block_index,  8 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 3], data + block_index, 12 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 4], data + block_index, 16 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 5], data + block_index, 20 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 6], data + block_index, 24 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 7], data + block_index, 28 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 8], data + block_index, 32 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 9], data + block_index, 36 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 10], data + block_index, 40 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 11], data + block_index, 44 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 12], data + block_index, 48 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 13], data + block_index, 52 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 14], data + block_index, 56 );
	GET_UINT32_BE( extended[pkt_idx*e_index + 15], data + block_index, 60 );

	// Same as "blk(i)" macro in openssl source.
	for (t = 16; t < 80; t++) {
		temp = extended[pkt_idx*e_index + t - 3] ^ extended[pkt_idx*e_index + t - 8] ^ extended[pkt_idx*e_index + t - 14] ^ extended[pkt_idx*e_index + t - 16];
		extended[pkt_idx*e_index + t] = S(temp,1);
	}

	__syncthreads();
	if(thread_index == 0){
		for(t = 0; t < 24; t++) 
			sha1_gpu_process(ctx, (unsigned int*)&extended[pkt_idx * t * 80]);
	}
}

//CKJUNG, ipsec_1514 ver.
__global__ void nf_ipsec_1514(struct pkt_buf *p_buf, int* pkt_cnt, unsigned int* ctr, unsigned char* d_nounce, unsigned int* d_key, unsigned char* d_sbox, unsigned char* d_GF2, int chain_seq, unsigned int* seq, unsigned int* extended)
{
	// <<< 13, 940 >>> threads. 
	//	94 threads for 1 pkt. (1510B pkt)
	// 940 / 94 = 10, 1TB has 940 threads each and manages 10 pkts.
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int cur_tid = threadIdx.x / AES_T_NUM;
	// 13 x 940 = 12,220
	// tid : 0 - 12,219 (12,220 threads)

	__shared__ unsigned char IV[PPB][16];
	__shared__ unsigned char aes_tmp[PPB][16*AES_T_NUM]; 
	__shared__ unsigned char rot_index; // This index is updated by "the last thread" of each TB to move forward to the NEXT 128 desc.
	// rot_index (0 - 3): {0 x 128(0) ~ 3 x 128(384)} + 127  == 0 ~ 511
	// IV : 10 * 16 =  160
	// aes_tmp : 10 * 16 * 94 = 15,040
	// ictx : 24 * 10 = 240
	// octx : 24 * 10 = 240
	// rot_index : 1
	//-------------------------- Total __shared__ mem Usage : 15,681 / 49,152 (48KB per TB)

	if(threadIdx.x == 0) // The first thread of EACH TB initialize rot_index(rotation_index) to "0". 
		rot_index = 0;

	if(tid == TOTAL_T_NUM - 1){
		START_RED
		printf("tid %d is alive!\n", tid);
		END
	}

	__syncthreads();

	while(true){ // Persistent Kernel (for every threads)
		// 94-threads to be grouped. 
		if(tid < 128*AES_T_NUM){ // Beyond this idx could exceed "rx_buf" boundary.
			//-------------------------- Multi threads Job --------------------------------------------
			if(readNoCache(&p_buf->rx_buf_idx[tid/AES_T_NUM + rot_index*128]) == chain_seq){

				//-------------------------- Single threads Job --------------------------------------------
				if(tid % AES_T_NUM == 0){ 
					///////////////////// ESP Tailer, padlen, next-hdr /////////////////////////
					int i;
					for(i = 1; i <= 6; i++)
						p_buf->rx_buf[0x1000 * (tid/AES_T_NUM + rot_index*128) + i] = 0; // padding
					
					p_buf->rx_buf[0x1000 * (tid/AES_T_NUM + rot_index*128) + 1510 + 6] = 6; // padlen 

					p_buf->rx_buf[0x1000 * (tid/AES_T_NUM + rot_index*128) + 1510 + 6 + 1] = IPPROTO_IPIP; // next-hdr (Meaning "IP within IP)

					/* For Reference...
						 IPPROTO_IP = 0
						 IPPROTO_ICMP = 1
						 IPPROTO_IPIP = 4
						 IPPROTO_TCP = 6
						 IPPROTO_UDP = 17
						 IPPROTO_ESP = 50
					 */
					atomicAdd(ctr, 1); // same "ctr" value for grouped 4-threads. (counter) AES-CTR Mode
					IV[cur_tid][15] = *ctr & 0xFF;
					IV[cur_tid][14] = (*ctr >> 8) & 0xFF; // CKJUNG, 1 Byte = 8bits means, Octal notation
					IV[cur_tid][13] = (*ctr >> 16) & 0xFF;
					IV[cur_tid][12] = (*ctr >> 24) & 0xFF;
					for(int i = 0; i < 12; i++)
						IV[cur_tid][i] = 0;

					// Copy our state into private memory
					unsigned char temp, temp2;
					unsigned char overflow = 0;
					char tmp[16];
					for(int i = 15; i != -1; i--) {
						temp = d_nounce[i];
						temp2 = IV[cur_tid][i];
						IV[cur_tid][i] += temp + overflow;
						overflow = ((int)temp2 + (int)temp + (int)overflow > 255);
					}

					AddRoundKey(IV[cur_tid], &d_key[0]);

					for(int i = 1; i < 10; i++)
					{
						SubBytes(IV[cur_tid], d_sbox);
						ShiftRows(IV[cur_tid]);
						MixColumns(IV[cur_tid], d_GF2, tmp);
						AddRoundKey(IV[cur_tid], &d_key[4 * i]);
					}
					SubBytes(IV[cur_tid], d_sbox);
					ShiftRows(IV[cur_tid]);
					AddRoundKey(IV[cur_tid], &d_key[4 * 10]);
				}
				__syncthreads();
				//-------------------------- Multi threads Job --------------------------------------------
				////////////////// Locating AES Encrypted parts into a pkt  ///////////////////////////////
				for(int i = 0; i < 16; i++){
					aes_tmp[cur_tid][((tid%AES_T_NUM)*16) + i] = p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM + rot_index*128)) + sizeof(struct ethhdr) + ((tid%AES_T_NUM) * 16) + i] ^ IV[cur_tid][i];
				}
				for(int i = 0; i < 16; i++){
					p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM + rot_index*128)) + sizeof(struct ethhdr) + sizeof(struct iphdr) + sizeof(struct esphdr) + ((tid%AES_T_NUM) * 16) + i] = aes_tmp[cur_tid][((tid%AES_T_NUM)*16) + i]; 
				}
				__syncthreads();
#if 1
				//-------------------------- Single threads Job --------------------------------------------
				if(tid % AES_T_NUM == 0){
					//////////// Proto_type = ESP set! ///////////
					p_buf->rx_buf[0x1000 * (tid/AES_T_NUM + rot_index*128) + sizeof(struct ethhdr) + 9] = IPPROTO_ESP; // IPPROTO_ESP = 50
					struct ethhdr* ethh;
					struct iphdr* iph;
					struct esphdr* esph;

					ethh = (struct ethhdr *)&p_buf->rx_buf[0x1000 * (tid/AES_T_NUM + rot_index*128)];
					iph = (struct iphdr *)(ethh + 1);
					esph = (struct esphdr *)((uint32_t *)iph + iph->ihl);

					// SPI (Security Parameter Index)
					uint32_t spi = 1085899777;
					HTONS32(spi);

					////////// Set ESP header SPI value ///////////////////
					memcpy(&esph->spi, &spi, 4);
					atomicAdd(seq, 1);

					//////////// Set ESP header SEQ value //////////
					memcpy(&esph->seq, seq, 4);
				}
				__syncthreads();
#endif
					// CKJUNG, HMAC-SHA1 From here! /////////////////////////////
					// RFC 2104, H(K XOR opad, H(K XOR ipad, text))
					/**** Inner Digest ****/
					// H(K XOR ipad, text) : 64 Bytes
					sha1_kernel_global_1514(&p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM + rot_index*128)) + sizeof(struct ethhdr) + sizeof(struct iphdr)], &ictx[cur_tid], extended, 64, (tid/AES_T_NUM + rot_index*128));
					/**** Outer Digest ****/
					// H(K XOR opad, H(K XOR ipad, text)) : 20 Bytes
					sha1_kernel_global_1514(&(ictx[cur_tid].c_state[0]), &octx[cur_tid], extended, 20, (tid/AES_T_NUM + rot_index*128));
			
				//-------------------------- Single threads Job --------------------------------------------
				if(tid % AES_T_NUM == 0){
					atomicAdd(&pkt_cnt[1], 1);	
					p_buf->rx_buf_idx[tid/AES_T_NUM + rot_index*128] = chain_seq+1;
					if(tid/AES_T_NUM == 127)
						rot_index += 1;
					if(rot_index == 4)
						rot_index = 0;
				}
			}
		}
		__syncthreads();
	}
}
