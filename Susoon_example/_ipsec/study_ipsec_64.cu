#include "hip/hip_runtime.h"
#include "ipsec.h"
#include "packet_man.h"
#include "gdnio.h"

#define TOTAL_T_NUM 4 * 3 * 128
#define AES_T_NUM 3
#define PPB 128
#define HMAC_T_NUM 1
#define PKT_SIZE 64
#define PAD_LEN 0

__device__ void sha1_kernel_global_64(unsigned char *data, sha1_gpu_context *ctx, uint32_t *extended, int len)
{
#if 1
	/* Initialization vector for SHA-1 */
	ctx->state[0] = 0x67452301;           
	ctx->state[1] = 0xEFCDAB89;           
	ctx->state[2] = 0x98BADCFE;           
	ctx->state[3] = 0x10325476;           
	ctx->state[4] = 0xC3D2E1F0;           
#endif

	uint32_t temp, t;

	/*
	 * Extend 32 block byte block into 80 byte block.
	 */

//sh_kim 20.03.11 : when data length is 20byte, we need padding
	if(len == 20)
	{
		memset(data + len - 1, 0, 44);
	}

	GET_UINT32_BE( extended[0], data,  0 );
	GET_UINT32_BE( extended[1], data,  4 );
	GET_UINT32_BE( extended[2], data,  8 );
	GET_UINT32_BE( extended[3], data, 12 );
	GET_UINT32_BE( extended[4], data, 16 );
	GET_UINT32_BE( extended[5], data, 20 );
	GET_UINT32_BE( extended[6], data, 24 );
	GET_UINT32_BE( extended[7], data, 28 );
	GET_UINT32_BE( extended[8], data, 32 );
	GET_UINT32_BE( extended[9], data, 36 );
	GET_UINT32_BE( extended[10], data, 40 );
	GET_UINT32_BE( extended[11], data, 44 );
	GET_UINT32_BE( extended[12], data, 48 );
	GET_UINT32_BE( extended[13], data, 52 );
	GET_UINT32_BE( extended[14], data, 56 );
	GET_UINT32_BE( extended[15], data, 60 );

	// Same as "blk(i)" macro in openssl source.
	for (t = 16; t < 80; t++) {
		temp = extended[t - 3] ^ extended[t - 8] ^ extended[t - 14] ^ extended[t - 16];
		extended[t] = S(temp,1);
	}

	sha1_gpu_process(ctx, extended);
}

// CKJUNG, 18.10.26 [NF#2:IPSec]-------------------------------------
__global__ void nf_ipsec_64(struct pkt_buf *p_buf, int* pkt_cnt, unsigned int* ctr, unsigned char* d_nounce, unsigned int* d_key, unsigned char* d_sbox, unsigned char* d_GF2, int chain_seq, unsigned int* seq)
{
	// <<< 4, 384 >>> threads. 
	//	3 threads for 1 pkt. (60B pkt)
	// 384 / 3 = 128, 1TB has 384 threads each and manages 128 pkts.
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int cur_tid = threadIdx.x / AES_T_NUM;
	// tid : 0 - 1,535 (1,536 threads) = 512 * 3

	__shared__ unsigned char IV[PPB][16];
	__shared__ unsigned char aes_tmp[PPB][16*AES_T_NUM]; 
	__shared__ sha1_gpu_context ictx[PPB];
	__shared__ sha1_gpu_context octx[PPB];
	// IV : 128 * 16 =  2,048 
	// aes_tmp : 128 * 16 * 3 = 6,144
	// ictx : 24 * 128 = 3,072
	// octx : 24 * 128 = 3,072
	// pkt_len : 4 * 128 = 512
	//-------------------------- Total __shared__ mem Usage : 14,336 + 512

	if(tid == TOTAL_T_NUM - 1){
		START_RED
		printf("[%s] tid %d is alive!\n", __FUNCTION__, tid);
		END
	}

	__syncthreads();
	while(true){ // Persistent Kernel (for every threads)
		// 3-threads to be grouped. ex) 0,1,2 --> idx 0; 3,4,5 --> idx 1; ...
		//-------------------------- Multi threads Job --------------------------------------------
		__syncthreads();
		if(readNoCache(&p_buf->rx_buf_idx[tid/AES_T_NUM]) == chain_seq){
			//-------------------------- Single threads Job --------------------------------------------
			//__syncthreads();
#if 1
			if(tid % AES_T_NUM == 0){
				p_buf->rx_buf[0x1000 * (tid/AES_T_NUM) + (PKT_SIZE - 4)] = 0; // padlen 
				p_buf->rx_buf[0x1000 * (tid/AES_T_NUM) + (PKT_SIZE - 4) + 1] = IPPROTO_IPIP; // next-hdr (Meaning "IP within IP)
				/* For Reference...
					 IPPROTO_IP = 0
					 IPPROTO_ICMP = 1
					 IPPROTO_IPIP = 4
					 IPPROTO_TCP = 6
					 IPPROTO_UDP = 17
					 IPPROTO_ESP = 50
				 */
				atomicAdd(ctr, 1); // same "ctr" value for grouped 3-threads. (counter) AES-CTR Mode
				IV[cur_tid][15] = *ctr & 0xFF;
				IV[cur_tid][14] = (*ctr >> 8) & 0xFF; // CKJUNG, 1 Byte = 8bits means, Octal notation
				IV[cur_tid][13] = (*ctr >> 16) & 0xFF;
				IV[cur_tid][12] = (*ctr >> 24) & 0xFF;
				for(int i = 0; i < 12; i++)
					IV[cur_tid][i] = 0;

				// Copy our state into private memory
				unsigned char temp, temp2;
				unsigned char overflow = 0;
				char tmp[16];
				for(int i = 15; i != -1; i--) {
					temp = d_nounce[i];
					temp2 = IV[cur_tid][i];
					IV[cur_tid][i] += temp + overflow;
					overflow = ((int)temp2 + (int)temp + (int)overflow > 255);
				}

				AddRoundKey(IV[cur_tid], &d_key[0]);

#if 1
				for(int i = 1; i < 10; i++)
				{
					SubBytes(IV[cur_tid], d_sbox);
					ShiftRows(IV[cur_tid]);
					MixColumns(IV[cur_tid], d_GF2, tmp);
					AddRoundKey(IV[cur_tid], &d_key[4 * i]);
				}
#endif
				SubBytes(IV[cur_tid], d_sbox);
				ShiftRows(IV[cur_tid]);
				AddRoundKey(IV[cur_tid], &d_key[4 * 10]);
			}
#endif
			__syncthreads();
			//-------------------------- Multi threads Job --------------------------------------------
			////////////////// Locating AES Encrypted parts into a pkt  ///////////////////////////////
			for(int i = 0; i < 16; i++){
				aes_tmp[cur_tid][((tid%AES_T_NUM)*16) + i] = p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM)) + sizeof(struct ethhdr) + ((tid%AES_T_NUM) * 16) + i] ^ IV[cur_tid][i];
			}
			for(int i = 0; i < 16; i++){
				p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM)) + sizeof(struct ethhdr) + sizeof(struct iphdr) + sizeof(struct esphdr) + ((tid%AES_T_NUM) * 16) + i] = aes_tmp[cur_tid][((tid%AES_T_NUM)*16) + i]; 
			}
			__syncthreads();
#if 1
			//-------------------------- Single threads Job --------------------------------------------
			if(tid % AES_T_NUM == 0){
				//////////// Proto_type = ESP set! ///////////
				p_buf->rx_buf[0x1000 * (tid/AES_T_NUM) + sizeof(struct ethhdr) + 9] = IPPROTO_ESP; // IPPROTO_ESP = 50
				struct ethhdr* ethh;
				struct iphdr* iph;
				struct esphdr* esph;

				ethh = (struct ethhdr *)&p_buf->rx_buf[0x1000 * (tid/AES_T_NUM)];
				iph = (struct iphdr *)(ethh + 1);
				esph = (struct esphdr *)((uint32_t *)iph + iph->ihl);

				// SPI (Security Parameter Index)
				uint32_t spi = 1085899777;
				HTONS32(spi);

				////////// Set ESP header SPI value ///////////////////
				memcpy(&esph->spi, &spi, 4);
				atomicAdd(seq, 1);

				//////////// Set ESP header SEQ value //////////
				memcpy(&esph->seq, seq, 4);

				// CKJUNG, HMAC-SHA1 From here! /////////////////////////////
				// RFC 2104, H(K XOR opad, H(K XOR ipad, text))
				/**** Inner Digest ****/
				// H(K XOR ipad, text) : 64 Bytes
				uint32_t extended[80];
				sha1_gpu_context ictx;

				sha1_kernel_global_64(&p_buf->rx_buf[(0x1000 * (tid/AES_T_NUM)) + sizeof(struct ethhdr) + sizeof(struct iphdr)], &ictx, extended, 64);
				/**** Outer Digest ****/
				// H(K XOR opad, H(K XOR ipad, text)) : 20 Bytes
				sha1_kernel_global_64(&(ictx.c_state[0]), &octx[cur_tid], extended, 20);
			}
#endif
			__syncthreads();
			//-------------------------- Multi threads Job --------------------------------------------
			// Attach 20-bytes HMAC-SHA authentication digest to packet.
			memcpy(&p_buf->rx_buf[0x1000 * (tid/AES_T_NUM) + 90 + ((tid%AES_T_NUM) * 8)], &(octx[cur_tid].c_state[((tid%AES_T_NUM)*8)]), 8);
			__syncthreads();
			//-------------------------- Single threads Job --------------------------------------------
			if(tid % AES_T_NUM == 0){
				atomicAdd(&pkt_cnt[1], 1);	
				p_buf->rx_buf_idx[tid/AES_T_NUM] = chain_seq+1;
			}
		}
	}
}
