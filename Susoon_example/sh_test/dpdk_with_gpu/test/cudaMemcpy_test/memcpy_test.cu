#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define START_RED printf("\033[1;31m");
#define START_GRN printf("\033[1;32m");
#define START_YLW printf("\033[1;33m");
#define END printf("\033[0m");

#define ONE_SEC (1000 * 1000 * 1000)
#define HALF (1024 * 32 * 64)

#define RAND 0

#define BATCH 0

#define LOOP 0

#if BATCH
#define CASE 11
#else
#define CASE 7
#endif

char * device_buf;
char * host_buf;

int test_cnt;

FILE * data;

uint64_t latency[11] = { 0 };
const char* pkt_size_str[7] = { "32", "64", "128", "256", "512", "1024", "1514"};
const char* batch_size_str[11] = { "64 * 32", "64 * 64", "64 * 128", "64 * 256",	\
		"64 * 512", "64 * 1024", "64 * 1024 * 2", "64 * 1024 * 4",		 	\
		"64 * 1024 * 8", "64 * 1024 * 16", "64 * 1024 * 32"};
int pkt_size[7] = { 32, 64, 128, 256, 512, 1024, 1514};
int batch_size[11] = { 64 * 32, 64 * 64, 64 * 128, 64 * 256, 64 * 512, 64 * 1024,\
 64 * 1024 * 2, 64 * 1024 * 4, 64 * 1024 * 8, 64 * 1024 * 16, 64 * 1024 * 32};

int start[11] = { 0 };
int end[11] = { 0 };

int monotonic_time() 
{
	struct timespec timespec;
	clock_gettime(CLOCK_MONOTONIC, &timespec);
	return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

void call_data(int size)
{
	data = fopen("data.txt", "r");

	fseek(data, 0, SEEK_SET);
	for(int i = 0; i < size; i++)
	{
		fscanf(data, "%c", host_buf + i);
	}

	fclose(data);
}

void once(void)
{
	int i = 0;
	int * size;

#if BATCH
	size = batch_size;
#else
	size = pkt_size;
#endif

	while(i < test_cnt)
	{
		for(int j = 0; j < CASE; j++)
		{
#if RAND
			call_data(size[j] * 2);
			start[j] = monotonic_time();
			cudaMemcpy(device_buf, host_buf + rand() % size[j], size[j], cudaMemcpyHostToDevice);
#else
			call_data(size[j]);
			start[j] = monotonic_time();
			hipMemcpy(device_buf, host_buf, size[j], hipMemcpyHostToDevice);
#endif
			end[j] = monotonic_time();
			latency[j] += end[j] - start[j];
			hipMemset(device_buf, 0, size[j]);
		}
		i++;
	}

	for(i = 0; i < CASE; i++)
	{
		latency[i] /= (uint64_t)test_cnt;
	}
}

void loop(int loop_cnt)
{
	int i = 0;

	int * size;
	
#if BATCH
	size = batch_size;
#else
	size = pkt_size;
#endif

	while(i < test_cnt)
	{
		for(int j = 0; j < CASE; j++) 
		{
#if RAND
			call_data(size[j] * 2);
			start[j] = monotonic_time();
			for(int k = 0; k < loop_cnt; k++)
			{
			cudaMemcpy(device_buf, host_buf + rand() % size[j], size[j], cudaMemcpyHostToDevice);
			}
#else
			call_data(size[j]);
			start[j] = monotonic_time();
			for(int k = 0; k < loop_cnt; k++)
			{
			hipMemcpy(device_buf, host_buf, size[j], hipMemcpyHostToDevice);
			}
#endif
			end[j] = monotonic_time();
			latency[j] += end[j] - start[j];
		}
		i++;
	}

	for(i = 0; i < CASE; i++)
	{
		latency[i] /= (uint64_t)test_cnt;
	}
}

void print_result(void)
{
	START_RED
	printf("\n\n___________________TEST START____________________\n\n");
	END

	START_YLW
#if RAND
	printf("RANDOM DATA TEST!\n");
#else
	printf("NORMAL DATA TEST!\n");
#endif
	END

	START_GRN
#if LOOP
	printf("TEST WAS RUNNED %d TIMES!\n", test_cnt);
#else
	printf("TEST WAS RUNNED ONCE!\n");
#endif
	END

	const char ** size_str;

#if BATCH 
	size_str = batch_size_str;
#else
	size_str = pkt_size_str;
#endif
 
	for(int i = 0; i < CASE; i++)
	{
		printf("data size : %s, latency : %ld\n", size_str[i], latency[i]);
	}

	START_RED
	printf("\n___________________TEST END____________________\n\n\n");
	END
}


int main(void)
{
	int loop_cnt;

	srand(time(NULL));

	host_buf = (char *)calloc(HALF * 2, sizeof(char));
	hipMalloc((void**)&device_buf, HALF * sizeof(char));
	hipMemset(device_buf, 0 ,HALF * sizeof(char)); 

	printf("Enter the test_cnt\n");
	scanf("%d", &test_cnt);

#if LOOP
	printf("Enter the loop_cnt\n");
	scanf("%d", &loop_cnt);
	loop(loop_cnt);
#else
	once();
#endif

	print_result();

	hipFree(device_buf);

	return 0;
}
