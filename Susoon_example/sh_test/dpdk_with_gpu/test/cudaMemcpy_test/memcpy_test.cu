#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define START_RED printf("\033[1;31m");
#define START_GRN printf("\033[1;32m");
#define START_YLW printf("\033[1;33m");
#define END printf("\033[0m");

#define ONE_SEC (1000 * 1000 * 1000)
#define HALF (1024 * 32 * 64)

#define RAND 0

#define LOOP 0

#define SEL 1

#define CASE 17

char * device_buf;
char * host_buf;

int test_cnt;

FILE * data = fopen("data.txt", "r");

uint64_t latency[17] = { 0 };
const char* size_str[17] = { "64", "128", "256", "512", "1024", "1514",\
		 "64 * 32", "64 * 64", "64 * 128", "64 * 256",	\
		"64 * 512", "64 * 1024", "64 * 1024 * 2", "64 * 1024 * 4",		 	\
		"64 * 1024 * 8", "64 * 1024 * 16", "64 * 1024 * 32"};
int size[17] = { 64, 128, 256, 512, 1024, 1514, 64 * 32, 64 * 64,\
			 64 * 128, 64 * 256, 64 * 512, 64 * 1024,\
			 64 * 1024 * 2, 64 * 1024 * 4, 64 * 1024 * 8,\
			 64 * 1024 * 16, 64 * 1024 * 32};

int start[17] = { 0 };
int end[17] = { 0 };

int monotonic_time() 
{
	struct timespec timespec;
	clock_gettime(CLOCK_MONOTONIC, &timespec);
	return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

void call_data(int size)
{
	fseek(data, 0, SEEK_SET);
	for(int i = 0; i < size; i++)
	{
		fscanf(data, "%c", host_buf + i);
	}
}

void once(void)
{
	int i = 0;
	
	int skip = 0;

	while(i < test_cnt)
	{
#if RAND
#else
		call_data(HALF * 2);
		skip = 0;
#endif
		for(int j = 0; j < CASE; j++)
		{
#if RAND
			call_data(size[j] * 2);
			start[j] = monotonic_time();
			cudaMemcpy(device_buf, host_buf + rand() % size[j], size[j], cudaMemcpyHostToDevice);
#else
			start[j] = monotonic_time();
			hipMemcpy(device_buf, host_buf + skip, size[j], hipMemcpyHostToDevice);
			skip += size[j];
#endif
			end[j] = monotonic_time();
			latency[j] += end[j] - start[j];
			hipMemset(device_buf, 0, size[j]);
		}
		i++;
	}

	for(i = 0; i < CASE; i++)
	{
		latency[i] /= (uint64_t)test_cnt;
	}
}

void loop(int loop_cnt)
{
	int i = 0;

	while(i < test_cnt)
	{
		for(int j = 0; j < CASE; j++) 
		{
#if RAND
			call_data(size[j] * 2);
			start[j] = monotonic_time();
			for(int k = 0; k < loop_cnt; k++)
			{
			cudaMemcpy(device_buf, host_buf + rand() % size[j], size[j], cudaMemcpyHostToDevice);
			}
#else
			call_data(size[j]);
			start[j] = monotonic_time();
			for(int k = 0; k < loop_cnt; k++)
			{
			hipMemcpy(device_buf, host_buf, size[j], hipMemcpyHostToDevice);
			}
#endif
			end[j] = monotonic_time();
			latency[j] += (end[j] - start[j]) / (uint64_t)loop_cnt;
		}
		i++;
	}

	for(i = 0; i < CASE; i++)
	{
		latency[i] /= (uint64_t)test_cnt;
	}
}

void same_cnt_loop(void)
{
	int i = 0;

	int loop_cnt = size[16];
	int cur_loop_cnt;

	while(i < test_cnt)
	{
		for(int j = 0; j < CASE; j++) 
		{
			cur_loop_cnt = loop_cnt / size[j];
#if RAND
			call_data(size[j] * 2);
			start[j] = monotonic_time();
			for(int k = 0; k < cur_loop_cnt; k++)
			{
			cudaMemcpy(device_buf, host_buf + rand() % size[j], size[j], cudaMemcpyHostToDevice);
			}
#else
			call_data(size[j]);
			start[j] = monotonic_time();
			for(int k = 0; k < cur_loop_cnt; k++)
			{
			hipMemcpy(device_buf, host_buf, size[j], hipMemcpyHostToDevice);
			}
#endif
			end[j] = monotonic_time();
			latency[j] += end[j] - start[j];
		}
		i++;
	}

	for(i = 0; i < CASE; i++)
	{
		latency[i] /= (uint64_t)test_cnt;
	}
}

void print_result(void)
{
	START_RED
	printf("\n\n___________________TEST START____________________\n\n");
	END

	START_YLW
#if RAND
	printf("RANDOM DATA TEST!\n");
#else
	printf("NORMAL DATA TEST!\n");
#endif
	END

	START_GRN
#if LOOP
	printf("TEST WAS RUNNED %d TIMES!\n", test_cnt);
#elif SEL
	printf("TEST WAS RUNNED SAME TIMES!\n");
#else
	printf("TEST WAS RUNNED ONCE!\n");
#endif
	END

	for(int i = 0; i < CASE; i++)
	{
		printf("data size : %s, latency : %ld\n", size_str[i], latency[i]);
	}

	START_RED
	printf("\n___________________TEST END____________________\n\n\n");
	END
}


int main(void)
{

	srand(time(NULL));

	host_buf = (char *)calloc(HALF * 2, sizeof(char));
	hipHostAlloc((void**)&device_buf, HALF * sizeof(char), hipHostMallocDefault);
	hipMemset(device_buf, 0 ,HALF * sizeof(char)); 

	printf("Enter the test_cnt\n");
	scanf("%d", &test_cnt);

#if LOOP
	int loop_cnt;

	printf("Enter the loop_cnt\n");
	scanf("%d", &loop_cnt);
	loop(loop_cnt);
#elif SEL
	same_cnt_loop();
#else
	once();
#endif

	print_result();

	hipFree(device_buf);

	fclose(data);

	return 0;
}
