//#include "thand.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

__global__ void Check_gpu(void)
{
#if 0
	while(true)
	{
		//printf("\n\n\n\n\n\n");
		printf("____________GPU function is called______________\n");
		//printf("\n\n\n\n\n\n");
	}
#else
		printf("____________GPU function is called______________\n");
#endif
}

extern "C"
void Check(void)
{
	printf("Check!!\n");
	Check_gpu<<<1,1>>>();
}

extern "C"
void cudasynch(void)
{
	hipDeviceSynchronize();
}

int main(void)
{
	printf("___1____\n");
	Check_gpu<<<1,1>>>();
	printf("___2____\n");
	hipDeviceSynchronize();
	printf("___3____\n");
	return 0;
}
