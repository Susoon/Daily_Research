//#include "thand.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <pthread.h>

int * count;

__global__ void Check_gpu(int * count)
{
#if 1
	while(true)
	{
		//printf("\n\n\n\n\n\n");
		//printf("____________GPU function is called______________\n");
		atomicAdd(count, 1);
		//printf("\n\n\n\n\n\n");
	}
#else
		printf("____________GPU function is called______________\n");
#endif
}

extern "C"
void Check(void)
{
	printf("Check!!\n");
	Check_gpu<<<1,512>>>(count);
}

extern "C"
void cudasynch(void)
{
	hipDeviceSynchronize();
}

void* get_cnt(void * data)
{
	int j = 0;
	while(j > -1)
	{
		int ret = 0, tmp;
		hipMemcpy(&ret, count, sizeof(int), hipMemcpyDeviceToHost);
		printf("In CPU : count = %d\n", ret);
		j++;
	}
}

int main(void)
{
	pthread_t thread;

	hipMalloc((void**)&count, sizeof(int));
	hipMemset(count, 0, sizeof(int));
	printf("___1____\n");
	Check();
	printf("___2____\n");
	pthread_create(&thread, NULL, get_cnt, NULL);
	//get_cnt();
	hipDeviceSynchronize();
	printf("___3____\n");
	return 0;
}
