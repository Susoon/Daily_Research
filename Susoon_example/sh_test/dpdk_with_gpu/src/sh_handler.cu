#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define RING_BATCH_SIZE 8
#define DPDK_RING_SIZE (BATCH_SIZE * RING_BATCH_SIZE)
#define RTE_ETH_CRC_LEN 5
#define TOTAL_PKT_SIZE (PKT_SIZE + RTE_ETH_CRC_LEN)
#define ONELINE 6
#define DUMP 0

unsigned char * rx_pkt_buf;
unsigned char * tx_pkt_buf;
static int idx;
static int * rx_pkt_cnt;
static int tx_idx;
static int * pkt_size;

static int * flag;

void check_error(hipError_t err)
{	
	if(err == hipSuccess)
		printf("Success!!!!!\n");
	else if(err == hipErrorLaunchTimeOut)
		printf("LaunchTimeout!!!!!!\n");
	else if(err == hipErrorInvalidDevicePointer)
		printf("InvalidDevicePointer");
	else
		printf("Cannot find cause!!!!!!\n");
}

__global__ void print_gpu(unsigned char* d_pkt_buf)
{
	int i;
	START_RED
	printf("[GPU]:\n");
	for(i = 0; i < BATCH_SIZE; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		if(i != 0 && i % PKT_SIZE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
	END
}

__device__ void mani_pkt_gpu(unsigned char * d_pkt_buf)
{
	int i;
	unsigned char tmp[6] = { 0 };

	// Swap mac
	for(i = 0; i < 6; i++){
		tmp[i] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 6];
		d_pkt_buf[i + 6] = tmp[i];
	}
	// Swap ip
	for(i = 26; i < 30; i++){
		tmp[i-26] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 4];
		d_pkt_buf[i + 4] = tmp[i-26];
	}
	// Swap port
	for(i = 34; i < 36; i++){
		tmp[i-34] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 2];
		d_pkt_buf[i + 2] = tmp[i-34];
	}	
	//Manipulatate data
	for(i = 36; i < PKT_SIZE; i++){
		d_pkt_buf[i] = 0;
	}
}

extern "C"
uint64_t monotonic_time() {
        struct timespec timespec;
        clock_gettime(CLOCK_MONOTONIC, &timespec);
        return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

extern "C"
void copy_to_gpu(unsigned char* buf, int size)
{
	hipMemcpy(rx_pkt_buf + (idx * BATCH_SIZE), buf, sizeof(unsigned char)* PKT_SIZE * size, hipMemcpyHostToDevice);

	check_error(hipMemset(flag, 1, sizeof(int)));
	check_error(hipMemcpy(pkt_size, &size, sizeof(int), hipMemcpyHostToDevice));

	idx++;
	if(idx == 512)
		idx = 0;

#if DUMP
	print_gpu<<<1,1>>>(rx_pkt_buf + (idx * BATCH_SIZE));
	hipDeviceSynchronize();
#endif
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;

	idx = 0;
	tx_idx = 0;

	ASSERTRT(hipMalloc((void**)&rx_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(rx_pkt_buf, 0, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&tx_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(tx_pkt_buf, 0, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&rx_pkt_cnt, sizeof(unsigned int)));
  	ASSERTRT(hipMemset(rx_pkt_cnt, 0, sizeof(unsigned int)));

	ASSERTRT(hipMalloc((void**)&pkt_size, sizeof(unsigned int)));
  	ASSERTRT(hipMemset(pkt_size, 0, sizeof(unsigned int)));

	ASSERTRT(hipMalloc((void**)&flag, sizeof(unsigned int)));
  	ASSERTRT(hipMemset(flag, 0, sizeof(unsigned int)));

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

extern "C"
int get_rx_cnt(void)
{
	int rx_cur_pkt = 0;
	
	hipMemcpy(&rx_cur_pkt, rx_pkt_cnt, sizeof(int), hipMemcpyDeviceToHost);
	hipMemset(rx_pkt_cnt, 0, sizeof(int));	

	return rx_cur_pkt;
}

extern "C"
void get_tx_buf(unsigned char* tx_buf)
{
	printf("get_tx_buf!!!!!\n");
	hipMemcpy(tx_buf, tx_pkt_buf + (tx_idx * BATCH_SIZE), sizeof(unsigned char) * BATCH_SIZE, hipMemcpyDeviceToHost);

	tx_idx++;
	if(tx_idx == RING_BATCH_SIZE)
		tx_idx = 0;
}

__global__ void gpu_monitoring_loop(unsigned char * rx_pkt_buf, unsigned char * tx_pkt_buf, int * rx_pkt_cnt, int * pkt_size, int * flag)
{
	int i = 0;
	volatile int infinity = 1;
	while(infinity)
	{
		/*
		printf("cur buf = %d\n", rx_pkt_buf[i * PKT_SIZE]);
		if(rx_pkt_buf[i * PKT_SIZE] == 0)
			continue;
		*/
		/*
		if(*flag)
			continue;
		*/

		//printf("rx_pkt_cnt = %d, pkt_size = %d\n", *rx_pkt_cnt, *pkt_size);
		atomicAdd(rx_pkt_cnt, *pkt_size);
		mani_pkt_gpu(rx_pkt_buf + (i * PKT_SIZE));
		memset(rx_pkt_buf + (i * PKT_SIZE), 0, PKT_SIZE); 		

		i++;
		if(i == DPDK_RING_SIZE / PKT_SIZE)
			i = 0;
			
		memcpy(tx_pkt_buf, rx_pkt_buf, PKT_SIZE);
		
		atomicAdd(flag, -1);

	}
}

extern "C"
void gpu_monitor(void)
{
	gpu_monitoring_loop<<<1,1>>>(rx_pkt_buf, tx_pkt_buf, rx_pkt_cnt, pkt_size, flag);
}

/*
extern "C"
void monitoring_loop(void){

#if 0
	int prev_pkt[2] = {0,}, cur_pkt[2] = {0,};
	double pkts[2];
	char units[] = {' ', 'K', 'M', 'G', 'T'};
	char pps[2][40];
	char bps[2][40];
	int buf_idx[512] = {0,};
	int p_size=0;
	int i, j;
#endif

	int buf_idx = 0;
	uint64_t last_stats_printed = monotonic_time();
	uint64_t time;
	bool copied_for_tx = false;
	
	while(true)                                           
	{
		buf_idx++;
		if(buf_idx == DPDK_RING_SIZE / PKT_SIZE)
			buf_idx = 0;

		if(rx_pkt_buf + (buf_idx * PKT_SIZE) == 0)
			continue;

		mani_pkt_gpu<<<1,1>>>(rx_pkt_buf + (buf_idx * PKT_SIZE), tmp, rx_pkt_cnt);
		time = monotonic_time();
		if(time - last_stats_printed > ONESEC){

			last_stats_printed = time;

			hipError_t err = hipMemcpy(&rx_cur_pkt, &rx_pkt_cnt, sizeof(int), hipMemcpyDeviceToHost);
			hipError_t err2 = hipMemcpy(&tx_cur_pkt, &tx_pkt_cnt, sizeof(int), hipMemcpyDeviceToHost);


			if(err != hipSuccess || err2 != hipSuccess)
			{
				printf("hipMemcpy, pkt_cnt, error!\n");
			}
			system("clear");	
			printf("receive packet total : %d\n", rx_cur_pkt);
		
			hipMemcpy(&tx_pkt_buf,  	

#if 0
			for(i = 0; i < 2; i++){
				double tmp_pps;
				double tmp;
				//double batch;
				if (prev_pkt[i] != cur_pkt[i]){ // If we got a traffic flow
					pkts[i] = (double)(cur_pkt[i] - prev_pkt[i]);

#if 0
					if(i == 0)
						printf("RX_pkts: %d\n", (int)pkts[i]); 
					else
						printf("TX_pkts: %d\n", (int)pkts[i]); 
#endif
					tmp = tmp_pps = pkts[i];
					//batch = tmp/BATCH;
					for(j = 0; tmp >= 1000 && j < sizeof(units)/sizeof(char) -1; j++)
						tmp /= 1000;
					sprintf(pps[i],"%.3lf %c" ,tmp, units[j]);
#if TX
					p_size = PKT_SIZE;
#endif

					//tmp = pkts[i] * p_size * 8; // Bytes -> Bits
					tmp = pkts[i] * p_size * 8 + tmp_pps * 20 * 8; // Add IFG also, 20.01.15, CKJUNG
					for(j = 0; tmp >= 1000 && j < sizeof(units)/sizeof(char) -1; j++)
						tmp /= 1000;
					sprintf(bps[i],"%.3lf %c" ,tmp, units[j]);

					if(i == 0)
						printf("[RX] pps: %spps %sbps, pkt_size: %d \n", pps[i], bps[i], p_size);
					else{
						printf("[TX] pps: %spps %sbps, pkt_size: %d \n", pps[i], bps[i], p_size);
					}
				}else{
					if(i == 0)
						printf("[RX] pps: None\n");
					else
						printf("[TX] pps: None\n");
				}
			}
			for(i = 0; i<512; i++)
			{
				if(i % 32 ==0)
					printf("\n");
				if(buf_idx[i] == 1){
					START_GRN
						printf("%d ", buf_idx[i]);
					END
				}else if(buf_idx[i] == 2){
					START_RED
						printf("%d ", buf_idx[i]);
					END
				}else if(buf_idx[i] == 3){
					START_BLU
						printf("%d ", buf_idx[i]);
					END
				}else{
					printf("%d ", buf_idx[i]);
				}
			}
			printf("\n");

			prev_pkt[0] = cur_pkt[0];
			prev_pkt[1] = cur_pkt[1];
		}
		//sleep(1); 
	} 
#endif                                                                 
}
*/
