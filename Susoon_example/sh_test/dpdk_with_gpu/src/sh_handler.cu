#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define RING_BATCH_NUM 512
#define DPDK_RING_SIZE (BATCH_SIZE * RING_BATCH_NUM)

#define ONELINE 6

#define DUMP 0
#define TX 0

unsigned char * rx_pkt_buf;
unsigned char * tx_pkt_buf;
static int idx;
int * rx_pkt_cnt;
int tx_idx;

int * batch_size;

extern "C"
int monotonic_time() {
        struct timespec timespec;
        clock_gettime(CLOCK_MONOTONIC, &timespec);
        return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

__global__ void print_gpu(unsigned char* d_pkt_buf)
{
	int i;
	START_RED
	printf("[GPU]:\n");
	for(i = 0; i < BATCH_SIZE; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		if(i != 0 && i % PKT_SIZE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
	END
}

__device__ void mani_pkt_gpu(unsigned char * d_pkt_buf)
{
	int i;
	unsigned char tmp[6] = { 0 };

	// Swap mac
	for(i = 0; i < 6; i++){
		tmp[i] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 6];
		d_pkt_buf[i + 6] = tmp[i];
	}
	// Swap ip
	for(i = 26; i < 30; i++){
		tmp[i-26] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 4];
		d_pkt_buf[i + 4] = tmp[i-26];
	}
	// Swap port
	for(i = 34; i < 36; i++){
		tmp[i-34] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 2];
		d_pkt_buf[i + 2] = tmp[i-34];
	}	
	//Manipulatate data
	for(i = 36; i < PKT_SIZE; i++){
		d_pkt_buf[i] = 0;
	}
}

extern "C"
void copy_to_gpu(unsigned char* buf, int size)
{
	hipMemcpy(rx_pkt_buf + (idx * BATCH_SIZE), buf, sizeof(unsigned char)* size, hipMemcpyHostToDevice);

	hipMemcpy(batch_size, &size, sizeof(int), hipMemcpyHostToDevice);

#if DUMP
	print_gpu<<<1,1>>>(rx_pkt_buf + (idx * BATCH_SIZE));
	hipDeviceSynchronize();
#endif

	idx++;
	if(idx == RING_BATCH_NUM)
		idx = 0;
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	idx = 0;
	tx_idx = 0;

	printf("DPDK_RING_SIZE = %d\n",DPDK_RING_SIZE);

	ASSERTRT(hipMalloc((void**)&rx_pkt_buf, DPDK_RING_SIZE));
  	ASSERTRT(hipMemset(rx_pkt_buf, 0, DPDK_RING_SIZE));

	ASSERTRT(hipMalloc((void**)&tx_pkt_buf, DPDK_RING_SIZE));
  	ASSERTRT(hipMemset(tx_pkt_buf, 0, DPDK_RING_SIZE));

	ASSERTRT(hipMalloc((void**)&rx_pkt_cnt, sizeof(int)));
  	ASSERTRT(hipMemset(rx_pkt_cnt, 0, sizeof(int)));

	ASSERTRT(hipMalloc((void**)&batch_size, sizeof(int)));
  	ASSERTRT(hipMemset(batch_size, 0, sizeof(int)));

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

extern "C"
int get_rx_cnt(void)
{
	int rx_cur_pkt = 0;
	static int turn = 0;

	ASSERTRT(hipMemcpy(&rx_cur_pkt, rx_pkt_cnt, sizeof(int), hipMemcpyDeviceToHost));

	hipMemset(rx_pkt_cnt, 0, sizeof(int));	
	turn++;

	return rx_cur_pkt;
}

extern "C"
void get_tx_buf(unsigned char* tx_buf)
{
	printf("get_tx_buf!!!!!\n");

	hipMemcpy(tx_buf, tx_pkt_buf + (tx_idx * BATCH_SIZE), sizeof(unsigned char) * BATCH_SIZE, hipMemcpyDeviceToHost);

	tx_idx++;
	if(tx_idx == RING_BATCH_NUM)
		tx_idx = 0;
}

__global__ void gpu_monitor(unsigned char * rx_pkt_buf, unsigned char * tx_pkt_buf, int * rx_pkt_cnt, int * batch_size)
{
	int mem_index = BATCH_SIZE * threadIdx.x;

	__syncthreads();
	if(rx_pkt_buf[mem_index] != 0)
	{
		__syncthreads();
		rx_pkt_buf[mem_index] = 0;

		__syncthreads();
		atomicAdd(rx_pkt_cnt, *batch_size);
#if TX
		__syncthreads();
		mani_pkt_gpu(rx_pkt_buf + mem_index);
				
		__syncthreads();
		memcpy(tx_pkt_buf + mem_index, rx_pkt_buf + mem_index, BATCH_SIZE);
#endif
	}
}

extern "C"
void gpu_monitor_loop(void)
{
	hipStream_t stream;
	ASSERTRT(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	while(true)
	{
		gpu_monitor<<<1, RING_BATCH_NUM, 0, stream>>>(rx_pkt_buf, tx_pkt_buf, rx_pkt_cnt, batch_size);
		hipDeviceSynchronize();
	}
}

