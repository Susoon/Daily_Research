#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define DPDK_RING_SIZE 4 * 1024 * 1024 //4MB
#define PKT_SIZE 64
#define RTE_ETH_CRC_LEN 4
#define TOTAL_PKT_SIZE (PKT_SIZE + RTE_ETH_CRC_LEN)

#define ORIGIN 0

#define ONELINE 6

__device__ unsigned char * dev_pkt_buf;
unsigned char * host_buf_ptr;

/*
__device__ uint8_t tmp_pkt[60] = {\
0x00, 0x1b, 0x21, 0xbc, 0x11, 0x52, 0xa0, 0x36, 0x9f, 0x03, 0x13, 0x86, 0x08, 0x00, 0x45, 0x10,\
0x00, 0x2e, 0x00, 0x00, 0x40, 0x00, 0x40, 0x11, 0x00, 0x00, 0x0a, 0x00, 0x00, 0x03, 0x0a, 0x00,\
0x00, 0x04, 0x04, 0xd2, 0x04, 0xd2, 0x00, 0x1a, 0x2c, 0xd6, 0x6f, 0x98, 0x26, 0x35, 0x02, 0xc9,\
0x83, 0xd7, 0x8b, 0xc3, 0xf7, 0xb5, 0x20, 0x8d, 0x48, 0x8d, 0xc0, 0x36};
*/

/* Suhwan pinning buffer 02/06 */
extern "C"
int sh_pin_buffer(void)
{
	int ret = 0;
	int retcode;

	retcode = hipHostAlloc((void**)&dev_pkt_buf, sizeof(unsigned char) * TOTAL_PKT_SIZE, hipHostMallocDefault);
	//hipHostGetDevicePointer(&host_buf_ptr, dev_pkt_buf, 0);
	if(retcode == hipErrorOutOfMemory)
	{
		ret = errno;
		printf("hipHostAlloc error (errno=%d)\n", ret);
	}

    return ret;
}

__global__ void print_gpu(unsigned char* d_pkt_buf)
{
	int i;
	printf("[GPU]:\n");
	for(i = 0; i < TOTAL_PKT_SIZE; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
}

#if ORIGIN

extern "C"
void copy_to_gpu(unsigned char* buf, int size)
{
	unsigned char * d_pkt_buf;
	hipMalloc((void**)&d_pkt_buf, sizeof(unsigned char)*1500);
	printf("____1__________copy_to_gpu____\n");
	hipMemcpy(d_pkt_buf+(512*0x1000)+(0x1000), buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
	print_gpu<<<1,1>>>(d_pkt_buf+(512*0x1000)+(0x1000));
	printf("____2__________copy_to_gpu____\n");
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t _pkt_buffer_size = DPDK_RING_SIZE;// 4MB, for rx,tx ring
	size_t pkt_buffer_size = (_pkt_buffer_size + GPU_PAGE_SIZE - 1) & GPU_PAGE_MASK;
	
	ASSERTRT(hipMalloc((void**)&dev_pkt_buf, pkt_buffer_size));
	ASSERTRT(hipMemset(dev_pkt_buf, 0, pkt_buffer_size));

	START_GRN
	printf("[Done]____GPU mem set for dpdk__\n");
	END
}

#else

extern "C"
void copy_to_pinned_buffer(unsigned char * d_pkt_buf, int size)
{
	printf("___1___________copy_to_pinned_buffer___\n");
	hipMemcpy(host_buf_ptr, d_pkt_buf, size, hipMemcpyDeviceToDevice);		
	printf("___2___________copy_to_pinned_buffer___\n");
}

extern "C" 
void copy_to_gpu(unsigned char* buf, int size)
{
	unsigned char * d_pkt_buf;
	hipMalloc((void**)&d_pkt_buf, sizeof(unsigned char) * TOTAL_PKT_SIZE);
	printf("____1__________copy_to_gpu____\n");
	hipMemcpy(d_pkt_buf, buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
	print_gpu<<<1,1>>>(d_pkt_buf);
	copy_to_pinned_buffer(d_pkt_buf, size);
	printf("____2__________copy_to_gpu____\n");
}

extern "C" 
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = TOTAL_PKT_SIZE;

	hipHostGetDevicePointer((void**)&dev_pkt_buf, (void*)host_buf_ptr, 0);

	ASSERTRT(hipMalloc((void**)&host_buf_ptr, pkt_buffer_size));
	ASSERTRT(hipMemset(host_buf_ptr, 0, pkt_buffer_size));

	START_GRN
	printf("[Done]____GPU mem set for dpdk__\n");
	END
}

__global__ void print_pinned_buffer(unsigned char * d_pkt_buf)
{
	int i;
	printf("[Pinned Buffer]:\n");
	for(i = 0; i < TOTAL_PKT_SIZE; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
}

#endif

__device__ void read_loop(void)
{
	while(1)
	{
		START_YLW
		printf("____________Dump Packet in GPU____________\n");
		END
		print_pinned_buffer<<<1,1>>>(dev_pkt_buf);
	}

}

