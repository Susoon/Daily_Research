#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define DPDK_RING_SIZE 2 * 1024 * 1024 //2MB
#define PKT_SIZE 64
#define RTE_ETH_CRC_LEN 4
#define TOTAL_PKT_SIZE (PKT_SIZE + RTE_ETH_CRC_LEN)

#define ONELINE 6

#define RING_CHECK 0

unsigned char * pinned_pkt_buf;
unsigned char * zero_arr;
static int idx;

__global__ void Check_gpu(void)
{
	printf("GPU function called\n");
}

void Check_buf(unsigned char * buf)
{
	printf("\n%dth pkt_dump: \n", idx);
	for(int i = 0; i < DPDK_RING_SIZE; i +=  0x1000)
	{
		if(i % (0x1000 * 32) == 0)
			printf("\n");
		if(i / 0x1000 == idx)
			START_GRN
		printf("%02x ", buf[i]);
		END
	}
	printf("\n");
}

void Dump_fct(unsigned char * buf, int size)
{
	printf("%dth pkt_dump: \n", idx);
	for(int i = 0; i < TOTAL_PKT_SIZE; i++){
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		printf("%02x ", buf[i]);
		}
	printf("\n");
}

__global__ void print_gpu(unsigned char* d_pkt_buf)
{
	int i;
	printf("[GPU]:\n");
	for(i = 0; i < TOTAL_PKT_SIZE; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
}

#if RING_CHECK

extern "C" 
void copy_to_gpu(unsigned char* buf, int size)
{
	memcpy(pinned_pkt_buf + (idx * 0x1000), buf, size);

	//Dump_fct(pinned_pkt_buf, size);
	print_gpu<<<1,1>>>(pinned_pkt_buf + (idx * 0x1000));
	
	Check_buf(pinned_pkt_buf);	
	memcpy(pinned_pkt_buf + (idx * 0x1000), zero_arr, size);

	idx++;
	if(idx == 512)
		idx = 0;
}


extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;
	idx = 0;

	pinned_pkt_buf = (unsigned char*)calloc(pkt_buffer_size, sizeof(unsigned char));
	zero_arr = (unsigned char*)calloc(pkt_buffer_size, sizeof(unsigned char));

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

#else

extern "C"
void copy_to_gpu(unsigned char* buf, int size)
{
	hipMemcpy(pinned_pkt_buf + (idx * 0x1000), buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);

	//print_gpu<<<1,1>>>(pinned_pkt_buf + (idx * 0x1000));

	//Check_gpu<<<1,1>>>();
	
	idx++;
	if(idx == 512)
		idx = 0;
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;
	idx = 0;
	ASSERTRT(hipMalloc((void**)&pinned_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(pinned_pkt_buf, 0, pkt_buffer_size));

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

#endif

