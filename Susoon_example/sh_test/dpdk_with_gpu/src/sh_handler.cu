#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define ONELINE 6

#define DUMP 0

unsigned char * rx_pkt_buf;
static int idx;
int * rx_pkt_cnt;

int * pkt_batch_num;

extern "C"
int monotonic_time() {
        struct timespec timespec;
        clock_gettime(CLOCK_MONOTONIC, &timespec);
        return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

__global__ void gpu_monitor(unsigned char * rx_pkt_buf, int * rx_pkt_cnt, int * pkt_batch_num);

#if DUMP

__global__ void print_gpu(unsigned char* d_pkt_buf, int * pkt_num)
{
	int i;
	int total_pkt_num = *pkt_num * PKT_SIZE;
	START_RED
	printf("[GPU]: pkt_num = %d\n", *pkt_num);
	for(i = 0; i < total_pkt_num; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		if(i != 0 && i % PKT_SIZE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n\n");
	END
}

#endif

extern "C"
int copy_to_gpu(unsigned char* buf, int pkt_num)
{

	ASSERTRT(hipMemcpy(rx_pkt_buf + (idx * PKT_BATCH_SIZE), buf, sizeof(unsigned char) * pkt_num * PKT_SIZE, hipMemcpyHostToDevice));

	hipMemcpy(pkt_batch_num + idx, &pkt_num, sizeof(int), hipMemcpyHostToDevice);
#if LAUNCH
	hipStream_t stream;
	ASSERTRT(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	gpu_monitor<<<1, THREAD_NUM, 0, stream>>>(rx_pkt_buf, rx_pkt_cnt, pkt_batch_num);
	hipDeviceSynchronize();
	hipStreamDestroy(stream);
#endif

#if DUMP
	print_gpu<<<1,1>>>(rx_pkt_buf + (idx * PKT_BATCH_SIZE), pkt_batch_num + idx);
	hipDeviceSynchronize();
#endif

	idx++;
	if(idx == BLOCK_NUM)
		idx = 0;
	
	return 1;
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	idx = 0;

	START_BLU
#if POLL
	printf("__________POLLING VERSION___________\n");
#else
	printf("__________KERNEL LAUNCH VERSION___________\n");
#endif
	printf("RING_SIZE = %d\n", RING_SIZE);
	printf("PKT_SIZE = %d, PKT_BATCH = %d + %d\n", PKT_SIZE, PKT_BATCH - RX_NB, RX_NB);
	END

	ASSERTRT(hipMalloc((void**)&rx_pkt_buf, RING_SIZE));
  	ASSERTRT(hipMemset(rx_pkt_buf, 0, RING_SIZE));

	ASSERTRT(hipMalloc((void**)&rx_pkt_cnt, sizeof(int)));
  	ASSERTRT(hipMemset(rx_pkt_cnt, 0, sizeof(int)));

	ASSERTRT(hipMalloc((void**)&pkt_batch_num, sizeof(int) * BLOCK_NUM));
  	ASSERTRT(hipMemset(pkt_batch_num, 0, sizeof(int) * BLOCK_NUM));

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

extern "C"
int get_rx_cnt(void)
{
	int rx_cur_pkt = 0;

	hipMemcpy(&rx_cur_pkt, rx_pkt_cnt, sizeof(int), hipMemcpyDeviceToHost);

	hipMemset(rx_pkt_cnt, 0, sizeof(int));	

	return rx_cur_pkt;
}

__global__ void gpu_monitor(unsigned char * rx_pkt_buf, int * rx_pkt_cnt, int * pkt_batch_num)
{
	int mem_index = PKT_BATCH_SIZE * threadIdx.x;

	__syncthreads();
	if(pkt_batch_num[threadIdx.x] != 0 && rx_pkt_buf[mem_index + ((pkt_batch_num[threadIdx.x] - 1) * PKT_SIZE)] != 0)
	{
		__syncthreads();
		rx_pkt_buf[mem_index + ((pkt_batch_num[threadIdx.x] - 1) * PKT_SIZE)] = 0;

		__syncthreads();
		atomicAdd(rx_pkt_cnt, pkt_batch_num[threadIdx.x]);
		
	}
}

extern "C"
void gpu_monitor_loop(void)
{
	hipStream_t stream;
	ASSERTRT(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	while(true)
	{
		gpu_monitor<<<1, THREAD_NUM, 0, stream>>>(rx_pkt_buf, rx_pkt_cnt, pkt_batch_num);
		hipDeviceSynchronize();
	}
}

