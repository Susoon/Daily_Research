#include "hip/hip_runtime.h"
#include "rx_handler.h"
#include "gdnio.h"
#include "packet_man.h"
#include "common.hpp"
#include "mydrv/mydrv.h"
#include "pkts.h"

#define PKT_SIZE 64

#define OUT cout
using namespace std;

unsigned char* d_pkt_buffer;
// 19.09.02. CKJUNG
unsigned char* gtx_desc;
struct pkt_buf *p_buf;
int *pkt_cnt;
int *pkt_size;          
unsigned int *ctr; // used in ipsec? 19.06.27      

static int idx;

void *ixgbe_bar0_host_addr, *io_addr, *tx_desc, *rx_desc, *tx_desc_addr,  *rx_desc_addr;
uint64_t *gddr_dma_addr;

/*
__device__ uint8_t tmp_pkt[60] = {\
0x00, 0x1b, 0x21, 0xbc, 0x11, 0x52, 0xa0, 0x36, 0x9f, 0x03, 0x13, 0x86, 0x08, 0x00, 0x45, 0x10,\
0x00, 0x2e, 0x00, 0x00, 0x40, 0x00, 0x40, 0x11, 0x00, 0x00, 0x0a, 0x00, 0x00, 0x03, 0x0a, 0x00,\
0x00, 0x04, 0x04, 0xd2, 0x04, 0xd2, 0x00, 0x1a, 0x2c, 0xd6, 0x6f, 0x98, 0x26, 0x35, 0x02, 0xc9,\
0x83, 0xd7, 0x8b, 0xc3, 0xf7, 0xb5, 0x20, 0x8d, 0x48, 0x8d, 0xc0, 0x36};
*/


// CKJUNG, 18.10.29. 
__device__ void DumpPacket_raw(unsigned char* buf, int len)
{
	int i;

	START_YLW
	printf("[START]___________________________________________\n");
	END
	printf("DumpPkt_____________________________________HEX___\n");
	for(i = 0; i < len; i++)
	{
		if(i % 16 == 0)
			printf("\n");

		printf("%02x ", buf[i]);
	}
	printf("\n____________________________________________HEX___\n\n");

#if 0
	 printf("[START]_____DumpPkt__char___\n");   
	  for(i = 0; i < len; i++)                
			 {                                       
				    if(i % 16 == 0)                       
							     printf("\n");                       
						                                         
						   printf("%02c ", buf[i]);              
							  }                                       
	printf("\n\n[END]_____DumpPkt__char___\n"); 
#endif
#if 0
	printf("DumpPkt_____________________________________DEC___\n");
	for(i = 0; i < len; i++)
	{
		if(i % 16 == 0)
			printf("\n");

		if(buf[i]>='a' && buf[i]<='z'){
			printf("%02d ", xlatcase[buf[i]]);
		}else{
			printf("%02d ", buf[i]);
		}
	}
	printf("\n____________________________________________DEC___\n");
#endif
	START_YLW
	printf("[END]___________________________________________\n\n\n");
	END

}




__device__ void DumpARPPacket(struct arphdr *arph)
//void DumpARPPacket(struct arphdr *arph)
{
	uint8_t *t;

	printf("ARP header: \n");
	printf("Hardware type: %d (len: %d), "
			"protocol type: %d (len: %d), opcode: %d\n", 
			//ntohs(arph->ar_hrd), arph->ar_hln, 
			NTOHS(arph->ar_hrd), arph->ar_hln, 
			//ntohs(arph->ar_pro), arph->ar_pln, ntohs(arph->ar_op));
			NTOHS(arph->ar_pro), arph->ar_pln, NTOHS(arph->ar_op));
	t = (uint8_t *)&arph->ar_sip;
	printf("Sender IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_sha[0], arph->ar_sha[1], arph->ar_sha[2], 
			arph->ar_sha[3], arph->ar_sha[4], arph->ar_sha[5]);
	t = (uint8_t *)&arph->ar_tip;
	printf("Target IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_tha[0], arph->ar_tha[1], arph->ar_tha[2], 
			arph->ar_tha[3], arph->ar_tha[4], arph->ar_tha[5]);
}

__device__ uint8_t * EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
{
	struct ethhdr *ethh;
	int i;

	ethh = (struct ethhdr *)buf;

#if 0
	printf("dst_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				dst_haddr[0], dst_haddr[1], 
				dst_haddr[2], dst_haddr[3], 
				dst_haddr[4], dst_haddr[5]);
	printf("src_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				src_haddr[0], src_haddr[1], 
				src_haddr[2], src_haddr[3], 
				src_haddr[4], src_haddr[5]);
#endif

	for (i = 0; i < ETH_ALEN; i++) {
		ethh->h_source[i] = src_haddr[i];
		ethh->h_dest[i] = dst_haddr[i];
	}
	ethh->h_proto = HTONS(h_proto);

	return (uint8_t *)(ethh + 1);
}

__device__ void 
DumpICMPPacket(const char* type, struct icmphdr *icmph, uint32_t saddr, uint32_t daddr)
{
  uint8_t* _saddr = (uint8_t*) &saddr;
  uint8_t* _daddr = (uint8_t*) &daddr;

	printf("ICMP header: \n");
  printf("Type: %d, "
      "Code: %d, ID: %d, Sequence: %d\n", 
      icmph->icmp_type, icmph->icmp_code,
      NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)));

  printf("Sender IP: %u.%u.%u.%u\n",
      *_saddr++, *_saddr++, *_saddr++, *_saddr);
  printf("Target IP: %u.%u.%u.%u\n",
      *_daddr++, *_daddr++, *_daddr++, *_daddr);

  printf("%s--------------------------------------------\n", type);
  for(int i=0; i<64; i+=2) {
    printf("%x ", *(((uint8_t*)icmph) + i));
    printf("%x ", *(((uint8_t*)icmph) + i+1));
    if(i%20==0)
      printf("\n");
  }
  printf("\n--------------------------------------------\n");
}

__device__ void 
DumpICMPPacket(struct icmphdr *icmph, uint32_t saddr, uint32_t daddr)
{
  uint8_t* _saddr = (uint8_t*) &saddr;
  uint8_t* _daddr = (uint8_t*) &daddr;

	printf("ICMP header: \n");
  printf("Type: %d, "
      "Code: %d, ID: %d, Sequence: %d\n", 
      icmph->icmp_type, icmph->icmp_code,
      NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)));

  printf("Sender IP: %u.%u.%u.%u\n",
      *_saddr++, *_saddr++, *_saddr++, *_saddr);
  printf("Target IP: %u.%u.%u.%u\n",
      *_daddr++, *_daddr++, *_daddr++, *_daddr);

  printf("--------------------------------------------\n");
  for(int i=0; i<100; i+=2) {
    printf("%x ", *(((uint8_t*)icmph) + i));
    printf("%x ", *(((uint8_t*)icmph) + i+1));
    if(i%20==0)
      printf("\n");
  }
  printf("\n--------------------------------------------\n");
}

__device__ void 
DumpICMPPacket(struct icmphdr *icmph, uint8_t* saddr, uint8_t* daddr)
{
	printf("\nICMP header: \n");
  printf("Type: %d, "
      "Code: %d, ID: %d, Sequence: %d\n", 
      icmph->icmp_type, icmph->icmp_code,
      NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)));
	printf("ICMP_checksum: 0x%x\n", icmph->icmp_checksum);
  printf("Sender IP: %u.%u.%u.%u\n",
      *saddr++, *saddr++, *saddr++, *saddr);
  printf("Target IP: %u.%u.%u.%u\n",
      *daddr++, *daddr++, *daddr++, *daddr);
}

__device__ void DumpPacket(uint8_t *buf, int len)
//void DumpPacket(uint8_t *buf, int len)
{
  printf("\n\n\n<<<DumpPacket>>>----------------------------------------\n");
	struct ethhdr *ethh;
	struct iphdr *iph;
	struct udphdr *udph;
	//struct tcphdr *tcph;
	uint8_t *t;

	ethh = (struct ethhdr *)buf;
	//if (ntohs(ethh->h_proto) != ETH_P_IP) {
	if (NTOHS(ethh->h_proto) != ETH_P_IP) {
		printf("%02X:%02X:%02X:%02X:%02X:%02X -> %02X:%02X:%02X:%02X:%02X:%02X ",
				ethh->h_source[0],
				ethh->h_source[1],
				ethh->h_source[2],
				ethh->h_source[3],
				ethh->h_source[4],
				ethh->h_source[5],
				ethh->h_dest[0],
				ethh->h_dest[1],
				ethh->h_dest[2],
				ethh->h_dest[3],
				ethh->h_dest[4],
				ethh->h_dest[5]);

		//printf("protocol %04hx  \n", ntohs(ethh->h_proto));
		printf("protocol %04hx  \n", NTOHS(ethh->h_proto));

    //if(ntohs(ethh->h_proto) == ETH_P_ARP)
    if(NTOHS(ethh->h_proto) == ETH_P_ARP)
      DumpARPPacket((struct arphdr *) (ethh + 1));
	//	goto done;
	}

	iph = (struct iphdr *)(ethh + 1);
	udph = (struct udphdr *)((uint32_t *)iph + iph->ihl);
	//tcph = (struct tcphdr *)((uint32_t *)iph + iph->ihl);

	t = (uint8_t *)&iph->saddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->source));
		printf("(%d)", NTOHS(udph->source));

	printf(" -> ");

	t = (uint8_t *)&iph->daddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->dest));
		printf("(%d)", NTOHS(udph->dest));
	else if (iph->protocol == IPPROTO_ICMP){
		struct icmphdr *icmph = (struct icmphdr *) IP_NEXT_PTR(iph);
		DumpICMPPacket(icmph, (uint8_t*)&(iph->saddr), (uint8_t*)&(iph->daddr));
	}

	//printf(" IP_ID=%d", ntohs(iph->id));
	printf(" IP_ID=%d", NTOHS(iph->id));
	printf(" TTL=%d ", iph->ttl);

	switch (iph->protocol) {
	case IPPROTO_TCP:
		printf("TCP ");
		break;
	case IPPROTO_UDP:
		printf("UDP ");
		break;
	default:
		printf("protocol %d ", iph->protocol);
		goto done;
	}
done:
	printf("len=%d\n", len);
  printf("<<<DumpPacket>>>-----------------------------------END--\n");

}

__device__ static int ARPOutput(uint8_t * d_tx_pkt_buffer, int opcode, uint32_t src_ip, uint32_t dst_ip, unsigned char *dst_haddr)
{
	if (!dst_haddr)
		return -1;

  //printf("\n\n\n[%s][%d] Enters\n", __FUNCTION__, __LINE__);
  // ckjung: 00:1b:21:bc:11:52
  //uint8_t src_haddr[ETH_ALEN] = {0x00, 0x1b, 0x21, 0xbc, 0x11, 0x52};
  uint8_t src_haddr[ETH_ALEN] = {0xa0, 0x36, 0x9f, 0x03, 0x13, 0x86};
	struct arphdr *arph = 
    (struct arphdr *)(uintptr_t)EthernetOutput(d_tx_pkt_buffer, ETH_P_ARP, src_haddr, dst_haddr, sizeof(struct arphdr));

	if (!arph) {
    printf("ERROR\n");
		return -1;
	}
	/* Fill arp header */
	arph->ar_hrd = HTONS(arp_hrd_ethernet);
	arph->ar_pro = HTONS(ETH_P_IP);
	arph->ar_hln = ETH_ALEN;
	arph->ar_pln = 4;
	arph->ar_op = HTONS(opcode);

	/* Fill arp body */
#if 0 // HONESTCHOI : TODO
	arph->ar_sip = CONFIG.eths[nif].ip_addr;
#endif 
	arph->ar_sip = src_ip;
	arph->ar_tip = dst_ip;

#if 0 // HONESTCHOI : TODO
	memcpy(arph->ar_sha, CONFIG.eths[nif].haddr, arph->ar_hln);
	if (target_haddr) {
		memcpy(arph->ar_tha, target_haddr, arph->ar_hln);
	} else {
		memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	}
#endif
  for(int i=0; i<arph->ar_hln; i++) {
    arph->ar_sha[i] = src_haddr[i];
    arph->ar_tha[i] = dst_haddr[i];
  }
	//memcpy(arph->ar_sha, src_haddr, arph->ar_hln);
  //memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	memset(arph->pad, 0, ARP_PAD_LEN);

#if 0
	DumpARPPacket(arph);
#endif

	return 0;
}

__device__ static int ProcessARPRequest(struct arphdr *arph, uint8_t* d_tx_pkt_buffer)
{
  //printf("[%s][%d] Enters", __FUNCTION__, __LINE__);
	ARPOutput(d_tx_pkt_buffer, arp_op_reply, arph->ar_tip, arph->ar_sip, arph->ar_sha);
  return 0;
}

// TODO
__device__ static int ProcessARPReply(struct arphdr *arph)
{
  //DumpARPPacket(arph);
	//unsigned char *temp;
	return 0;
}

__device__ int ProcessARPPacket(unsigned char* d_tx_pkt_buffer, unsigned char *pkt_data, int len)
{
	struct arphdr *arph = (struct arphdr *)(pkt_data + sizeof(struct ethhdr));

  switch (NTOHS(arph->ar_op)) {
    case arp_op_request:
      //printf("[%s][%d] arp_op_request\n", __FUNCTION__, __LINE__);
      ProcessARPRequest(arph, d_tx_pkt_buffer);
      break;

    case arp_op_reply:
      //printf("[%s][%d] arp_op_reply\n", __FUNCTION__, __LINE__);
      // TODO
      ProcessARPReply(arph);
      break;

    default:
      printf("[%s][%d] ERROR. KNOWN OP CODE (%d)\n", __FUNCTION__, __LINE__, NTOHS(arph->ar_op));
      //DumpPacket(pkt_data, 1500);
      break;
  }

  return 1;
}

int my_pin_buffer(my_t g, unsigned long addr, size_t size, uint64_t p2p_token, uint32_t va_space, my_mh_t *handle, uint64_t *ret_dma_addr)
{
    int ret = 0;
    int retcode;

    struct MYDRV_IOC_PIN_BUFFER_PARAMS params;
    params.addr = addr;
    params.size = size;
    params.p2p_token = p2p_token;
    params.va_space = va_space;
    params.handle = 0;
    //params.buf_name = bname;

    retcode = ioctl(g->fd, MYDRV_IOC_PIN_BUFFER, &params);
    if (0 != retcode) {
        ret = errno;
        my_err("ioctl error (errno=%d)\n", ret);
    }
    *handle = params.handle;

		// 19.07.17. CKJUNG                                        
		START_YLW                                                  
			printf("[CKJUNG] ret_dma_addr: %p\n", params.ret_dma_addr);
		END                                                        

		*ret_dma_addr = params.ret_dma_addr;

    return ret;
}

int my_pin_desc(my_t g, unsigned long addr, size_t size, uint64_t p2p_token, uint32_t va_space, my_mh_t *handle)
{
    int ret = 0;
    int retcode;

    struct MYDRV_IOC_PIN_DESC_PARAMS params;
    params.addr = addr;
    params.size = size;
    params.p2p_token = p2p_token;
    params.va_space = va_space;
    params.handle = 0;
    //params.buf_name = bname;

    retcode = ioctl(g->fd, MYDRV_IOC_PIN_DESC, &params);
    if (0 != retcode) {
        ret = errno;
        my_err("ioctl error (errno=%d)\n", ret);
    }
    *handle = params.handle;
		return ret;
}

my_t my_open()
{
    my_t m = NULL;
    //const char *myinode = "/dev/mydrv";
    const char *myinode = "/dev/ixgbe";

    m = (my_t) calloc(1, sizeof(*m));
    if (!m) {
        //my_err("error while allocating memory\n");
        return NULL;
    }

    int fd = open(myinode, O_RDWR);
    if (-1 == fd ) {
        int ret = errno;
        //my_err("error opening driver (errno=%d/%s)\n", ret, strerror(ret));
        free(m);
        return NULL;
    }

    m->fd = fd;

    return m;
}

__device__ int clean_index;
__device__ int tx_index;
__device__ int sendable;


__global__ void clean_buffer(unsigned char* buffer, int size, struct pkt_buf *p_buf) 
{
  //for(int i=0; i<size; i++) {
	// CKJUNG 18.03.01
  for(int i=0; i<size; i++) {
    buffer[i] = 0;
  }

	// 19.06.25. Mapping "p_buf" to "d_pkt_buffer"
	p_buf->tx_buf = buffer;
	p_buf->rx_buf = p_buf->tx_buf + offset_for_rx;

#if 1
	for(int i=0; i<512; i++) {
		switch(PKT_SIZE){
			case 64:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_60B, PKT_SIZE);
				break;
			case 128:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_124B, PKT_SIZE);
				break;
			case 256:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_252B, PKT_SIZE);
				break;
			case 512:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_508B, PKT_SIZE);
				break;
			case 1024:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_1020B, PKT_SIZE);
				break;
			case 1514:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_1510B, PKT_SIZE);
				break;
		}
	}
#endif

	// Initialize "Tx" related indices.
	clean_index = 0;
	tx_index = 0;
	sendable = 1;

	printf("%p, %p\n", p_buf->tx_buf, buffer+(512*0x1000));
	
	START_GRN
		printf("[Done]____clean_buffer__\n");
	END

}

__device__ static inline __sum16 csum_fold(unsigned int csum)
{
	u32 sum = (__force u32)csum;;

	sum += (sum << 16);
	csum = (sum < csum);
	sum >>= 16;
	sum += csum;

	return (__force __sum16)~sum;
}

__device__ static inline __sum16 ip_fast_csum(const void *iph, unsigned int ihl)
{
	const unsigned int *word = (const unsigned int*) iph;
	const unsigned int *stop = word + ihl;
	unsigned int csum = 0;
	int carry;

  uint32_t tmp = 0;
  memcpy(&tmp, (uint16_t*)&word[0], 4);
  csum = tmp;
  memcpy(&tmp, (uint16_t*)&word[1], 4);
	csum += tmp;
	carry = (csum < tmp);
	csum += carry;

  memcpy(&tmp, (uint16_t*)&word[2], 4);
	csum += tmp;
	carry = (csum < tmp);
	csum += carry;

  memcpy(&tmp, (uint16_t*)&word[3], 4);
	csum += tmp;
	carry = (csum < tmp);
	csum += carry;

	word += 4;
	do {
    memcpy(&tmp, (uint16_t*)word, 4);
		csum += tmp;
		carry = (csum < tmp);
		csum += carry;
		word++;
	} while (word != stop);

	return csum_fold(csum);
}

__device__ uint8_t *IPOutputStandalone(unsigned char* d_tx_pkt_buffer, uint8_t protocol,uint16_t ip_id, uint32_t saddr, uint32_t daddr, uint16_t payloadlen)
{
	struct iphdr *iph;
	//int nif;
	//unsigned char * haddr;
	//int rc = -1;

// TODO: when daddr is not known yet.
// This should be done with handling arp reply
#if 0
	nif = GetOutputInterface(daddr);
	if (nif < 0)
		return NULL;

	haddr = GetDestinationHWaddr(daddr);
	if (!haddr) {
#if 0
		uint8_t *da = (uint8_t *)&daddr;
		TRACE_INFO("[WARNING] The destination IP %u.%u.%u.%u "
				"is not in ARP table!\n",
				da[0], da[1], da[2], da[3]);
#endif
		RequestARP(mtcp, daddr, nif, mtcp->cur_ts);
		return NULL;
	}
#endif

  //TODO for now, statically sets mac addrs
  //uint8_t src_haddr[ETH_ALEN] = {0x00, 0x1b, 0x21, 0xbc, 0x11, 0x52};
  uint8_t src_haddr[ETH_ALEN] = {0xa0, 0x36, 0x9f, 0x03, 0x13, 0x86};
  // hw addr for yoon
  //uint8_t dst_haddr[ETH_ALEN] = {0x3c, 0xa3, 0x15, 0x04, 0x88, 0xd3};
  //uint8_t dst_haddr[ETH_ALEN] = {0x3c, 0xa3, 0x15, 0x04, 0x86, 0x76};
  uint8_t dst_haddr[ETH_ALEN] = {0x00, 0x1b, 0x21, 0xbc, 0x11, 0x52};
  //uint8_t dst_haddr[ETH_ALEN] = {0xa0, 0x36, 0x9f, 0x9c, 0x93, 0x06};
	iph = (struct iphdr *)EthernetOutput(d_tx_pkt_buffer, ETH_P_IP, src_haddr, dst_haddr, payloadlen + IP_HEADER_LEN);
	if (!iph) {
		return NULL;
	}

	iph->ihl = IP_HEADER_LEN >> 2;
	iph->version = 4;
	iph->tos = 0;
	iph->tot_len = HTONS(IP_HEADER_LEN + payloadlen);

  // to avoid memory misalignment
  // maybe, __be16 is not recognized as 2bytes by cuda
  // maybe, we need to redevine ip header structure using __align__
	*(uint16_t*)(&(iph->id)) = HTONS(ip_id);

  // XXX ??? why undefined?
#define IP_DF   0x4000
	*(uint16_t*)(&(iph->frag_off)) = (uint16_t)HTONS(IP_DF);	// no fragmentation
	*(uint8_t*)&iph->ttl = 64;
	*(uint8_t*)&iph->protocol = protocol;
	//iph->saddr = saddr;
	memcpy((uint16_t*)&iph->saddr,&saddr,4);
	//*(uint32_t*)&iph->saddr = saddr;
	//*(uint32_t*)&iph->daddr = daddr;
	memcpy((uint16_t*)&iph->daddr,&daddr,4);

	// XXX CKJUNG 18.03.15. Shoud understand why problem occurs from seq 512
  iph->check = 0;
  //uint16_t tmp = ip_fast_csum(iph, iph->ihl);
	//printf("CKJUNG___tmp:0x%x\n", ip_fast_csum(iph, iph->ihl));
	//memcpy((uint16_t*)&iph->check, &tmp, 2);
	//printf("CKJUNG___iph->check:0x%x\n", iph->check);
	iph->check = ip_fast_csum(iph, iph->ihl);
	return (uint8_t *)(iph + 1);
}

__device__ static uint16_t ICMPChecksum(uint16_t *icmph, int len)
{
#if 1
	uint16_t ret = 0;
	uint32_t sum = 0;
	uint16_t odd_byte;
	
	while (len > 1) {
		sum += *icmph++;
		len -= 2;
	}
	
	if (len == 1) {
		*(uint8_t*)(&odd_byte) = * (uint8_t*)icmph;
		sum += odd_byte;
	}
	
	sum =  (sum >> 16) + (sum & 0xffff);
	sum += (sum >> 16);
	ret =  ~sum;
	return ret; 
#endif
#if 0
	uint16_t result = 0x12a5;
	return result;
#endif
}

__device__ static int ICMPOutput(unsigned char* d_tx_pkt_buffer, uint32_t saddr, uint32_t daddr,uint8_t icmp_type, uint8_t icmp_code, uint16_t icmp_id, uint16_t icmp_seq, uint8_t *icmpd, uint16_t len)
{
	struct icmphdr *icmph;

	icmph = (struct icmphdr *)IPOutputStandalone(
      d_tx_pkt_buffer, IPPROTO_ICMP, 0, saddr, daddr, sizeof(struct icmphdr) + len);
	if (!icmph)
		return -1;
	/* Fill in the icmp header */
	icmph->icmp_type = icmp_type;
	icmph->icmp_code = icmp_code;
	icmph->icmp_checksum = 0;
	ICMP_ECHO_SET_ID(icmph, HTONS(icmp_id));
	ICMP_ECHO_SET_SEQ(icmph, HTONS(icmp_seq));
	
	/* Fill in the icmp data */
	if (len > 0)
		memcpy((void *)(icmph + 1), icmpd, len);
	
#if 1
	//DumpICMPPacket("ICMPChecksum", icmph, saddr, daddr);
	/* Calculate ICMP Checksum with header and data */
//	icmph->icmp_checksum = 0x12a5;
#if 1
	icmph->icmp_checksum = 
		ICMPChecksum((uint16_t *)icmph, sizeof(struct icmphdr) + len);
#endif
#endif
//	printf("CKJUNG___%s__icmp_checksum:0x%x\n", __FUNCTION__, icmph->icmp_checksum);	
	//DumpICMPPacket("TX", icmph, saddr, daddr);

#if 0
	if (ICMPChecksum((uint16_t *) icmph, 64) ) {
    printf("ICMPChecksum returns ERROR\n");
  }
#endif
	return 0;
}

	__device__ static int ProcessICMPECHORequest(unsigned char* d_tx_pkt_buffer, struct iphdr *iph, int len)
{
	int ret = 0;
	struct icmphdr *icmph = (struct icmphdr *) IP_NEXT_PTR(iph);
	
  // TODO
#if 1 
	if (ICMPChecksum((uint16_t *) icmph, len - (iph->ihl << 2)) ) {
		ret = ERROR;
  }
	else 
#endif
  {
    // RESOLVING MISALINGED ERROR
		// 18.06.14, CKJUNG, Mechanism????
    uint16_t* _saddr = (uint16_t*)&(iph->saddr);
    uint16_t* _daddr = (uint16_t*)&(iph->daddr);
    uint32_t saddr = 0;
    uint32_t daddr = 0;
    memcpy(&saddr, _saddr, 4);
    memcpy(&daddr, _daddr, 4);

    ICMPOutput(d_tx_pkt_buffer, daddr, saddr, ICMP_ECHOREPLY, 0, 
        NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)), 
        (uint8_t *) (icmph + 1),
        (uint16_t) (len - (iph->ihl << 2) - sizeof(struct icmphdr)) );
  }

  return ret;
}


__device__ int ProcessUDPPacket(struct iphdr *iph, int len)
{
//	atomicAdd(&u_cnt, 1);
//	u_cnt++;
//	printf("udp_cnt: %d\n", u_cnt);
	return 0;
}
// ~ckjung

__device__ int ProcessICMPPacket(unsigned char* d_tx_pkt_buffer, struct iphdr *iph, int len)
{
  //uint8_t* _saddr = (uint8_t*) &(iph->saddr);
  //uint8_t* _daddr = (uint8_t*) &(iph->daddr);

	// CKJUNG, #define IP_NEXT_PTR(iph) ((uint8_t *)iph + (iph->ihl << 2))
	// iph->ihl value is 5 in most cases. So, 5 << 2 == 5 * 4 = 20 Bytes (IP header length)
	struct icmphdr *icmph = (struct icmphdr *) IP_NEXT_PTR(iph);
	//int i;
  // TODO : should we do the following?
#if 0
	int to_me = -1;
	
	/* process the icmp messages destined to me */
	for (i = 0; i < CONFIG.eths_num; i++) {
		if (iph->daddr == CONFIG.eths[i].ip_addr) {
			to_me = TRUE;
		}
	}
	
	if (!to_me)
		return TRUE;
#endif
	
  // need to re-align for cuda
#if 0
  uint16_t* _saddr = (uint16_t*)&(iph->saddr);
  uint16_t* _daddr = (uint16_t*)&(iph->daddr);
  uint32_t saddr = 0;
  uint32_t daddr = 0;
  memcpy(&saddr, _saddr, 4);
  memcpy(&daddr, _daddr, 4);
#endif

  switch (icmph->icmp_type) {
        case ICMP_ECHO:
          ProcessICMPECHORequest(d_tx_pkt_buffer, iph, len);
          break;
        case ICMP_DEST_UNREACH:
          printf("[INFO] ICMP Destination Unreachable message received\n");
          break;
        case ICMP_TIME_EXCEEDED:
          printf("[INFO] ICMP Time Exceeded message received\n");
          break;
        default:
          printf("[INFO] Unsupported ICMP message type %x received\n", icmph->icmp_type);
          break;
  }
  return TRUE;
}


__device__ inline int ProcessIPv4Packet(unsigned char* d_tx_pkt_buffer, unsigned char *pkt_data, int len, int* pkt_size)
{
  //printf("[%s][%d]\n",__FUNCTION__, __LINE__);
	/* check and process IPv4 packets */
	struct iphdr* iph = (struct iphdr *)(pkt_data + sizeof(struct ethhdr));
	int ip_len = NTOHS(iph->tot_len);
	*pkt_size = ip_len + 18; // CKJUNG, 18.10.20. + Eth, mac headers(18 bytes)

//	printf("[GPU]_________________ ip_len: %d,  pkt_size: %d\n", ip_len, *pkt_size);

	/* drop the packet shorter than ip header */
	if (ip_len < sizeof(struct iphdr))
    // TODO: define ERROR and FALSE
		//return ERROR;
    return -1;

  // TODO: should handle checksum and promiscuous mode
#if 0
#ifndef DISABLE_HWCSUM
	if (mtcp->iom->dev_ioctl != NULL)
		rc = mtcp->iom->dev_ioctl(mtcp->ctx, ifidx, PKT_RX_IP_CSUM, iph);
	if (rc == -1 && ip_fast_csum(iph, iph->ihl))
		return ERROR;
#else
	UNUSED(rc);
	if (ip_fast_csum(iph, iph->ihl))
		return ERROR;
#endif

#if !PROMISCUOUS_MODE
	/* if not promiscuous mode, drop if the destination is not myself */
	if (iph->daddr != CONFIG.eths[ifidx].ip_addr)
		//DumpIPPacketToFile(stderr, iph, ip_len);
		return TRUE;
#endif
#endif 

	// CKJUNG 18.09.15. For checking ip-pkt length
	//printf("CKJUNG________total-length: %d\n", NTOHS(iph->tot_len));

	
#if 0
	// CKJUNG, 18.08.31 NF#1. IPv4 lookup, DIR-24-8 Algorithm -----------------------------------
	uint16_t* _daddr = (uint16_t*)&(iph->daddr);
	uint32_t daddr = 0;
	memcpy(&daddr, _daddr, 4);
#if 0 // CKJUNG, Printout DST IPs, For check
	uint8_t* t = (uint8_t *)&daddr;
	printf("CKJUNG___________________dst_ip: %u.%u.%u.%u\n", t[0], t[1], t[2], t[3]);
#endif
	//ToDo.
	//IP lookup here!
	//d_interface_lookup(&daddr, d_mtable, d_stable);
#endif
	// ~ CKJUNG----------------------------------------------------------------------------------


	// see if the version is correct
	if (iph->version != 0x4 ) {
    // TODO: define ERROR and FALSE
		//return FALSE;
		return -1;
	}
	
  switch (iph->protocol) {
#if 0
		case IPPROTO_TCP:
			return ProcessTCPPacket(mtcp, cur_ts, ifidx, iph, ip_len);
#endif
// ckjung, 180617 Adding UDP 
//		case IPPROTO_UDP:
//			return ProcessUDPPacket(iph, ip_len);
// ~ ckjung
		case IPPROTO_ICMP:
			ProcessICMPPacket(d_tx_pkt_buffer, iph, ip_len);
			return 1;
		default:
			/* currently drop other protocols */
      // TODO: define FALSE
			return 0;
      //return FALSE
	}
  //return FALSE
}

//__device__ unsigned char* d_tx_buf;
__device__ struct pkt_buf* d_p_buf;
__device__ volatile uint8_t* d_io_addr;
__device__ volatile union ixgbe_adv_tx_desc* d_tx_desc;
__device__ uint64_t* d_gddr_dma_addr;
__device__ int* d_tx_cnt;
__device__ int d_tx_flag[512];
__device__  int prev_t_id;
__device__ int batch_num; //19.08.15. For batch order
__device__ int tx_warp[16];

//#define wrap_ring(index, ring_size) (uint16_t) ((index + 1) & (ring_size - 1))
#define wrap_ring(index, add, ring_size) (uint16_t) ((index + add) & (ring_size - 1))
#define CLEAN_BATCH 32
#define SEND_BATCH 32
//__device__ void send(int pkt_size, int num, int t_id, int batch)
__device__ void send(int pkt_size, volatile uint8_t *io_addr)
{
#if 1
	int num = atomicAdd(d_tx_cnt, 1);
	int t_id = num % 512;
#else
	num = atomicAdd(d_tx_cnt, 1);
	t_id = num % 512;
	//printf("tx_tail_shared: %p, tx_tail_global: %p\n", *tx_tail_shared, *tx_tail_for_queue_zero);
#endif
  //memcpy(&d_p_buf->tx_buf[0x1000 * t_id], tmp_pkt, pkt_size);
	//printf("t_id: %d\n", t_id); 
#if 1
#if 1
	unsigned int paylen = pkt_size; // [Need #2] tx_pkt_size
	int batch = 32;

	(d_tx_desc + t_id)->read.cmd_type_len = IXGBE_TXD_CMD_EOP | IXGBE_TXD_CMD_RS | IXGBE_TXD_CMD_IFCS | IXGBE_TXD_CMD_DEXT | IXGBE_ADVTXD_DTYP_DATA | paylen;
	(d_tx_desc + t_id)->read.olinfo_status = paylen << IXGBE_ADVTXD_PAYLEN_SHIFT;
#if 0
	//if((t_id % 64) == (64 - 1)) {
	if((t_id % batch) == (batch - 1)) { // doorbelling thread
		while(true){
			int32_t cleanable = tx_index - clean_index;
			if(cleanable < 0){
				cleanable = 512 + cleanable;
			}
			if(cleanable < CLEAN_BATCH){
				break;
			}
			int32_t cleanup_to = clean_index + CLEAN_BATCH - 1;
			if(cleanup_to >= 512){
				cleanup_to -= 512;
			}
			if((d_tx_desc + cleanup_to)->wb.status & IXGBE_TXD_STAT_DD){
				int32_t i = clean_index;
				while(true){
					if(i == cleanup_to)
						break;
					i = wrap_ring(i, 512);
				}
				clean_index = wrap_ring(cleanup_to, 512);
			}else{
				break;
			}
		}
	}
#endif
#if 1
	if((t_id % batch) == (batch - 1)) { // doorbelling thread
		while(true){
			// For "writel" every 32-threads(warp) batch.
			if((t_id+1)/batch == readNoCache(&batch_num)){
						//printf("threadIdx.x: %d___(t_id:%d)___batchnum: %d___________________tx: %d, clean: %d\n", threadIdx.x, t_id, (t_id+1)/batch, tx_index, clean_index);
				// If the Index we are trying to writel now is NOT "need-to-clean area" of the ring.

				//[TODO] Make Senderble as cleanable!!!
				
				if(clean_index != t_id){
				//if(sendable){
#if 1
					if(t_id == 511)
						*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(0);
					//*(volatile unsigned long*)((volatile uint8_t*)io_addr + IXGBE_TDT(0)) = (unsigned long)(0);
					else
						*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(t_id + 1);
					//*(volatile unsigned long*)((volatile uint8_t*)io_addr + IXGBE_TDT(0)) = (unsigned long)(t_id + 1);
#endif

					// Set "tx_index" for kernel "clean_topia".
					if(t_id == 511)
						tx_index = 0;
					else
						tx_index = t_id + 1; // +1 because this means "How many" not "Which order".

					batch_num = (batch_num + 1);
					if(batch_num == (512/batch)+1)
						batch_num = 1;
					break;
				}
			}
		}
	}
#else
	if((t_id % batch) == (batch - 1)) {
		printf("threadIdx.x: %d___(t_id:%d)___batchnum: %d___________________clean: %d, tx: %d\n", threadIdx.x, t_id, (t_id+1)/batch, clean_index, tx_index);
		if(clean_index != tx_index + 1){
			if(t_id == 511)
				*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(0);
			else
				*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(t_id + 1);

			if(t_id == 511)
				tx_index = 0;
			else
				tx_index = t_id + 1;
		}
	}

#endif
#else
	//[TODO] 19.08.15. We don't have to check all desc's STAT_DD flag!
	// We only need to concern about the LAST one!
	//d_tx_flag[t_id] = 1;
	unsigned int paylen = pkt_size; // [Need #2] tx_pkt_size
	//[TODO] IF values below are set, HW never set this desc to STAT_DD!, 19.08.15.

	int batch = 32;
	//if((t_id % batch) != (batch - 1)) {
	d_tx_flag[t_id] = 1;
	//atomicAdd(&d_tx_flag[ ((t_id / batch) + 1) * batch - 1 ], 1);
	//[TODO] 2-codelines below should be moved down.
	// Is is better to access adjacent PCIe address from 1-thread instead of multiple threads?, 19.08.15.
	//(desc + t_id)->read.cmd_type_len = IXGBE_TXD_CMD_EOP | IXGBE_TXD_CMD_RS | IXGBE_TXD_CMD_IFCS | IXGBE_TXD_CMD_DEXT | IXGBE_ADVTXD_DTYP_DATA | paylen;
	//(desc + t_id)->read.olinfo_status = paylen << IXGBE_ADVTXD_PAYLEN_SHIFT;
	//}

	if((t_id % batch) == (batch - 1)) { // doorbelling thread
		bool wait = true;
		while(wait){
			int sum = 0;
			for(int i = t_id - (batch - 1); i < t_id; i++) {
				sum += d_tx_flag[i];
			}
			if(sum == (batch-1)) {
				for(int i = t_id - (batch - 1); i < t_id; i++) {
					d_tx_flag[i] = 0;
					//[TODO] cmd_type_len is critical!!! 13.4 Mpps -> 8 Mpps !!
					//(d_tx_desc + i)->read.cmd_type_len |= paylen;
					(d_tx_desc + i)->read.cmd_type_len = IXGBE_TXD_CMD_EOP | IXGBE_TXD_CMD_RS | IXGBE_TXD_CMD_IFCS | IXGBE_TXD_CMD_DEXT | IXGBE_ADVTXD_DTYP_DATA | paylen;
					(d_tx_desc + i)->read.olinfo_status = paylen << IXGBE_ADVTXD_PAYLEN_SHIFT;
				}
				//printf("threadIdx.x: %d___(t_id:%d)___batchnum: %d\n", threadIdx.x, t_id, (t_id+1)/batch);
				while(true){
					if((t_id+1)/batch == readNoCache(&batch_num)){
						uint32_t status = (d_tx_desc + t_id)->wb.status;
						if(status & IXGBE_TXD_STAT_DD) {
							if(clean_index == tx_index + 1)
								break;
							//printf("_____________________________________________threadIdx.x: %d___(t_id:%d)___writel(%d)\n", threadIdx.x, t_id, (t_id+1)/batch);
							d_tx_flag[t_id] = 0;
							//(d_tx_desc + t_id)->read.cmd_type_len |= paylen;
							(d_tx_desc + t_id)->read.cmd_type_len = IXGBE_TXD_CMD_EOP | IXGBE_TXD_CMD_RS | IXGBE_TXD_CMD_IFCS | IXGBE_TXD_CMD_DEXT | IXGBE_ADVTXD_DTYP_DATA | paylen;
							(d_tx_desc + t_id)->read.olinfo_status = paylen << IXGBE_ADVTXD_PAYLEN_SHIFT;
							if(t_id == 511)
								//*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(0);
								*(volatile unsigned long*)((volatile uint8_t*)io_addr + IXGBE_TDT(0)) = (unsigned long)(0);
							else
								//*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(t_id + 1);
								*(volatile unsigned long*)((volatile uint8_t*)io_addr + IXGBE_TDT(0)) = (unsigned long)(t_id + 1);
							wait = false;

							tx_index = t_id + 1;

							batch_num = (batch_num + 1);
							if(batch_num == (512/batch)+1)
								batch_num = 1;
							break;
						}
						//printf("Can't triggering writel______________t_id: %d\n", t_id);
					}
				}
				//printf("return______________threadIdx.x: %d_____________batchnum: %d (t_id:%d)\n", threadIdx.x, (t_id+1)/batch, t_id);
			}else{
				//printf("[FAIL] sum : %d\n", sum);
			}
		}
	}
#endif
#endif
}

__device__ void swap_src_dst(uint8_t *buf, struct pkt_buf *p_buf, int idx)
{
	int i;
	struct ethhdr *ethh;
	struct iphdr *iph;
	ethh = (struct ethhdr *)buf;
	iph = (struct iphdr *)(buf + sizeof(struct ethhdr));

	uint8_t tmp[ETH_ALEN] = {0,};
	uint32_t tmp_ip = 0;
	uint16_t* _saddr = (uint16_t*)&(iph->saddr);
	uint16_t* _daddr = (uint16_t*)&(iph->daddr);
	uint32_t saddr = 0;
	uint32_t daddr = 0;
	memcpy(&saddr, _saddr, 4);
	memcpy(&daddr, _daddr, 4);
	tmp_ip = saddr;
	saddr = daddr;
	daddr = tmp_ip;
	
	
#if 1
	for(i = 0; i < ETH_ALEN; i++) {
		ethh->h_source[i] = p_buf->mac_dst[idx][i];
		ethh->h_dest[i] = p_buf->mac_src[idx][i];
	
//		tmp[i] = ethh->h_source[i];
//		ethh->h_source[i] = ethh->h_dest[i];
//		ethh->h_dest[i] = tmp[i];
	}

//	memcpy((uint16_t*)&iph->saddr, &saddr, 4);
//	memcpy((uint16_t*)&iph->daddr, &daddr, 4);
	
#endif
#if 0
	printf("Source MAC:\n");
	for(i = 0; i < ETH_ALEN; i++) {
		printf("%02x ", p_buf->mac_src[idx][i]);
	}
	printf("\n");
	
	printf("Dest MAC:\n");
	for(i = 0; i < ETH_ALEN; i++) {
		printf("%02x ", p_buf->mac_dst[idx][i]);
	}
	printf("\n");
#endif
}


__global__ void writeler(void)
{
	int i;
	int idx = 0;
	for(i = 0; i < 16; i++)
		tx_warp[i] = 0;
	while(true)
	{
		//for(i = 0; i < 16; i++)
		//	printf("%d ", tx_warp[i]);
		//printf("\n");
		if(tx_warp[idx] == 1){
			printf("idx: %d\n", idx);
			if(clean_index == tx_index + 1)
				continue;
			int t_id = (idx + 1)*32 - 1;

			if(t_id == 511)
				*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(0);
			else
				*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(t_id + 1);

			if(t_id == 511)
				tx_index = 0;
			else
				tx_index = t_id + 1;
			tx_warp[idx] = 0;
			idx++;
		}
	}
}

#define LOG 1
//__global__ void tx_handler(volatile union ixgbe_adv_tx_desc* tx_desc, unsigned char* gtx_desc, volatile uint8_t* io_addr, int* pkt_cnt)
__global__ void tx_handler(union ixgbe_adv_tx_desc* gtx_desc, volatile uint8_t* io_addr, int* pkt_cnt, uint64_t *gddr_dma_addr)
{
	__shared__ int clean[512];
	__shared__ int tx[512];
	__shared__ int desc[512];
	__shared__ int clean_index; // sum of clean[512] 
	__shared__ int tx_index; // sum of tx[512]
	__shared__ int cleanable;
	__shared__ int cleanup_to;
	__shared__ int do_clean;

	__shared__ clock_t t1, t2;
	__shared__ int yhoon_num;

	int tx_hang = 0; // For each threads. 19.08.31.

// Initialize variables
	if(threadIdx.x == 0){
		int i;
		for(i = 0; i < 512; i++){
			clean[i] = 1; // Every desc is "Clean".
			tx[i] = 0; // Every desc is usable.
			desc[i] = 0;
		}
		clean_index = 0;
		tx_index = 0;
		cleanable = 0;
		cleanup_to = 0;
		do_clean = 0;
		t1 = 0;
		t2 = 0;
		for(i = 0; i < 512; i++){
			(gtx_desc + i)->read.buffer_addr = *gddr_dma_addr + 0x1000*i;
			(gtx_desc + i)->read.cmd_type_len = IXGBE_TXD_CMD_EOP | IXGBE_TXD_CMD_RS | IXGBE_TXD_CMD_IFCS | IXGBE_TXD_CMD_DEXT | IXGBE_ADVTXD_DTYP_DATA;
		}
	}
	// Persistent Loop
	int count = 0;
	while(true){
#if 0
		if(threadIdx.x == 0)
			printf("__________________________________________________start of Persist.loop\n");
#endif
/////////////////////////////////// desc-clean-routine //////////////////////////////////////
			// tx_index, clean_index, cleanable, cleanup_to
#if 0
		if(threadIdx.x == 0){
		//	if(count++ % 10000 == 0) {
				int i;
		
				for(i = 0; i < 512; i++){
					if(i % 32 == 0)
						printf("\n");
					printf("%d ", desc[i]);
				}
				
				printf("\n\n");
		
				//printf("tx_index: %d, clean_index: %d\n", tx_index, clean_index);
				/*
				for(i = 0; i < 512; i++){
					if(i % 32 == 0)
						printf("\n");
					printf("%d ", (tx_desc + i)->wb.status & IXGBE_TXD_STAT_DD);
				}
				printf("\n\n");
			*/
		//	}
		}
			__syncthreads();
#endif
			__syncthreads(); //[TODO] If we have path divergence below, this line is necessary!
#if 0
			if(threadIdx.x % CLEAN_BATCH == (CLEAN_BATCH-1) && desc[threadIdx.x] == 1) {  // --> Need to clean (used for Tx).
				//printf("threadId: %d\n", (threadIdx.x + 1)/32);
				//if(desc[threadIdx.x] == 1) {  // --> Need to clean (used for Tx).
				//if(readNoCache(& ((tx_desc + threadIdx.x)->wb.status)) & IXGBE_TXD_STAT_DD) {
				if(((gtx_desc + threadIdx.x)->wb.status) & IXGBE_TXD_STAT_DD) {
					for(int i=0; i<CLEAN_BATCH; i++)
						desc[threadIdx.x-i] = 0;
				}
			}
#endif
#if 1
			//if(threadIdx.x == (clean_index + CLEAN_BATCH - 1)){
			if(threadIdx.x == wrap_ring(clean_index, CLEAN_BATCH - 1, 512)){      
				if(((gtx_desc + threadIdx.x)->wb.status) & IXGBE_TXD_STAT_DD) {
					for(int i=0; i<CLEAN_BATCH; i++)
						desc[threadIdx.x-i] = 0;
				clean_index = wrap_ring(threadIdx.x, 1, 512);
				//atomicAdd(&pkt_cnt[1], SEND_BATCH);
				}
			}
#else
			if(threadIdx.x == wrap_ring(clean_index, CLEAN_BATCH - 1, 512)){      
				int cnt = 0;                                                        
				while(true){                                                        
					if(cnt > 3){                                                      
						for(int i=0; i<CLEAN_BATCH; i++)                              
							desc[threadIdx.x-i] = 0;                                    
						clean_index = wrap_ring(threadIdx.x, 1, 512);                 
						break;                                                        
					}                                                                 
					int cleanable = tx_index - clean_index;                           
					if(cleanable < 0)                                                 
						cleanable = 512 + cleanable;                                    
					if(cleanable < CLEAN_BATCH) break;                                
					if(((gtx_desc + threadIdx.x)->wb.status) & IXGBE_TXD_STAT_DD) {   
						for(int i=0; i<CLEAN_BATCH; i++)                                
							desc[threadIdx.x-i] = 0;                                      
						clean_index = wrap_ring(threadIdx.x, 1, 512);                   
					}                                                                 
					cnt++;                                                            
				}                                                                   
			}                                                                     
#endif
			__syncthreads();
			//	if(desc[threadIdx.x] == 0) { // --> Able to tx (cleaned).
			if(threadIdx.x >= tx_index && threadIdx.x < tx_index + SEND_BATCH) {
				if(desc[threadIdx.x] == 0){
					unsigned int paylen = PKT_SIZE;
					(gtx_desc + threadIdx.x)->read.cmd_type_len = IXGBE_TXD_CMD_EOP | IXGBE_TXD_CMD_RS | IXGBE_TXD_CMD_IFCS | IXGBE_TXD_CMD_DEXT | IXGBE_ADVTXD_DTYP_DATA | paylen;
					(gtx_desc + threadIdx.x)->read.olinfo_status = paylen << IXGBE_ADVTXD_PAYLEN_SHIFT;
					desc[threadIdx.x] = 1;
					if(threadIdx.x == tx_index + SEND_BATCH - 1){
						//tx_index = wrap_ring(threadIdx.x, 512);
						tx_index = wrap_ring(threadIdx.x, 1, 512);
						*(volatile unsigned long*)((volatile uint8_t*)io_addr + IXGBE_TDT(0)) = (unsigned long)(tx_index);
						//printf("writel, tx_index: %d\n", tx_index);
						atomicAdd(&pkt_cnt[1], SEND_BATCH);
					}
				}
			}
			__syncthreads();
	} // while(true)
}

#define C_SIZE 64
__global__ void tx_test(int* pkt_cnt, volatile uint8_t* io_addr)
{
	__shared__ int num_turns;
	//__shared__ clock_t start;

	__shared__ int num;
	__shared__ int t_id;
	__shared__ int batch;

	if(threadIdx.x == 0){
		num = 0;
		t_id = 0;
		batch = 32;
	}


	clock_t start; // For each thread
	start = clock64();

	while(num_turns < NUM_TURN_rx_handler){
#if 0
		if(threadIdx.x == 0)
		{
			while(1){
				//if(3000.0 <= (float)(((clock64() - start)/1480000000.0)*1000.0)){
				if(0.02 <= (float)(((clock64() - start)/1480000000.0)*1000.0)){
					start = clock64();
					break;
				}
			}
		}
		__syncthreads();
#endif
#if 0
		while(1){
			if(0.03 <= (float)(((clock64() - start)/1480000000.0)*1000.0)){
				start = clock64();
				break;
			}
		}
#endif
		//send(64, num, t_id, batch);
		send(64, io_addr);


	}
}


__global__ void var_map(struct pkt_buf *p_buf, volatile uint8_t* io_addr, volatile union ixgbe_adv_tx_desc* tx_desc, uint64_t *gddr_dma_addr, int* pkt_cnt)
{
	//d_tx_buf = p_buf->tx_buf;
	d_p_buf = p_buf;
	d_io_addr = io_addr;
	d_tx_desc = tx_desc;
	d_gddr_dma_addr = gddr_dma_addr;
	d_tx_cnt = &pkt_cnt[1];
	for(int i=0; i<512; i++) {
		d_tx_flag[i] = 0;
		(d_tx_desc+i)->wb.status |= IXGBE_TXD_STAT_DD;
	}
	batch_num = 1;
	tx_tail_for_queue_zero = (volatile uint8_t*)io_addr + IXGBE_TDT(0);
	//rx_tail_for_queue_zero = (volatile uint8_t*)io_addr + IXGBE_RDT(0);
}


__global__ void old_tx_handler(struct pkt_buf *p_buf, int* pkt_cnt, volatile uint8_t* io_addr, volatile union ixgbe_adv_tx_desc* tx_desc, uint64_t *gddr_dma_addr)
{
	__shared__ int num_turns;
	__shared__ int begin;
	__shared__ int batch_map[16];
	__shared__ int warp_idx;

	int idx = 0;
	if(threadIdx.x == 0){
		int i;
		for(i = 0; i < 16; i++)
			batch_map[i] = 0;
		warp_idx = 0;
	}

	tx_tail_for_queue_zero = io_addr + IXGBE_TDT(0);
	while(num_turns < NUM_TURN_rx_handler) {
#if 0
		if(threadIdx.x == 0){
			if(batch_map[idx] == 1){
				int t_id = ((idx+1)*32) - 1;
									
				if(t_id == 511)
					*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(0);
				else
					*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(t_id + 1);
			
				batch_map[idx] = 0;
				idx += 1;
				if(idx == 17)
					idx = 1;
			}
		}
#endif
		if(readNoCache(&(p_buf->tx_buf_idx[threadIdx.x])) == 1){ // [Need #1] tx_buf_idx
			p_buf->tx_buf_idx[threadIdx.x] = 0;
			//printf("[tx_handler] got pkt. tid: %d\n", t_id);

			//	p_buf->rx_no_poll[t_id] = 1; // pause rx_handler for this buf.
#if 1
#if 1
	//		memcpy(&p_buf->tx_buf[0x1000 * threadIdx.x], tmp_pkt, p_buf->tx_pkt_size[threadIdx.x]);
#else
			memcpy(&p_buf->tx_buf[0x1000 * threadIdx.x], &p_buf->rx_buf[0x1000 * threadIdx.x], p_buf->tx_pkt_size[threadIdx.x]);


			swap_src_dst((uint8_t*)(&p_buf->tx_buf[0x1000 * threadIdx.x]), p_buf, threadIdx.x);
			struct iphdr *iph;
			iph = (struct iphdr *)(&(p_buf->rx_buf[0x1000 * threadIdx.x]) + sizeof(struct ethhdr));
			iph->check = ip_fast_csum(iph, iph->ihl);
#endif

#if 0
			START_GRN
0		printf("_________________________________________[TX_handler]\n");
			END
			DumpPacket_raw(&p_buf->tx_buf[0x1000 * t_id], p_buf->tx_pkt_size[t_id]);
#endif
			//volatile union ixgbe_adv_tx_desc *desc = tx_desc + t_id;

#if 1
			unsigned int paylen = p_buf->tx_pkt_size[threadIdx.x]; // [Need #2] tx_pkt_size
			//(tx_desc + threadIdx.x)->read.cmd_type_len = IXGBE_TXD_CMD_EOP | IXGBE_TXD_CMD_RS | IXGBE_TXD_CMD_IFCS | IXGBE_TXD_CMD_DEXT | IXGBE_ADVTXD_DTYP_DATA | paylen;
			(tx_desc + threadIdx.x)->read.cmd_type_len |= paylen;
			(tx_desc + threadIdx.x)->read.olinfo_status = paylen << IXGBE_ADVTXD_PAYLEN_SHIFT;
#endif
#if 0
			if(threadIdx.x % 32 == 31){
				batch_map[((threadIdx.x+1)/32)-1] = 1;
				//printf("tid: %d\n", ((threadIdx.x+1)/32)-1);
			}
#endif
			int num = atomicAdd(&pkt_cnt[1], 1);
			int t_id = num % 512;

			//printf("t_id: %d\n", t_id);
			
#if 1
			if(threadIdx.x % 32 == 31){

					//printf("t_id: %d\n", (threadIdx.x+1)/32);
					//[TODO] Order is trash itself! WTF.
					int n = atomicAdd(&warp_idx, 1);
					int tt = n % 16;
				
					printf("tt: %d_________warp_idx: %d\n", tt, ((tt+1)*32-1));
						if(((tt+1)*32-1) == 511)
							*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(0);
						else
							*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)((tt+1)*32);
			}
#endif
			// [TODO] clean buffer (re-map)
				//for(i = 0; i < C_SIZE; i++){
				//	(desc + threadIdx.x)->read.buffer_addr = *gddr_dma_addr + threadIdx.x*0x1000; 
				//}
			//}
			//*(volatile unsigned long*)tx_tail_for_queue_zero = (unsigned long)(t_id);
			//*(uint16_t*)(&p_buf->rx_buf[0x1000*(t_id)]) = 0;

			// After Tx,
			//p_buf->tx_pkt_size[t_id] = 0;

#if 0
			//19.07.11, CKJUNG, If we do here, rx pps goes CRAZY!!!
			p_buf->rx_buf[0x1000 * t_id] = 0; // Prohibiting read.
			p_buf->rx_buf[(0x1000 * t_id)+1] = 0; // Prohibiting read. 
#endif
#if 0
			// Tx needs more time? 19.07.11 (writel?)
			if(t_id == 0){
				p_buf->tx_buf[0x1000 * 511] = 0; // Prohibiting read.
				p_buf->tx_buf[(0x1000 * 511)+1] = 0; // Prohibiting read. 
			}else{
				p_buf->tx_buf[0x1000 * (t_id-1)] = 0; // Prohibiting read.
				p_buf->tx_buf[(0x1000 * (t_id-1))+1] = 0; // Prohibiting read. 
			}
			__threadfence();
#endif
		//	p_buf->rx_no_poll[t_id] = 0; // resume rx_handler for this buf.

#endif
		}
	}
}

__global__ void rx_handler(struct pkt_buf *p_buf, int* pkt_cnt, int* pkt_size, volatile uint8_t* io_addr, volatile union ixgbe_adv_rx_desc* rx_desc, uint64_t *gddr_dma_addr)
{
	__shared__ int num_turns;
	__shared__ int max;
	__shared__ int flag[512];
	__shared__ volatile uint8_t *rx_tail_for_queue_zero_s;
	int i;
	// CKJUNG, 18.10.19,  Connect "d_pkt_buffer" with "p_buf" here in rx_handler. 
	// We'll just use p_buf from NFs.
	//p_buf->tx_buf = d_pkt_buffer;
	//p_buf->rx_buf = p_buf->tx_buf + offset_for_rx;

	BEGIN_SINGLE_THREAD_PART{
		for(i = 0; i < 512; i++)
			flag[i] = 0;
		num_turns = 0;
		max = 0;
	} END_SINGLE_THREAD_PART;
	//if(threadIdx.x == 0)
		//printf("[GPU]rx_handler, gddr_dma_addr: %p\n", *gddr_dma_addr);
	int mem_index = 0x1000 * threadIdx.x;
	uint64_t rx_dma_addr = *gddr_dma_addr + 0x1000*512 + threadIdx.x*0x1000;
	rx_tail_for_queue_zero_s = io_addr + IXGBE_RDT(0);
	while(num_turns < NUM_TURN_rx_handler) { // Persistent kernel
			//printf("[Rx_Hdlr] Infinite_loop num_turns: %d\n", num_turns);
			BEGIN_SINGLE_THREAD_PART{
#if 0
				if(*(&pkt_cnt[0]) != 0 && *(&pkt_cnt[0]) % 512 == 0){ // Reset index to "0", when exceeds 512. 
					for(i = 0; i < 512; i++)
						p_buf->rx_buf_idx[i] = 0;
				}
#endif		
#if 0
				if(*(&pkt_cnt[0]) != 0 && *(&pkt_cnt[0]) % BATCH == 0){ // Set index to "2", batch 32.
					for(i = 0; i < 512; i++){
						if(p_buf->rx_buf_idx[i] == 1)
							p_buf->rx_buf_idx[i] = 2;
					}
				}
#endif
			} END_SINGLE_THREAD_PART;

				//printf("[rx_handler] got pkt. tid: %d, %d\n", threadIdx.x, (uint8_t)p_buf->rx_buf[mem_index]);

#if 0
			BEGIN_SINGLE_THREAD_PART{
				if(pkt_cnt[0] > 0 && pkt_cnt[0] % 500 == 0)
					*(volatile unsigned long*)rx_tail_for_queue_zero = (unsigned long)(last_idx);
			} END_SINGLE_THREAD_PART;
#endif
#if 0
#define POLLING_THS 128
			if(threadIdx.x % (512/POLLING_THS) == (512/POLLING_THS)-1)
			{
				for(i = threadIdx.x - ((512/POLLING_THS)-1); i <= threadIdx.x; i++)
				{
					if((readNoCache((uint16_t*)&p_buf->rx_buf[0x1000*i]) != 0))
					{
						flag[i] = 1;
					}
				}

			}
			if((readNoCache(&flag[threadIdx.x]) != 0)){
#else
			if(readNoCache((uint16_t*)&p_buf->rx_buf[mem_index]) != 0){
#endif

				//p_buf->rx_no_poll[threadIdx.x] = 1; // pause rx_handler for this buf.
				//if(readNoCache((uint16_t*)&p_buf->rx_buf[mem_index]) != 0 && p_buf->tx_buf_idx[threadIdx.x] == 0 ){
				//printf("[rx_handler] got pkt. tid: %d\n", threadIdx.x);

#if 0
				START_BLU
				printf("_________________________________________[RX_handler]\n");
				END
				DumpPacket_raw(&p_buf->rx_buf[0x1000 * threadIdx.x], 64);
#endif
#if 0
				struct ethhdr *ethh;
				ethh = (struct ethhdr *)&p_buf->rx_buf[mem_index];
				int i;
				for(i = 0; i < ETH_ALEN; i++)
				{
					p_buf->mac_src[threadIdx.x][i] = ethh->h_source[i];
					p_buf->mac_dst[threadIdx.x][i] = ethh->h_dest[i];
				}
#endif

				atomicAdd(&pkt_cnt[0], 1);

				// 19.04.11, CKJUNG

				//atomicAdd(num_turns, 1);
#if 0
				atomicAdd(&cnt, 1);
				acum_cnt[threadIdx.x] = cnt;
				rx_ed = clock64();
				//printf("\033[1;32m[Rx_Hdlr] Buf_id[%d]: %lf\033[0m\n", threadIdx.x,(double)(((double)rx_ed - (double)rx_st)/1480000000.0)*1000.0);
				printf("%3d %lf %3d\n", threadIdx.x,(double)(((double)rx_ed - (double)rx_st)/1480000000.0)*1000.0, acum_cnt[threadIdx.x]);
#endif

				//[TODO] buf_idx to 1 at each threads (parallelism)??
#if 0
				if(pkt_cnt[0] % BATCH == 0 && pkt_cnt[0] != 0){
					for(i = threadIdx.x; i > threadIdx.x - BATCH; i--)
						p_buf->rx_buf_idx[i] = 1; // [Step 1] If we receive something.
				}
#else					
				__syncthreads();
				p_buf->rx_buf_idx[threadIdx.x] = 1; // [Step 1] If we receive something.
#endif	
				p_buf->rx_buf[mem_index] = 0; // Prohibiting read.
				p_buf->rx_buf[mem_index+1] = 0; // Prohibiting read. 
				//flag[threadIdx.x] = 0;

				__syncthreads();
				volatile union ixgbe_adv_rx_desc *desc = rx_desc + threadIdx.x;
				desc->read.pkt_addr = rx_dma_addr;
				desc->wb.upper.length = 0;

#if 1
				// "rx_desc" is stored in Global memory. 19.08.07. CKJUNG 
				//__threadfence();
				//int batch_size = 32*8;
				//if(threadIdx.x % batch_size  == (batch_size - 1)){
				
				__syncthreads();
				if(threadIdx.x % (32*8)  == ((32*8) - 1)){
					*(volatile unsigned long*)rx_tail_for_queue_zero_s = (unsigned long)(threadIdx.x);
				}
				//__syncthreads();
#else
				if(pkt_cnt[0] > 0 && pkt_cnt[0] % 32 == 0){
					printf("tid: %d, pkt_cnt: %d\n", threadIdx.x, pkt_cnt[0]);
					*(volatile unsigned long*)rx_tail_for_queue_zero = (unsigned long)(threadIdx.x);
				}

#endif

				//p_buf->rx_done[threadIdx.x] = 1;

				unsigned char* rx_packet = &p_buf->rx_buf[0x1000*threadIdx.x];
				unsigned char* tx_packet = &p_buf->tx_buf[0x1000*threadIdx.x];
				if(*(uint16_t*)(rx_packet+12) != 0) {
					struct ethhdr *ethh = (struct ethhdr *)rx_packet;
					u_short ip_proto = NTOHS(ethh->h_proto);
					
					struct iphdr *iph = (struct iphdr *)(rx_packet + sizeof(struct ethhdr));
					int ip_len = NTOHS(iph->tot_len);
					*pkt_size = ip_len + 18; // + mac header (18 Bytes)

					if (ip_proto == ETH_P_ARP) {
						ProcessARPPacket(tx_packet, rx_packet, 60); // [TODO] Need to off NFs..
						p_buf->tx_buf_idx[threadIdx.x] = 1; // Send!
						p_buf->tx_pkt_size[threadIdx.x] = 60;
						atomicAdd(&pkt_cnt[1], 1);
					}else if(ip_proto == ETH_P_IP) {
						// TODO: passing len from below
#if 0
						if(ProcessIPv4Packet(tx_packet, rx_packet, 1500, pkt_size)){
							//printf("ICMP] tid: %d\n", threadIdx.x);
							p_buf->tx_buf_idx[threadIdx.x] = 1; // Send!
							p_buf->tx_pkt_size[threadIdx.x] = *pkt_size;
							//printf("pkt_size: %d\n", *pkt_size);
							atomicAdd(&pkt_cnt[1], 1);
						}
#endif
						;
					}else {
						printf("[%s][%d] %d thread unknown protocol\n", __FUNCTION__, __LINE__, threadIdx.x);
					}
					//*(uint16_t*)(rx_packet+12) = 0;
				}
			}
	} // ~ while
	if(threadIdx.x == 0)
		printf("End of rx_handler!\n");
}

// YHOON~ for test
int tx_rx_ring_setup()
{
  const char *myinode = "/dev/ixgbe";
  int fd = open(myinode, O_RDWR);
  ioctl(fd, 1);
  return fd;
}

void yhoon_finalizer(void* ixgbe_bar0_host_addr, void* desc_addr)
{
  const size_t IXGBE_BAR0_SIZE = 4096*8; // A rough calculation
	printf("Finalizer called!\n");
  hipHostUnregister(desc_addr);
  hipHostUnregister(ixgbe_bar0_host_addr);
  munmap(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE*5);
}



void yhoon_initializer(int fd, void *ixgbe_bar0_host_addr, void *tx_desc_addr, void *rx_desc_addr, void **io_addr, void **tx_desc, void **rx_desc)
{
	const size_t IXGBE_BAR0_SIZE = 4096*8; // A rough calculation

	ixgbe_bar0_host_addr = mmap(0, 4096*12, PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
	//ixgbe_bar0_host_addr = mmap(0, IXGBE_BAR0_SIZE*5 , PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
	if(ixgbe_bar0_host_addr == MAP_FAILED) {
		  START_RED
				  printf("mmap Error!\n");
			  END
					  exit(1);
	}

	printf("[mmap] ixgbe_bar0_host_addr: %p\n", ixgbe_bar0_host_addr);
	hipError_t desc_addr_error = hipHostRegister(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE, hipHostRegisterIoMemory);
	ASSERTRT(desc_addr_error);
	if(desc_addr_error != hipSuccess) {
		  START_RED
				    fprintf(stdout, "%s\n", hipGetErrorName(desc_addr_error));
			  END
					 // yhoon_finalizer(ixgbe_bar0_host_addr, desc_addr);
				  exit(1);
	}
	ASSERTRT(hipHostGetDevicePointer((void**)io_addr, (void*)ixgbe_bar0_host_addr, 0));
	printf("[GetDevicePtr] io_addr: %p\n", *io_addr);

	tx_desc_addr = (void*)((unsigned char*)ixgbe_bar0_host_addr + IXGBE_BAR0_SIZE);
	printf("[offset] tx_desc_addr: %p\n", tx_desc_addr);
	desc_addr_error = hipHostRegister(tx_desc_addr, 0x1000*2, hipHostRegisterIoMemory);
	if(desc_addr_error != hipSuccess) {
		  START_RED
				    printf("%s\n", hipGetErrorName(desc_addr_error));
			  END
					//  yhoon_finalizer(ixgbe_bar0_host_addr, desc_addr);
				  exit(1);
	}
	ASSERTRT(hipHostGetDevicePointer((void**)tx_desc, (void*)tx_desc_addr, 0));
	if(*tx_desc != NULL){
		  START_GRN
				  printf("tx_desc ready :)\n");
			END
	}

	rx_desc_addr = (void*)((unsigned char*)ixgbe_bar0_host_addr + IXGBE_BAR0_SIZE + 4096*2);
	printf("[offset] rx_desc_addr: %p\n", rx_desc_addr);                                    
	desc_addr_error = hipHostRegister(rx_desc_addr, 0x1000*2, hipHostRegisterIoMemory);   
	if(desc_addr_error != hipSuccess) {                                                    
		START_RED                                                                             
			printf("%s\n", hipGetErrorName(desc_addr_error));                                  
		END                                                                                   
			//  yhoon_finalizer(ixgbe_bar0_host_addr, desc_addr);                               
			exit(1);                                                                            
	}                                                                                       
	ASSERTRT(hipHostGetDevicePointer((void**)rx_desc, (void*)rx_desc_addr, 0));            
	if(*rx_desc != NULL){                                                                   
		START_GRN                                                                             
			printf("rx_desc ready :)\n");                                                       
		END                                                                                   
	}                                                                                       

	printf("[GetDevicePtr] tx_desc: %p\n", *tx_desc);
	printf("[GetDevicePtr] rx_desc: %p\n", *rx_desc);
}

extern "C"
void initialize_gdnio(void)
{
	printf("____[Initialize]__GDNIO__\n");
	int dev_id = 0;
	size_t _pkt_buffer_size = 2*512*4096; // 4MB, for rx,tx ring
	size_t pkt_buffer_size = (_pkt_buffer_size + GPU_PAGE_SIZE - 1) & GPU_PAGE_MASK;
	int n_devices = 0;
	ASSERTRT(hipGetDeviceCount(&n_devices));

	hipDeviceProp_t prop;
	for (int n=0; n<n_devices; ++n) {
		hipGetDeviceProperties(&prop,n);
		OUT << "GPU id:" << n << " name:" << prop.name 
			<< " PCI domain: " << prop.pciDomainID 
			<< " bus: " << prop.pciBusID 
			<< " device: " << prop.pciDeviceID << endl;
	}
	OUT << "selecting device " << dev_id << endl;
	OUT << "_pkt_buffer_size: " << _pkt_buffer_size << "  pkt_buffer_size: " << pkt_buffer_size << endl;


	int peak_clk = 1; // in kHz
	//CKJUNG 18.03.17
	ASSERTRT(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	OUT << "GPU___Peak_clockrate:" << peak_clk << " kHz" << endl;
	// ~CKJUNG

	ASSERTRT(hipSetDevice(dev_id));

	// 18.10.25, CKJUNG, We Reset the device to fresh each run.
	ASSERTRT(hipDeviceReset());
  ASSERTRT(hipSetDeviceFlags(hipDeviceMapHost));


//	unsigned char* d_pkt_buffer;
  ASSERTRT(hipMalloc((void**)&d_pkt_buffer, pkt_buffer_size));
  ASSERTRT(hipMemset(d_pkt_buffer, 0, pkt_buffer_size));

// Allocate "Tx-desc" in GDDR, 19.09.02. CKJUNG
  ASSERTRT(hipMalloc((void**)&gtx_desc, sizeof(union ixgbe_adv_tx_desc)*512));
  ASSERTRT(hipMemset(gtx_desc, 0, sizeof(union ixgbe_adv_tx_desc)*512));


	unsigned int flag = 1;
	ASSERTDRV(hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t) d_pkt_buffer));
	ASSERTDRV(hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t) gtx_desc));

	my_t g = my_open();

	ASSERT_NEQ(g, (void*)0);

	uint64_t ret_dma_addr;
	my_mh_t mh;
	printf("[GDNIO]Pinning Pkt_Buffer in GDDR\n");
	if (my_pin_buffer(g, (hipDeviceptr_t)d_pkt_buffer, pkt_buffer_size, 0, 0, &mh, &ret_dma_addr)  != 0)
		OUT << "[my_pin_buffer] NOT_EQ" << endl;

	printf("[GDNIO]Pinning Tx_desc in GDDR\n");
	if (my_pin_desc(g, (hipDeviceptr_t)gtx_desc, sizeof(union ixgbe_adv_tx_desc), 0, 0, &mh)  != 0)
		OUT << "[my_pin_desc] NOT_EQ" << endl;

#if 0
	*ixgbe_bar0_host_addr = 0;
	*io_addr = 0;
	*tx_desc = 0;
	*rx_desc = 0;
	*tx_desc_addr = 0;
	*rx_desc_addr = 0;
#endif
	int fd = tx_rx_ring_setup();
	yhoon_initializer(fd, ixgbe_bar0_host_addr, tx_desc_addr, rx_desc_addr, &io_addr, &tx_desc, &rx_desc);
	
	hipStream_t cuda_stream1;
	hipStream_t cuda_stream5;
	hipStream_t cuda_stream6;
	ASSERT_CUDA(hipStreamCreateWithFlags(&cuda_stream1,hipStreamNonBlocking));
	ASSERT_CUDA(hipStreamCreateWithFlags(&cuda_stream5,hipStreamNonBlocking));
	ASSERT_CUDA(hipStreamCreateWithFlags(&cuda_stream6,hipStreamNonBlocking));
	
	// CKJUNG, 18.10.17 p_buf struct added.
	//int *pkt_cnt; // pkt_cnt[0]:RX, pkt_cnt[1]:TX
	//int *pkt_size;
	//struct pkt_buf *p_buf;
	//unsigned int *ctr;

  ASSERTRT(hipMalloc((void**)&pkt_cnt, sizeof(int)*2));
  ASSERTRT(hipMalloc((void**)&pkt_size, sizeof(int)));
  ASSERTRT(hipMalloc((void**)&p_buf, sizeof(struct pkt_buf)));
  ASSERTRT(hipMalloc((void**)&ctr, sizeof(unsigned int)));

	ASSERTRT(hipMalloc((void**)&gddr_dma_addr, sizeof(uint64_t)));
	ASSERTRT(hipMemcpy(gddr_dma_addr, &ret_dma_addr, sizeof(uint64_t), hipMemcpyHostToDevice));
	

	ASSERT_CUDA(hipMemset(pkt_cnt, 0, sizeof(int)*2));
	ASSERT_CUDA(hipMemset(pkt_size, 0, sizeof(int)));
	ASSERT_CUDA(hipMemset(p_buf, 0, sizeof(struct pkt_buf)));
	ASSERT_CUDA(hipMemset(ctr, 0, sizeof(unsigned int)));


  clean_buffer<<< 1, 1 >>> (d_pkt_buffer, pkt_buffer_size, p_buf);
	var_map<<<1, 1 >>> (p_buf, (volatile uint8_t*)io_addr, (volatile union ixgbe_adv_tx_desc*)tx_desc, gddr_dma_addr, pkt_cnt); 

	hipDeviceSynchronize();
#if 1
	START_RED
	printf("[GDNIO]#0: Rx_handler\n");
	END
	rx_handler<<< 1, 512, 0, cuda_stream1 >>> (p_buf, pkt_cnt, pkt_size, (volatile uint8_t*)io_addr, (volatile union ixgbe_adv_rx_desc*)rx_desc, gddr_dma_addr);
#endif
#if 0
	START_BLU
	printf("[GDNIO]#0: Tx_handler\n");
	END
	//tx_handler<<< 1, 512, 0, cuda_stream5 >>> (p_buf, pkt_cnt, (volatile uint8_t*)io_addr, (volatile union ixgbe_adv_tx_desc*)tx_desc, gddr_dma_addr); 
	tx_handler<<< 1, 512, 0, cuda_stream5 >>> ((union ixgbe_adv_tx_desc*)gtx_desc, (volatile uint8_t*)io_addr, pkt_cnt, gddr_dma_addr); 
#endif
#if 0
	START_YLW
	printf("[GDNIO]#0: tx_test\n");
	END
	tx_test<<< 1, 512, 0, cuda_stream6 >>> (pkt_cnt, (volatile uint8_t*)io_addr);
#endif

#if 0
	START_RED
	printf("[GDNIO]#0: writeler\n");
	END
	writeler<<< 1, 1, 0, cuda_stream5 >>> ();
#endif




	START_GRN
	printf("[Done]____[Initialize]__GDNIO__\n");
	END
}

extern "C"
void wait_for_gpu(void)
{
	hipDeviceSynchronize();
}

// returns a timestamp in nanoseconds
// based on rdtsc on reasonably configured systems and is hence fast
uint64_t monotonic_time() {
	struct timespec timespec;
	clock_gettime(CLOCK_MONOTONIC, &timespec);
	return timespec.tv_sec * 1000 * 1000 * 1000 + timespec.tv_nsec;
}

extern "C"
void monitoring_loop(void)
{
	START_GRN
		printf("Control is returned to CPU! :)\n");
	END
		// CKJUNG, 18.06.30
		struct timeval prev, cur;
	int prev_pkt[2] = {0,}, cur_pkt[2] = {0,};
	double pkts[2];
	char units[] = {' ', 'K', 'M', 'G', 'T'};
	char pps[2][40];
	char bps[2][40];
	int buf_idx[512] = {0,};
	int p_size=0;
	int i, j;
	int data[1024] = {0,};

	uint64_t last_stats_printed = monotonic_time();
	uint64_t time;
	// Replace "gettimeofday" to "clock_gettime", CKJUNG, 20.01.13
	//gettimeofday(&prev, NULL);
	while(1)                                           
	{
		time = monotonic_time();
		if(time - last_stats_printed > 1000 * 1000 * 1000){
			printf("elapsed time: %d\n", (time - last_stats_printed)/(1000*1000));
			last_stats_printed = time;
			//gettimeofday(&cur, NULL);
			//if(((cur.tv_sec - prev.tv_sec) * 1000000) + (cur.tv_usec - prev.tv_usec) > 1000000){
			//prev = cur;
			//printf("%p, %p\n", p_buf->tx_buf, d_pkt_buffer+(512*0x1000));
			hipError_t err = hipMemcpy(&cur_pkt[0], &pkt_cnt[0], sizeof(int), hipMemcpyDeviceToHost);
			hipError_t err2 = hipMemcpy(&cur_pkt[1], &pkt_cnt[1], sizeof(int), hipMemcpyDeviceToHost);
			hipError_t err3 = hipMemcpy(&p_size, pkt_size, sizeof(int), hipMemcpyDeviceToHost);
			hipError_t err4 = hipMemcpy(buf_idx, (p_buf->rx_buf_idx), sizeof(int)*512, hipMemcpyDeviceToHost);
			//hipError_t err5 = hipMemcpy(data, (p_buf->rx_buf), sizeof(int)*1024, hipMemcpyDeviceToHost);
			//hipError_t err5 = hipMemcpy(data, d_pkt_buffer+(512*0x1000), sizeof(int)*1024, hipMemcpyDeviceToHost);


			// CKJUNG, 18.08.07 For check
			//printf("Error-code of hipMemcpy: %d\n", err);
			//if(err != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess || err5 != hipSuccess)
			if(err != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess)
			{
				printf("hipMemcpy, pkt_cnt or buf_idx, error!\n");
			}
			system("clear");	
#if 0
			printf("[CKJUNG] buf #0\n");
			for(i = 0; i < 1024; i++){
				printf("%d ", data[i]);
			}
			printf("\n\n");
#endif
			for(i = 0; i < 2; i++){
				double tmp_pps;
				double tmp;
				double batch;
				if (prev_pkt[i] != cur_pkt[i]){ // If we got a traffic flow
					pkts[i] = (double)(cur_pkt[i] - prev_pkt[i]);

#if 0
					if(i == 0)
						printf("RX_pkts: %d\n", (int)pkts[i]); 
					else
						printf("TX_pkts: %d\n", (int)pkts[i]); 
#endif
					tmp = tmp_pps = pkts[i];
					//batch = tmp/BATCH;
					for(j = 0; tmp >= 1000 && j < sizeof(units)/sizeof(char) -1; j++)
						tmp /= 1000;
					sprintf(pps[i],"%.3lf %c" ,tmp, units[j]);

					//tmp = pkts[i] * p_size * 8; // Bytes -> Bits
					tmp = pkts[i] * p_size * 8 + tmp_pps * 20 * 8; // Add IFG also, 20.01.15, CKJUNG
					for(j = 0; tmp >= 1000 && j < sizeof(units)/sizeof(char) -1; j++)
						tmp /= 1000;
					sprintf(bps[i],"%.3lf %c" ,tmp, units[j]);

					if(i == 0)
						printf("[RX] pps: %spps %sbps, pkt_size: %d \n", pps[i], bps[i], p_size);
					else{
						printf("[TX] pps: %spps %sbps, pkt_size: %d \n", pps[i], bps[i], p_size);
					}
				}else{
					if(i == 0)
						printf("[RX] pps: None\n");
					else
						printf("[TX] pps: None\n");
				}
			}
			for(i = 0; i<512; i++)
			{
				if(i % 32 ==0)
					printf("\n");
				if(buf_idx[i] == 1){
					START_GRN
						printf("%d ", buf_idx[i]);
					END
				}else if(buf_idx[i] == 2){
					START_RED
						printf("%d ", buf_idx[i]);
					END
				}else if(buf_idx[i] == 3){
					START_BLU
						printf("%d ", buf_idx[i]);
					END
				}else{
					printf("%d ", buf_idx[i]);
				}
			}
			printf("\n");

			prev_pkt[0] = cur_pkt[0];
			prev_pkt[1] = cur_pkt[1];
		}
		//sleep(1); 
		}                                                                  
	}

__global__ void print_gpu(unsigned char* d_pkt_buf, int size)
{
	int i;
	printf("[GPU]:\n");
	for(i = 0; i < size; i++)
		printf("%02x ", d_pkt_buf[i]);
	printf("\n");
}


extern "C"
void copy_to_gpu(unsigned char* buf, int size)
{
	unsigned char *d_pkt_buf;
	hipMalloc((void**)&d_pkt_buf, sizeof(unsigned char)*1500);
	printf("____1__________copy_to_gpu____\n");
	//hipMemcpy(&p_buf->rx_buf+(0x1000*idx), buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_pkt_buffer+(512*0x1000)+(0x1000*idx), buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
	print_gpu<<<1,1>>>(d_pkt_buffer+(512*0x1000)+(0x1000*idx), sizeof(unsigned char)*size);
	//printf("p_buf->rx_buf: %p\n", p_buf->rx_buf);
	//hipMemcpy(p_buf->rx_buf, buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
	idx++;
	if(idx == 512)
		idx = 0;
	printf("____2__________copy_to_gpu____\n");
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t _pkt_buffer_size = 2*512*4096; // 4MB, for rx,tx ring
	size_t pkt_buffer_size = (_pkt_buffer_size + GPU_PAGE_SIZE - 1) & GPU_PAGE_MASK;
	
  ASSERTRT(hipMalloc((void**)&d_pkt_buffer, pkt_buffer_size));
  ASSERTRT(hipMemset(d_pkt_buffer, 0, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&pkt_cnt, sizeof(int)*2));
  ASSERTRT(hipMalloc((void**)&pkt_size, sizeof(int)));
  ASSERTRT(hipMalloc((void**)&p_buf, sizeof(struct pkt_buf)));
  ASSERTRT(hipMalloc((void**)&ctr, sizeof(unsigned int)));


	ASSERT_CUDA(hipMemset(pkt_cnt, 0, sizeof(int)*2));
	ASSERT_CUDA(hipMemset(pkt_size, 0, sizeof(int)));
	ASSERT_CUDA(hipMemset(p_buf, 0, sizeof(struct pkt_buf)));
	ASSERT_CUDA(hipMemset(ctr, 0, sizeof(unsigned int)));

  clean_buffer<<< 1, 1 >>> (d_pkt_buffer, pkt_buffer_size, p_buf);

	START_GRN
	printf("[Done]____GPU mem set for dpdk__\n");
	END
}
