#include "hip/hip_runtime.h"
#include "my_handler.h"
#include "gdnio.h"
#include "packet_man.h"
#include "common.hpp"
#include "mydrv/mydrv.h"
#include "pkts.h"

#define PKT_SIZE 64

#define OUT cout
using namespace std;

unsigned char* d_pkt_buffer;
// 19.09.02. CKJUNG
struct pkt_buf *p_buf;
int *pkt_cnt;
int *pkt_size;          
unsigned int *ctr; // used in ipsec? 19.06.27      

static int idx;

/*
__device__ uint8_t tmp_pkt[60] = {\
0x00, 0x1b, 0x21, 0xbc, 0x11, 0x52, 0xa0, 0x36, 0x9f, 0x03, 0x13, 0x86, 0x08, 0x00, 0x45, 0x10,\
0x00, 0x2e, 0x00, 0x00, 0x40, 0x00, 0x40, 0x11, 0x00, 0x00, 0x0a, 0x00, 0x00, 0x03, 0x0a, 0x00,\
0x00, 0x04, 0x04, 0xd2, 0x04, 0xd2, 0x00, 0x1a, 0x2c, 0xd6, 0x6f, 0x98, 0x26, 0x35, 0x02, 0xc9,\
0x83, 0xd7, 0x8b, 0xc3, 0xf7, 0xb5, 0x20, 0x8d, 0x48, 0x8d, 0xc0, 0x36};
*/

int my_pin_buffer(my_t g, unsigned long addr, size_t size, uint64_t p2p_token, uint32_t va_space, my_mh_t *handle, uint64_t *ret_dma_addr)
{
    int ret = 0;
    int retcode;

    struct MYDRV_IOC_PIN_BUFFER_PARAMS params;
    params.addr = addr;
    params.size = size;
    params.p2p_token = p2p_token;
    params.va_space = va_space;
    params.handle = 0;
    //params.buf_name = bname;

    retcode = ioctl(g->fd, MYDRV_IOC_PIN_BUFFER, &params);
    if (0 != retcode) {
        ret = errno;
        my_err("ioctl error (errno=%d)\n", ret);
    }
    *handle = params.handle;

		// 19.07.17. CKJUNG                                        
		START_YLW                                                  
			printf("[CKJUNG] ret_dma_addr: %p\n", params.ret_dma_addr);
		END                                                        

		*ret_dma_addr = params.ret_dma_addr;

    return ret;
}

my_t my_open()
{
    my_t m = NULL;
    //const char *myinode = "/dev/mydrv";
    const char *myinode = "/dev/ixgbe";

    m = (my_t) calloc(1, sizeof(*m));
    if (!m) {
        //my_err("error while allocating memory\n");
        return NULL;
    }

    int fd = open(myinode, O_RDWR);
    if (-1 == fd ) {
        int ret = errno;
        //my_err("error opening driver (errno=%d/%s)\n", ret, strerror(ret));
        free(m);
        return NULL;
    }

    m->fd = fd;

    return m;
}

__device__ int clean_index;
__device__ int tx_index;
__device__ int sendable;


__global__ void clean_buffer(unsigned char* buffer, int size, struct pkt_buf *p_buf) 
{
  //for(int i=0; i<size; i++) {
	// CKJUNG 18.03.01
  for(int i=0; i<size; i++) {
    buffer[i] = 0;
  }

	// 19.06.25. Mapping "p_buf" to "d_pkt_buffer"
	p_buf->tx_buf = buffer;
	p_buf->rx_buf = p_buf->tx_buf + offset_for_rx;

#if 1
	for(int i=0; i<512; i++) {
		switch(PKT_SIZE){
			case 64:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_60B, PKT_SIZE);
				break;
			case 128:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_124B, PKT_SIZE);
				break;
			case 256:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_252B, PKT_SIZE);
				break;
			case 512:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_508B, PKT_SIZE);
				break;
			case 1024:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_1020B, PKT_SIZE);
				break;
			case 1514:
				memcpy(&p_buf->tx_buf[0x1000 * i], pkt_1510B, PKT_SIZE);
				break;
		}
	}
#endif

	// Initialize "Tx" related indices.
	clean_index = 0;
	tx_index = 0;
	sendable = 1;

	printf("%p, %p\n", p_buf->tx_buf, buffer+(512*0x1000));
	
	START_GRN
		printf("[Done]____clean_buffer__\n");
	END

}

extern "C"
void wait_for_gpu(void)
{
	hipDeviceSynchronize();
}

// returns a timestamp in nanoseconds
// based on rdtsc on reasonably configured systems and is hence fast
uint64_t monotonic_time() {
	struct timespec timespec;
	clock_gettime(CLOCK_MONOTONIC, &timespec);
	return timespec.tv_sec * 1000 * 1000 * 1000 + timespec.tv_nsec;
}

__global__ void print_gpu(unsigned char* d_pkt_buf, int size)
{
	int i;
	printf("[GPU]:\n");
	for(i = 0; i < size; i++)
		printf("%02x ", d_pkt_buf[i]);
	printf("\n");
}


extern "C"
void copy_to_gpu(unsigned char* buf, int size)
{
	unsigned char *d_pkt_buf;
	hipMalloc((void**)&d_pkt_buf, sizeof(unsigned char)*1500);
	printf("____1__________copy_to_gpu____\n");
	//hipMemcpy(&p_buf->rx_buf+(0x1000*idx), buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_pkt_buffer+(512*0x1000)+(0x1000*idx), buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
	print_gpu<<<1,1>>>(d_pkt_buffer+(512*0x1000)+(0x1000*idx), sizeof(unsigned char)*size);
	//printf("p_buf->rx_buf: %p\n", p_buf->rx_buf);
	//hipMemcpy(p_buf->rx_buf, buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
	idx++;
	if(idx == 512)
		idx = 0;
	printf("____2__________copy_to_gpu____\n");
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t _pkt_buffer_size = 2*512*4096; // 4MB, for rx,tx ring
	size_t pkt_buffer_size = (_pkt_buffer_size + GPU_PAGE_SIZE - 1) & GPU_PAGE_MASK;
	
  ASSERTRT(hipMalloc((void**)&d_pkt_buffer, pkt_buffer_size));
  ASSERTRT(hipMemset(d_pkt_buffer, 0, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&pkt_cnt, sizeof(int)*2));
  ASSERTRT(hipMalloc((void**)&pkt_size, sizeof(int)));
  ASSERTRT(hipMalloc((void**)&p_buf, sizeof(struct pkt_buf)));
  ASSERTRT(hipMalloc((void**)&ctr, sizeof(unsigned int)));


	ASSERT_CUDA(hipMemset(pkt_cnt, 0, sizeof(int)*2));
	ASSERT_CUDA(hipMemset(pkt_size, 0, sizeof(int)));
	ASSERT_CUDA(hipMemset(p_buf, 0, sizeof(struct pkt_buf)));
	ASSERT_CUDA(hipMemset(ctr, 0, sizeof(unsigned int)));

  clean_buffer<<< 1, 1 >>> (d_pkt_buffer, pkt_buffer_size, p_buf);

	START_GRN
	printf("[Done]____GPU mem set for dpdk__\n");
	END
}
